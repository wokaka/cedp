#include "hip/hip_runtime.h"
#ifndef __GPUFI_KERNEL__
#define __GPUFI_KERNEL__

#include <stdio.h>

#define MAX2(x,y) ((x>y)?x:y)
#define MAX3(x,y,z) ((x>y)?MAX2(x,z):MAX2(y,z))

struct _gpufi_data_ gpufi_host = {0, };
__device__ struct _gpufi_data_ *gpufi_dev;


#if GPUFI_FUNC_INIT
////////////////////////////////////////////////////
// CPU PART
////////////////////////////////////////////////////

int variable_count;
int kernel_count;

/*
 * @about   Fetch a fault injection command (including fault type and time)
 *          from a file, namely fi_cmd.txt
 */
int GPUFI_INIT(int kernel_cnt, int variable_cnt)
{
    FILE    *fp;
    char    cmd[32];
		int i;
	
    kernel_count = kernel_cnt + 1;
    variable_count = variable_cnt;

    memset(&gpufi_host, 0, sizeof(struct _gpufi_data_));

    fp = fopen("fi_cmd.txt", "rt");

    if(!fp){
        printf("file open error\n");
        system("pwd");
        return -1;
    }

    fscanf(fp, "%s", cmd);
    printf("%s ", cmd);
    if(!strcmp(cmd, "profile")){
        gpufi_host.current.blid = 0;
        gpufi_host.current.thid = 0;

        gpufi_host.fault.mode = GPUFI_PROFILE;
        gpufi_host.current.profile_index = -1;
        fscanf(fp, "%s", cmd);
	      printf("%s ", cmd);
        if(!strcmp(cmd, "none")){
	        gpufi_host.current.profile_mode = PROFILE_MODE_NONE;
	      }
        else if(!strcmp(cmd, "value")){
	        fscanf(fp, "%s", cmd);
  	      printf("%s ", cmd);
					if(!strcmp(cmd, "loop"))
		        gpufi_host.current.profile_mode = PROFILE_MODE_VALUE_LOOP;
	        else if(!strcmp(cmd, "kernel"))
		        gpufi_host.current.profile_mode = PROFILE_MODE_VALUE_KERNEL;
	        else if(!strcmp(cmd, "thread")){
		        gpufi_host.current.profile_mode = PROFILE_MODE_VALUE_THREAD;
		        fscanf(fp, "%d", &gpufi_host.current.blid);
	      		printf(" %d ", gpufi_host.current.blid);
	        }
	        else if(!strcmp(cmd, "block")){
		        gpufi_host.current.profile_mode = PROFILE_MODE_VALUE_BLOCK;
		      }
	      }
	      printf(" (%d)\n", gpufi_host.current.profile_mode);
    }

    fclose(fp);

    return 0;
}

#else 

int GPUFI_INIT(int x, int y)
{
    return 0;
}

#endif


#if GPUFI_FUNC_HALT

char *gpufi_type_name[6] = {"unknown", "int", "fp", "int_pt", "fp_pt", "unknown_pt"};

int GPUFI_HALT(char *fname)
{
    FILE    *fp;
    int i, j, k, m;

    if(gpufi_host.fault.mode == GPUFI_PROFILE){
        //char    cmd[32];

        fp = fopen(fname, "wt");
        if(!fp){
        	  printf("can't write to an output file, %s\n", fname);
            return -1;
        }

        for(i=0; i<MAX_KERNEL; i++){
            if(gpufi_host.profile.kernel[i].state == 1)
                fprintf(fp, "kernel\t%d\t%s\t%d\n",
                        gpufi_host.profile.kernel[i].id,
                        gpufi_host.profile.kernel[i].name,
                        gpufi_host.profile.kernel[i].instance);
            else
                break;

            for(j=0; j<variable_count; j++){
                if(gpufi_host.profile.variable[i][j].call_count != 0){
                  fprintf(fp, "variable\t%d\t%s\t%d\t%d\t%s\t",
                          j,
             							gpufi_host.profile.variable_bitmap[j]?gpufi_host.profile.variable_name[j]:"",
                          gpufi_host.profile.variable[i][j].call_count,
                          gpufi_host.profile.variable[i][j].loop_id,
                          gpufi_type_name[gpufi_host.profile.variable[i][j].type]);

				        		fprintf(fp, "\n");
								}
            }
            fprintf(fp, "\n");
        }

        fclose(fp);
    }

    return 0;
}

#else


int GPUFI_HALT(char *x)
{
    return 0;
}


#endif



////////////////////////////////////////////////////
// DEVICE PART
////////////////////////////////////////////////////

#if GPUFI_FUNC_LIB

__device__
void GPUFI_KERNEL_DEC(int *count)
{
    if(*count != 0)
        *count--;
    // else
    /* be ready to inject fault when GPUFI_KERNEL_INJECT is called */
}

__device__
void GPUFI_KERNEL_SET(int *count, int value)
{
    *count = value;
}

/*
 * @about   Event generated when a GPU kernel is executed
 */
__device__
void CudaStrcpy(char *dst, char *src, int max)
{
    int i;
	
    for(i=0; src[i]!=0 && i<max-1; i++)
        dst[i] = src[i];
    dst[i] = 0;
}

__device__
int GPUFI_STRCMP(char *src, char *dst)
{
	int	i;

	for(i=0; src[i] && dst[i]; i++){
		if(src[i] != dst[i])
			return 1;
	}
	
	if(src[i] == dst[i])
		return 0;

	return 1;
}

__device__
void GPUFI_DEBUG_LOC(struct _gpufi_data_ *gpufi_dev)
{
#if 0
    if(blockIdx.x != gpufi_dev->current.blid || threadIdx.x != gpufi_dev->current.thid)
        return;

	if(!gpufi_dev->debug.injected){
								gpufi_dev->debug.injected = 1;
								gpufi_dev->debug.kernel = gpufi_dev->current.kernel;
								gpufi_dev->debug.instance = gpufi_dev->current.instance;
								gpufi_dev->debug.loop = gpufi_dev->current.loop;
								gpufi_dev->debug.iteration = gpufi_dev->current.iteration;
	}
#endif
}

__device__ 
int GPUFI_EXIT(int errno)
{
	int *addr = 0x0;
	
	*addr = 0x80;
	return 100 / errno;
}

#else

__device__
int GPUFI_STRCMP(char *src, char *dst)
{
	return 1;
}

__device__
void CudaStrcpy(char *dst, char *src, int max)
{
}

__device__ int GPUFI_EXIT(int errno)
{
	return 0;
}

__device__
void GPUFI_KERNEL_DEC(int *count)
{
}

__device__
void GPUFI_KERNEL_SET(int *count, int value)
{
}

#endif

#if GPUFI_FUNC_KERNEL

__device__
void GPUFI_KERNEL(struct _gpufi_data_ *gpufi_dev, int begin, int type, char *name)
{
    if(blockIdx.x != gpufi_dev->current.blid || threadIdx.x != gpufi_dev->current.thid)
        return;

    if(gpufi_dev->fault.mode == GPUFI_PROFILE){
        if(begin == GPUFI_KERNEL_BEGIN){
            /* 1: update current */
            
            if(gpufi_dev->profile.kernel_bitmap[type] == 0)
                gpufi_dev->profile.kernel_bitmap[type] = 1; /* kernel */
            gpufi_dev->profile.kernel_instance[type]++; /* instance */

            /* update current */
            gpufi_dev->current.kernel = type;
            gpufi_dev->current.instance = gpufi_dev->profile.kernel_instance[type] - 1;
            gpufi_dev->current.loop = -1;

            /* 2: profiling */
           	gpufi_dev->current.profile_index++;

						if(gpufi_dev->current.profile_index < MAX_KERNEL){
	            gpufi_dev->profile.kernel[gpufi_dev->current.profile_index].state  = 1;
	            gpufi_dev->profile.kernel[gpufi_dev->current.profile_index].id  = gpufi_dev->current.kernel;
	            CudaStrcpy(gpufi_dev->profile.kernel[gpufi_dev->current.profile_index].name, name, MAX_NAME);
	            gpufi_dev->profile.kernel[gpufi_dev->current.profile_index].instance  = gpufi_dev->current.instance;
					  }
        }
    }
}

#else

// DEVICE PART

__device__
void GPUFI_KERNEL(struct _gpufi_data_ *gpufi_dev, int begin, int type, char *name)
{
}

#endif


#if GPUFI_FUNC_LOOP

__device__
void GPUFI_KERNEL_LOOP(struct _gpufi_data_ *gpufi_dev, int begin)
{
    if(blockIdx.x != gpufi_dev->current.blid || threadIdx.x != gpufi_dev->current.thid)
        return;

    if(gpufi_dev->fault.mode == GPUFI_PROFILE){
        if(begin == GPUFI_LOOP_BEGIN){
            //gpufi_dev->profile.loop[gpufi_dev->current.kernel][gpufi_dev->current.instance]++;
            gpufi_dev->current.loop = gpufi_dev->current.loop_count;// = gpufi_dev->profile.loop[gpufi_dev->current.kernel][gpufi_dev->current.instance] - 1;
            gpufi_dev->current.iteration = -1;
            gpufi_dev->current.loop_count++;
        }
        else{
            //gpufi_dev->profile.loop[gpufi_dev->current.kernel][gpufi_dev->current.instance]++;
            gpufi_dev->current.loop = -1;
        }
    }
}

__device__
void GPUFI_KERNEL_ITERATION(struct _gpufi_data_ *gpufi_dev)
{
    if(blockIdx.x != gpufi_dev->current.blid || threadIdx.x != gpufi_dev->current.thid)
        return;

    if(gpufi_dev->fault.mode == GPUFI_PROFILE){
//        gpufi_dev->profile.iteration[gpufi_dev->current.kernel][gpufi_dev->current.instance][gpufi_dev->current.loop]++;
        gpufi_dev->current.iteration++;
    }
}

#else

__device__
void GPUFI_KERNEL_LOOP(struct _gpufi_data_ *gpufi_dev, int begin)
{
}

__device__
void GPUFI_KERNEL_ITERATION(struct _gpufi_data_ *gpufi_dev)
{
}

#endif


#if GPUFI_FUNC_VARIABLE

__device__
void GPUFI_KERNEL_VARIABLE(struct _gpufi_data_ *gpufi_dev, int varid, char *name, int *variable, int var_type)
{
    int i;
    if(blockIdx.x != gpufi_dev->current.blid || threadIdx.x != gpufi_dev->current.thid)
        return;

    if(gpufi_dev->fault.mode == GPUFI_PROFILE){
    		if(gpufi_dev->current.profile_index < MAX_KERNEL && varid < MAX_VARIABLE){
	        	atomicAdd(&gpufi_dev->profile.variable[gpufi_dev->current.profile_index][varid].call_count, 1);
	        	if(gpufi_dev->profile.variable_bitmap[varid] == 0){
	        		gpufi_dev->profile.variable_bitmap[varid] = 1;
	        		CudaStrcpy(gpufi_dev->profile.variable_name[varid], name, MAX_NAME);
	        	}
	        	gpufi_dev->profile.variable[gpufi_dev->current.profile_index][varid].loop_id = gpufi_dev->current.loop;
	        	gpufi_dev->profile.variable[gpufi_dev->current.profile_index][varid].type = var_type;
      	}
    }
}

__device__
void GPUFI_KERNEL_VARIABLE_STREAM(struct _gpufi_data_ *gpufi_dev, int varid, char *name, int *variable, int length, int count)
{
#if 0
    int i;

    if(blockIdx.x != gpufi_dev->current.blid || threadIdx.x != gpufi_dev->current.thid)
        return;

    if(gpufi_dev->fault.mode == GPUFI_PROFILE){
//        gpufi_dev->profile.variable[gpufi_dev->current.kernel][gpufi_dev->current.instance][gpufi_dev->current.loop][varid]++;
    }
#endif
}

#else

__device__
void GPUFI_KERNEL_VARIABLE(struct _gpufi_data_ *gpufi_dev, int varid, char *name, int *variable, int var_type)
{
}

__device__
void GPUFI_KERNEL_VARIABLE_FORCE(struct _gpufi_data_ *gpufi_dev, int varid, char *name, int *variable)
{
}

#endif

#endif