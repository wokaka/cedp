/* Instrumented by FaultInjector */
#include "gpufi.h"

/*
**************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 **************************************************************************
*/
#include "gpufi.h"

#include <stdio.h>

#include <stdlib.h>

#include <sys/time.h>

#include <inttypes.h>

#include "parboil.h"

#include "hip/hip_runtime.h"

#include "sad.h"

#include "sad4.h"

#include "largerBlocks.h"

#include "file.h"

#include "image.h"

#include "largerBlocks.cu"

#include "sad4.cu"

#include "gpufi.h"

static unsigned short * load_sads(char * filename);
static void write_sads(char * filename, int image_size_macroblocks, unsigned short * sads);
static void write_sads_directly(char * filename, int width, int height, unsigned short * sads);
/* FILE I/O */
unsigned short * load_sads(char * filename)
{
    FILE * infile;
    unsigned short * sads;
    int w;
    int h;
    int sads_per_block;
    infile=fopen(filename, "r");
    if (( ! infile))
    {
        fprintf(stderr, "Cannot find file '%s'\n", filename);
        exit(( - 1));
    }
    /* Read image dimensions (measured in macroblocks) */
    w=read16u(infile);
    h=read16u(infile);
    /*
    Read SAD values.  Only interested in the 4x4 SAD values, which are
       * at the end of the file.
    */
    sads_per_block=(1096*(w*h));
    fseek(infile, ((25*sads_per_block)*sizeof (unsigned short)), 1);
    sads=((unsigned short * )malloc(((sads_per_block*16)*sizeof (unsigned short))));
    fread(sads, sizeof (unsigned short), (sads_per_block*16), infile);
    fclose(infile);
    return sads;
}

/*
Compare the reference SADs to the expected SADs.
 
*/
void check_sads(unsigned short * sads_reference, unsigned short * sads_computed, int image_size_macroblocks)
{
    int block;
    /*
    Check the 4x4 SAD values.  These are in sads_reference.
       * Ignore the data at the beginning of sads_computed.
    */
    sads_computed+=((25*1096)*image_size_macroblocks);
    for (block=0; block<image_size_macroblocks; block ++ )
    {
        int subblock;
        for (subblock=0; subblock<16; subblock ++ )
        {
            int sad_index;
            for (sad_index=0; sad_index<1089; sad_index ++ )
            {
                int index = ((((block*16)+subblock)*1096)+sad_index);
                if ((sads_reference[index]!=sads_computed[index]))
                {
                    goto mismatch;
                }
            }
        }
    }
    printf("Success.\n");
    return ;
    {
mismatch:
        printf("Computed SADs do not match expected values.\n");
    }
}

/*
Extract the SAD data for a particular block type for a particular
 * macroblock from the array of SADs of that block type.
*/
static inline void write_subblocks(FILE * outfile, unsigned short * subblock_array, int macroblock, int count)
{
    int block;
    int pos;
    for (block=0; block<count; block ++ )
    {
        unsigned short * vec = (subblock_array+((block+(macroblock*count))*1096));
        /* Write all SADs for this sub-block */
        for (pos=0; pos<1089; pos ++ )
        {
            write16u(outfile, ( * (vec ++ )));
        }
    }
}

void write_sads(char * filename, int image_size_macroblocks, unsigned short * sads)
{
    FILE * outfile = fopen(filename, "w");
    int block;
    if ((outfile==NULL))
    {
        fprintf(stderr, "Cannot open output file\n");
        exit(( - 1));
    }
    /* Write size in macroblocks */
    write32u(outfile, image_size_macroblocks);
    /* Write zeros */
    write32u(outfile, 0);
    /* Each macroblock */
    for (block=0; block<image_size_macroblocks; block ++ )
    {
        int blocktype;
        /* Write SADs for all sub-block types */
        for (blocktype=1; blocktype<=7; blocktype ++ )
        {
            write_subblocks(outfile, (sads+((blocktype==1) ? 0 : ((blocktype==2) ? (image_size_macroblocks*1096) : ((blocktype==3) ? (image_size_macroblocks*(3*1096)) : ((blocktype==4) ? (image_size_macroblocks*(5*1096)) : ((blocktype==5) ? (image_size_macroblocks*(9*1096)) : ((blocktype==6) ? (image_size_macroblocks*(17*1096)) : (image_size_macroblocks*(25*1096))))))))), block, ((blocktype==1) ? 1 : ((blocktype==2) ? 2 : ((blocktype==3) ? 2 : ((blocktype==4) ? 4 : ((blocktype==5) ? 8 : ((blocktype==6) ? 8 : 16)))))));
        }
    }
    fclose(outfile);
}

/* FILE I/O for debugging */
static void write_sads_directly(char * filename, int width, int height, unsigned short * sads)
{
    FILE * f = fopen(filename, "w");
    int n;
    write16u(f, width);
    write16u(f, height);
    for (n=0; n<((41*1096)*(width*height)); n ++ )
    {
        write16u(f, sads[n]);
    }
    fclose(f);
}

static void print_test_sad_vector(unsigned short * base, int macroblock, int count)
{
    int n;
    int searchpos = ((17*33)+17);
    for (n=0; n<count; n ++ )
    {
        printf(" %d", base[((((count*macroblock)+n)*1096)+searchpos)]);
    }
}

static void print_test_sads(unsigned short * sads_computed, int mbs)
{
    int macroblock = 5;
    int blocktype;
    for (blocktype=1; blocktype<=7; blocktype ++ )
    {
        printf("%d:", blocktype);
        print_test_sad_vector((sads_computed+((blocktype==1) ? 0 : ((blocktype==2) ? (mbs*1096) : ((blocktype==3) ? (mbs*(3*1096)) : ((blocktype==4) ? (mbs*(5*1096)) : ((blocktype==5) ? (mbs*(9*1096)) : ((blocktype==6) ? (mbs*(17*1096)) : (mbs*(25*1096))))))))), macroblock, ((blocktype==1) ? 1 : ((blocktype==2) ? 2 : ((blocktype==3) ? 2 : ((blocktype==4) ? 4 : ((blocktype==5) ? 8 : ((blocktype==6) ? 8 : 16)))))));
        puts("\n");
    }
}

/* MAIN */
int stream_data = 0;
int main(int argc, char *  * argv)
{
    struct image_i16 * ref_image;
    struct image_i16 * cur_image;
    unsigned short * sads_computed;
    /* SADs generated by the program */
    GPUFI_INIT(3, 129);


    int image_size_bytes;
    int image_width_macroblocks, image_height_macroblocks;
    int image_size_macroblocks;
    struct pb_TimerSet timers;
    struct pb_Parameters * params;
    pb_InitializeTimerSet(( & timers));
    params=pb_ReadParameters(( & argc), argv);
    if ((pb_Parameters_CountInputs(params)!=2))
    {
        fprintf(stderr, "Expecting two input filenames\n");
        exit(( - 1));
    }
    /* Read input files */
    pb_SwitchToTimer(( & timers), pb_TimerID_IO);
    ref_image=load_image(params->inpFiles[0]);
    cur_image=load_image(params->inpFiles[1]);
    pb_SwitchToTimer(( & timers), pb_TimerID_COMPUTE);
    if (((ref_image->width!=cur_image->width)||(ref_image->height!=cur_image->height)))
    {
        fprintf(stderr, "Input images must be the same size\n");
        exit(( - 1));
    }
    if (((ref_image->width%16)||(ref_image->height%16)))
    {
        fprintf(stderr, "Input image size must be an integral multiple of 16\n");
        exit(( - 1));
    }
    /* Compute parameters, allocate memory */
    image_size_bytes=((ref_image->width*ref_image->height)*sizeof (short));
    image_width_macroblocks=(ref_image->width>>4);
    image_height_macroblocks=(ref_image->height>>4);
    image_size_macroblocks=(image_width_macroblocks*image_height_macroblocks);
    sads_computed=((unsigned short * )malloc((((41*1096)*image_size_macroblocks)*sizeof (short))));
    /* Run the kernel code */
    {
        struct hipArray * ref_ary;
        /* Reference image on the device */
        short * d_cur_image;
        /* Current image on the device */
        unsigned short * d_sads;
        /* SADs on the device */
        dim3 macroblock_grid( image_width_macroblocks,  image_height_macroblocks);
        pb_SwitchToTimer(( & timers), pb_TimerID_COPY);
        hipMalloc(((void *  * )( & d_cur_image)), image_size_bytes);
        stream_data+=image_size_bytes;
        {
            hipError_t err = hipGetLastError();
            if (err)
            {
                fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        hipMallocArray(( & ref_ary), ( & get_ref().channelDesc), ref_image->width, ref_image->height);
        stream_data+=(ref_image->width*ref_image->height);
        {
            hipError_t err = hipGetLastError();
            if (err)
            {
                fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        /* Transfer current image to device */
        hipMemcpy(d_cur_image, cur_image->data, image_size_bytes, hipMemcpyHostToDevice);
        {
            hipError_t err = hipGetLastError();
            if (err)
            {
                fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        /* Transfer reference image to device */
        hipMemcpy2DToArray(ref_ary, 0, 0, ref_image->data, (ref_image->width*sizeof (unsigned short)), (ref_image->width*sizeof (unsigned short)), ref_image->height, hipMemcpyHostToDevice);
        {
            hipError_t err = hipGetLastError();
            if (err)
            {
                fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        hipBindTextureToArray(get_ref(), ref_ary);
        {
            hipError_t err = hipGetLastError();
            if (err)
            {
                fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        /* Allocate SAD data on the device */
        hipMalloc(((void *  * )( & d_sads)), (((41*1096)*image_size_macroblocks)*sizeof (unsigned short)));
        stream_data+=(((41*1096)*image_size_macroblocks)*sizeof (unsigned short));
        printf("stream_data %d\n", stream_data);
        {
            hipError_t err = hipGetLastError();
            if (err)
            {
                fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        hipMemset(d_sads, 0, (((41*1096)*image_size_macroblocks)*sizeof (unsigned short)));
        {
            hipError_t err = hipGetLastError();
            if (err)
            {
                fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        if (params->synchronizeGpu)
        {
            hipDeviceSynchronize();
        }
        pb_SwitchToTimer(( & timers), pb_TimerID_GPU);
        /* Run the 4x4 kernel */
        /* Fault Injection Instrumentation Begin */
        hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
        CUDA_ERRCK
        hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

        mb_sad_calc<<<dim3((((ref_image->width/4)+(1-1))/1), (((ref_image->height/4)+(1-1))/1)), dim3(((((1089+(18-1))/18)*1)*1)), (((1*1)*1096)*sizeof (unsigned short))>>> (d_sads, ((unsigned short * )d_cur_image), image_width_macroblocks, image_height_macroblocks, gpufi_dev);
        /* Fault Injection Instrumentation Begin */
        hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
        CUDA_ERRCK
        hipFree(gpufi_dev);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

        {
            hipError_t err = hipGetLastError();
            if (err)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        /* Run the larger-blocks kernels */
        /* Fault Injection Instrumentation Begin */
        hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
        CUDA_ERRCK
        hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

        larger_sad_calc_8<<<macroblock_grid, dim3(32, 4)>>> (d_sads, image_width_macroblocks, image_height_macroblocks, gpufi_dev);
        /* Fault Injection Instrumentation Begin */
        hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
        CUDA_ERRCK
        hipFree(gpufi_dev);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

        {
            hipError_t err = hipGetLastError();
            if (err)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        /* Fault Injection Instrumentation Begin */
        hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
        CUDA_ERRCK
        hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

        larger_sad_calc_16<<<macroblock_grid, dim3(32, 1)>>> (d_sads, image_width_macroblocks, image_height_macroblocks, gpufi_dev);
        /* Fault Injection Instrumentation Begin */
        hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
        CUDA_ERRCK
        hipFree(gpufi_dev);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

        {
            hipError_t err = hipGetLastError();
            if (err)
        {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        if (params->synchronizeGpu)
        {
            hipDeviceSynchronize();
        }
        pb_SwitchToTimer(( & timers), pb_TimerID_COPY);
        /* Transfer SAD data to the host */
        /* + 25 * MAX_POS_PADDED * image_size_macroblocks,
         */
        /* + 25 * MAX_POS_PADDED * image_size_macroblocks,
         */
        hipMemcpy(sads_computed, d_sads, (((41*1096)*image_size_macroblocks)*sizeof (unsigned short)), hipMemcpyDeviceToHost);
        {
            hipError_t err = hipGetLastError();
            if (err)
            {
                fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        /* Free GPU memory */
        hipFree(d_sads);
        {
            hipError_t err = hipGetLastError();
            if (err)
            {
                fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        hipUnbindTexture(get_ref());
        {
            hipError_t err = hipGetLastError();
            if (err)
            {
                fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        hipFreeArray(ref_ary);
        {
            hipError_t err = hipGetLastError();
            if (err)
            {
                fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        hipFree(d_cur_image);
        {
            hipError_t err = hipGetLastError();
            if (err)
            {
                fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            }
        }
        if (params->synchronizeGpu)
        {
            hipDeviceSynchronize();
        }
        pb_SwitchToTimer(( & timers), pb_TimerID_COMPUTE);
    }
    /* Print output */
    if (params->outFile)
    {
        pb_SwitchToTimer(( & timers), pb_TimerID_IO);
        write_sads(params->outFile, image_size_macroblocks, sads_computed);
        pb_SwitchToTimer(( & timers), pb_TimerID_COMPUTE);
    }
    /* Free memory */
    free(sads_computed);
    free_image(ref_image);
    free_image(cur_image);
    pb_SwitchToTimer(( & timers), pb_TimerID_NONE);
    pb_PrintTimerSet(( & timers));
    pb_FreeParameters(params);
    GPUFI_HALT("fi_profile.txt");
    return 0;
    GPUFI_HALT("fi_profile.txt");
}
