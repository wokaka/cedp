#include "hip/hip_runtime.h"
/* Instrumented by FaultInjector */
#include "gpufi.h"

/*
**************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 **************************************************************************
*/
#include "gpufi.h"

#include "sad.h"

#include "sad4.h"

/* Macros to access temporary frame storage in shared memory */
/* Macros to access temporary SAD storage in shared memory */
/*
When reading from this array, we use an "index" rather than a
   search position.  Also, the number of array elements is divided by
   four relative to SAD_LOC_GET() since this is an array of 8byte
   data, while SAD_LOC_GET() sees an array of 2byte data.
*/
/*
The size of one row of sad_loc_8b.  This is the group of elements
 * holding SADs for all search positions for one 4x4 block.
*/
/*
The presence of this preprocessor variable controls which
 * of two means of computing the current search position is used.
*/
/* A local copy of the current 4x4 block */
__shared__ unsigned short frame_loc[((1*1)*16)];
/* The part of the reference image that is in the search range */
texture<unsigned short, 2, hipReadModeElementType> ref;
/*
The local SAD array on the device.  This is an array of short ints.  It is
 * interpreted as an array of 8-byte data for global data transfers.
*/
extern __shared__ unsigned short sad_loc[];
extern __shared__ struct vec8b sad_loc_8b[];
/* The compute kernel. */
/*
The macros THREADS_W and THREADS_H specify the width and height of the
 * area to be processed by one thread, measured in 4-by-4 pixel blocks.
 * Larger numbers mean more computation per thread block.
 *
 * The macro POS_PER_THREAD specifies the number of search positions for which
 * an SAD is computed.  A larger value indicates more computation per thread,
 * and fewer threads per thread block.  It must be a multiple of 3 and also
 * must be at most 33 because the loop to copy from shared memory uses
 * 32 threads per 4-by-4 pixel block.
 *
 
*/
__global__ void mb_sad_calc(unsigned short * blk_sad, unsigned short * frame, int mb_width, int mb_height,  struct _gpufi_data_ *gpufi_dev)
{
    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_BEGIN, 0, "mb_sad_calc");
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 0, "blk_sad", (int*)&blk_sad, GPUFI_DATATYPE_UNKNOWN_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 1, "frame", (int*)&frame, GPUFI_DATATYPE_UNKNOWN_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 2, "mb_width", (int*)&mb_width, GPUFI_DATATYPE_INTEGER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 3, "mb_height", (int*)&mb_height, GPUFI_DATATYPE_INTEGER);


    int txy_tmp = (threadIdx.x/((1089+(18-1))/18));
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 4, "txy_tmp", (int*)&txy_tmp, GPUFI_DATATYPE_INTEGER);


    int ty = (txy_tmp/1);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 5, "ty", (int*)&ty, GPUFI_DATATYPE_INTEGER);


    int tx = (txy_tmp-__umul24(ty, 1));
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 6, "tx", (int*)&tx, GPUFI_DATATYPE_INTEGER);


    int bx = blockIdx.x;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 7, "bx", (int*)&bx, GPUFI_DATATYPE_INTEGER);


    int by = blockIdx.y;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 8, "by", (int*)&by, GPUFI_DATATYPE_INTEGER);


    /* Macroblock and sub-block coordinates */
    int mb_x = ((tx+__umul24(bx, 1))>>2);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 9, "mb_x", (int*)&mb_x, GPUFI_DATATYPE_INTEGER);


    int mb_y = ((ty+__umul24(by, 1))>>2);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 10, "mb_y", (int*)&mb_y, GPUFI_DATATYPE_INTEGER);


    int block_x = ((tx+__umul24(bx, 1))&3);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 11, "block_x", (int*)&block_x, GPUFI_DATATYPE_INTEGER);


    int block_y = ((ty+__umul24(by, 1))&3);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 12, "block_y", (int*)&block_y, GPUFI_DATATYPE_INTEGER);


    /*
    Block-copy data into shared memory.
       * Threads are grouped into sets of 16, leaving some threads idle.
    */
    if (((threadIdx.x>>4)<(1*1)))
    {
        int ty = ((threadIdx.x>>4)/1);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 13, "ty", (int*)&ty, GPUFI_DATATYPE_INTEGER);


        int tx = ((threadIdx.x>>4)-__umul24(ty, 1));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 14, "tx", (int*)&tx, GPUFI_DATATYPE_INTEGER);


        int tgroup = (threadIdx.x&15);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 15, "tgroup", (int*)&tgroup, GPUFI_DATATYPE_INTEGER);


        /* Width of the image in pixels */
        int img_width = (mb_width*16);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 16, "img_width", (int*)&img_width, GPUFI_DATATYPE_INTEGER);


        /* Pixel offset of the origin of the current 4x4 block */
        int frame_x = ((tx+__umul24(bx, 1))<<2);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 17, "frame_x", (int*)&frame_x, GPUFI_DATATYPE_INTEGER);


        int frame_y = ((ty+__umul24(by, 1))<<2);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 18, "frame_y", (int*)&frame_y, GPUFI_DATATYPE_INTEGER);


        /* Origin in the current frame for this 4x4 block */
        int cur_o = ((frame_y*img_width)+frame_x);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 19, "cur_o", (int*)&cur_o, GPUFI_DATATYPE_INTEGER);


        /* If this is an invalid 4x4 block, do nothing */
        if ((((frame_x>>4)<mb_width)&&((frame_y>>4)<mb_height)))
        {
            /* Copy one pixel into 'frame' */
            frame_loc[(((__umul24(ty, 1)+tx)<<4)+tgroup)]=frame[((cur_o+((tgroup>>2)*img_width))+(tgroup&3))];
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 20, "frame_loc[(((__umul24(ty, 1)+tx)<<4)+tgroup)]", (int*)&frame_loc[(((__umul24(ty, 1)+tx)<<4)+tgroup)], GPUFI_DATATYPE_UNKNOWN);


        }
    }
    __syncthreads();
    /* If this thread is assigned to an invalid 4x4 block, do nothing */
    if (((mb_x<mb_width)&&(mb_y<mb_height)))
    {
        /* Pixel offset of the origin of the current 4x4 block */
        int frame_x = (((mb_x<<2)+block_x)<<2);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 21, "frame_x", (int*)&frame_x, GPUFI_DATATYPE_INTEGER);


        int frame_y = (((mb_y<<2)+block_y)<<2);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 22, "frame_y", (int*)&frame_y, GPUFI_DATATYPE_INTEGER);


        /* Origin of the search area for this 4x4 block */
        int ref_x = (frame_x-16);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 23, "ref_x", (int*)&ref_x, GPUFI_DATATYPE_INTEGER);


        int ref_y = (frame_y-16);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 24, "ref_y", (int*)&ref_y, GPUFI_DATATYPE_INTEGER);


        /* Origin in the current frame for this 4x4 block */
        int cur_o = ((ty*1)+tx);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 25, "cur_o", (int*)&cur_o, GPUFI_DATATYPE_INTEGER);


        int search_pos;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 26, "search_pos", (int*)&search_pos, GPUFI_DATATYPE_INTEGER);


        int search_pos_base = ((threadIdx.x%((1089+(18-1))/18))*18);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 27, "search_pos_base", (int*)&search_pos_base, GPUFI_DATATYPE_INTEGER);


        int search_pos_end = (search_pos_base+18);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 28, "search_pos_end", (int*)&search_pos_end, GPUFI_DATATYPE_INTEGER);


        int sotmp = (search_pos_base/((2*16)+1));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 29, "sotmp", (int*)&sotmp, GPUFI_DATATYPE_INTEGER);


        int local_search_off_x = (search_pos_base-((sotmp<<5)+sotmp));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 30, "local_search_off_x", (int*)&local_search_off_x, GPUFI_DATATYPE_INTEGER);


        int search_off_y = (ref_y+sotmp);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 31, "search_off_y", (int*)&search_off_y, GPUFI_DATATYPE_INTEGER);


        /* Don't go past bounds */
        if ((search_pos_end>1089))
        {
            search_pos_end=1089;
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 32, "search_pos_end", (int*)&search_pos_end, GPUFI_DATATYPE_INTEGER);


        }
        /* For each search position, within the range allocated to this thread */
        /* Fault Injector Instrumentation Begin */
        GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_BEGIN);
        /* Fault Injector Instrumentation End */

        for (search_pos=search_pos_base; search_pos<search_pos_end; search_pos+=3)
        {
            /*
            It is also beneficial to fuse (jam) the enclosed loops if this loop
            	 * is unrolled.
            */
            /* Fault Injector Instrumentation Begin */
            GPUFI_KERNEL_ITERATION(gpufi_dev);
            /* Fault Injector Instrumentation End */

            unsigned short sad1 = 0, sad2 = 0, sad3 = 0;
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 35, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


            int search_off_x = (ref_x+local_search_off_x);
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 36, "search_off_x", (int*)&search_off_x, GPUFI_DATATYPE_INTEGER);


            /* 4x4 SAD computation */
            int y;
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 37, "y", (int*)&y, GPUFI_DATATYPE_INTEGER);


            /*
             
            	for(y=0; y<4; y++) {
            	  int t;
            	  t = tex2D(ref, search_off_x, search_off_y + y);
            	  sad1 += abs(t - FRAME_GET(cur_o, 0, y));
             
            	  t = tex2D(ref, search_off_x + 1, search_off_y + y);
            	  sad1 += abs(t - FRAME_GET(cur_o, 1, y));
            	  sad2 += abs(t - FRAME_GET(cur_o, 0, y));
             
            	  t = tex2D(ref, search_off_x + 2, search_off_y + y);
            	  sad1 += abs(t - FRAME_GET(cur_o, 2, y));
            	  sad2 += abs(t - FRAME_GET(cur_o, 1, y));
            	  sad3 += abs(t - FRAME_GET(cur_o, 0, y));
             
            	  t = tex2D(ref, search_off_x + 3, search_off_y + y);
            	  sad1 += abs(t - FRAME_GET(cur_o, 3, y));
            	  sad2 += abs(t - FRAME_GET(cur_o, 2, y));
            	  sad3 += abs(t - FRAME_GET(cur_o, 1, y));
             
            	  t = tex2D(ref, search_off_x + 4, search_off_y + y);
            	  sad2 += abs(t - FRAME_GET(cur_o, 3, y));
            	  sad3 += abs(t - FRAME_GET(cur_o, 2, y));
             
            	  t = tex2D(ref, search_off_x + 5, search_off_y + y);
            	  sad3 += abs(t - FRAME_GET(cur_o, 3, y));
            	}
            	
            */
            {
                /* y = 0 */
                int t;
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 38, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                y=0;
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 39, "y", (int*)&y, GPUFI_DATATYPE_INTEGER);


                t=tex2D(ref, search_off_x, (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 40, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+0)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 41, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+1), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 42, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+1)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 43, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+0)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 44, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+2), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 45, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+2)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 46, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+1)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 47, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+0)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 48, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+3), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 49, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+3)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 50, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+2)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 51, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+1)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 52, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+4), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 53, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+3)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 54, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+2)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 55, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+5), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 56, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+3)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 57, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                /* y = 1 */
                y=1;
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 58, "y", (int*)&y, GPUFI_DATATYPE_INTEGER);


                t=tex2D(ref, search_off_x, (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 59, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+0)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 60, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+1), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 61, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+1)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 62, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+0)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 63, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+2), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 64, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+2)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 65, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+1)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 66, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+0)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 67, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+3), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 68, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+3)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 69, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+2)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 70, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+1)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 71, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+4), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 72, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+3)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 73, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+2)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 74, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+5), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 75, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+3)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 76, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                /* y = 2 */
                y=2;
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 77, "y", (int*)&y, GPUFI_DATATYPE_INTEGER);


                t=tex2D(ref, search_off_x, (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 78, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+0)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 79, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+1), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 80, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+1)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 81, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+0)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 82, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+2), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 83, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+2)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 84, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+1)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 85, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+0)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 86, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+3), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 87, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+3)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 88, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+2)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 89, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+1)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 90, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+4), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 91, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+3)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 92, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+2)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 93, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+5), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 94, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+3)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 95, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                /* y = 3 */
                y=3;
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 96, "y", (int*)&y, GPUFI_DATATYPE_INTEGER);


                t=tex2D(ref, search_off_x, (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 97, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+0)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 98, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+1), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 99, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+1)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 100, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+0)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 101, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+2), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 102, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+2)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 103, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+1)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 104, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+0)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 105, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+3), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 106, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad1+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+3)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 107, "sad1", (int*)&sad1, GPUFI_DATATYPE_UNKNOWN);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+2)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 108, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+1)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 109, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+4), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 110, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad2+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+3)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 111, "sad2", (int*)&sad2, GPUFI_DATATYPE_UNKNOWN);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+2)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 112, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


                t=tex2D(ref, (search_off_x+5), (search_off_y+y));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 113, "t", (int*)&t, GPUFI_DATATYPE_INTEGER);


                sad3+=abs((t-frame_loc[(((cur_o<<4)+(y<<2))+3)]));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 114, "sad3", (int*)&sad3, GPUFI_DATATYPE_UNKNOWN);


            }
            /* Save this value into the local SAD array */
            sad_loc[(((__umul24(ty, 1)+tx)*1096)+search_pos)]=sad1;
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 115, "sad_loc[(((__umul24(ty, 1)+tx)*1096)+search_pos)]", (int*)&sad_loc[(((__umul24(ty, 1)+tx)*1096)+search_pos)], GPUFI_DATATYPE_UNKNOWN);


            sad_loc[(((__umul24(ty, 1)+tx)*1096)+(search_pos+1))]=sad2;
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 116, "sad_loc[(((__umul24(ty, 1)+tx)*1096)+(search_pos+1))]", (int*)&sad_loc[(((__umul24(ty, 1)+tx)*1096)+(search_pos+1))], GPUFI_DATATYPE_UNKNOWN);


            sad_loc[(((__umul24(ty, 1)+tx)*1096)+(search_pos+2))]=sad3;
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 117, "sad_loc[(((__umul24(ty, 1)+tx)*1096)+(search_pos+2))]", (int*)&sad_loc[(((__umul24(ty, 1)+tx)*1096)+(search_pos+2))], GPUFI_DATATYPE_UNKNOWN);


            local_search_off_x+=3;
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 118, "local_search_off_x", (int*)&local_search_off_x, GPUFI_DATATYPE_INTEGER);


            if ((local_search_off_x>=((2*16)+1)))
            {
                local_search_off_x-=((2*16)+1);
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 119, "local_search_off_x", (int*)&local_search_off_x, GPUFI_DATATYPE_INTEGER);


                search_off_y ++ ;
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 120, "search_off_y", (int*)&search_off_y, GPUFI_DATATYPE_INTEGER);


            }
        }
        /* Automatically inserted by Cetus_Cuda */
        GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_END);
        /* Fault Injector Instrumentation End *//* End of Cetus_Cuda code */

        GPUFI_KERNEL_VARIABLE(gpufi_dev, 34, "search_pos", (int*)&search_pos, GPUFI_DATATYPE_INTEGER);


    }
    __syncthreads();
    /*
    Block-copy data into global memory.
       * Threads are grouped into sets of 32, leaving some threads idle.
    */
    if (((threadIdx.x>>5)<(1*1)))
    {
        int tgroup = (threadIdx.x&31);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 121, "tgroup", (int*)&tgroup, GPUFI_DATATYPE_INTEGER);


        int ty = ((threadIdx.x>>5)/1);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 122, "ty", (int*)&ty, GPUFI_DATATYPE_INTEGER);


        int tx = ((threadIdx.x>>5)-__umul24(ty, 1));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 123, "tx", (int*)&tx, GPUFI_DATATYPE_INTEGER);


        int index;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 124, "index", (int*)&index, GPUFI_DATATYPE_INTEGER);


        /* Macroblock and sub-block coordinates */
        int mb_x = ((tx+__umul24(bx, 1))>>2);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 125, "mb_x", (int*)&mb_x, GPUFI_DATATYPE_INTEGER);


        int mb_y = ((ty+__umul24(by, 1))>>2);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 126, "mb_y", (int*)&mb_y, GPUFI_DATATYPE_INTEGER);


        int block_x = ((tx+__umul24(bx, 1))&3);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 127, "block_x", (int*)&block_x, GPUFI_DATATYPE_INTEGER);


        int block_y = ((ty+__umul24(by, 1))&3);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 128, "block_y", (int*)&block_y, GPUFI_DATATYPE_INTEGER);


        if (((mb_x<mb_width)&&(mb_y<mb_height)))
        {
            /*
            All SADs from this thread are stored in a contiguous chunk
            	 * of memory starting at this offset
            */
            blk_sad+=(((__umul24(__umul24(mb_width, mb_height), 25)+((__umul24(mb_y, mb_width)+mb_x)*16))+((4*block_y)+block_x))*1096);
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 129, "blk_sad", (int*)&blk_sad, GPUFI_DATATYPE_UNKNOWN_POINTER);


            /* Block copy, 32 threads at a time */
            /* Fault Injector Instrumentation Begin */
            GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_BEGIN);
            /* Fault Injector Instrumentation End */

            for (index=tgroup; index<(1096/4); index+=32)
            {
                /* Fault Injector Instrumentation Begin */
                GPUFI_KERNEL_ITERATION(gpufi_dev);
                /* Fault Injector Instrumentation End */

                ((struct vec8b * )blk_sad)[index]=sad_loc_8b[(((__umul24(ty, 1)+tx)*(1096/4))+index)];
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 132, "((struct vec8b * )blk_sad)[index]", (int*)&((struct vec8b * )blk_sad)[index], GPUFI_DATATYPE_UNKNOWN);


            }
            /* Automatically inserted by Cetus_Cuda */
            GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_END);
            /* Fault Injector Instrumentation End *//* End of Cetus_Cuda code */

            GPUFI_KERNEL_VARIABLE(gpufi_dev, 131, "index", (int*)&index, GPUFI_DATATYPE_INTEGER);


        }
    }
    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_END, 0, "mb_sad_calc");
}

texture<unsigned short, 2, hipReadModeElementType> &
get_ref(void )
{
    return ref;
}
