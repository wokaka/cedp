#include "hip/hip_runtime.h"
#ifndef __GPUFI_KERNEL__
#define __GPUFI_KERNEL__

#include <stdio.h>

#define MAX2(x,y) ((x>y)?x:y)
#define MAX3(x,y,z) ((x>y)?MAX2(x,z):MAX2(y,z))

struct _gpufi_data_ gpufi_host = {0, };
__device__ struct _gpufi_data_ *gpufi_dev;



#if GPUFI_FUNC_INIT
////////////////////////////////////////////////////
// CPU PART
////////////////////////////////////////////////////

int variable_count;
int kernel_count;

/*
 * @about   Fetch a fault injection command (including fault type and time)
 *          from a file, namely fi_cmd.txt
 */
int GPUFI_INIT(int kernel_cnt, int variable_cnt)
{
    FILE    *fp;
    char    cmd[32];
	
    kernel_count = kernel_cnt + 1;
    variable_count = variable_cnt;

    memset(&gpufi_host, 0, sizeof(struct _gpufi_data_));

    fp = fopen("fi_cmd.txt", "rt");

    if(!fp){
        printf("file open error\n");
        system("pwd");
        return -1;
    }

    fscanf(fp, "%s", cmd);
    printf("%s ", cmd);
		if(!strcmp(cmd, "fi")){
        gpufi_host.fault.mode = GPUFI_FI;
        gpufi_host.current.profile_index = -1;

        fscanf(fp, "%d %d %d %d 0x%x\n",
                &gpufi_host.fault.kernel,
                &gpufi_host.fault.instance,
								&gpufi_host.fault.varid,
                &gpufi_host.fault.call,
                &gpufi_host.fault.mask);

        gpufi_host.fault.mask_type = MASK_XOR;
				gpufi_host.fault.injected = 0;

        gpufi_host.current.blid = 0;
        gpufi_host.current.thid = 0;

       	printf("fi_cmd %d %d %d %d 0x%x\n", gpufi_host.fault.kernel, gpufi_host.fault.instance,
    		gpufi_host.fault.varid, gpufi_host.fault.call, gpufi_host.fault.mask);
    }

    fclose(fp);

    return 0;
}
#else 
int GPUFI_INIT(int x, int y)
{
    return 0;
}
#endif

#if GPUFI_FUNC_HALT

int GPUFI_HALT(char *fname)
{
    int i;

    printf("* injected: %d\n", gpufi_host.fault.injected);
    
#if DEBUG_INJECT_LOC
    if(gpufi_host.fault.injected){
			printf("%d %d %d %d %d\n", gpufi_host.injected.kernel,
				gpufi_host.injected.instance,
				gpufi_host.injected.loop,
				gpufi_host.injected.iteration,
				gpufi_host.injected.varid);
		}
#endif

	  printf("fi %d %d %d %d 0x%x\n", gpufi_host.fault.kernel, gpufi_host.fault.instance,
			gpufi_host.fault.varid, gpufi_host.fault.call, gpufi_host.fault.mask);
	
	  if(!gpufi_host.fault.injected)
	    for(i=0; i<MAX_KERNEL; i++){
	        if(gpufi_host.profile.kernel[i].state == 1)
	            printf("kernel\t%d\tinstance\t%d\tcall\t%d\n",
	                    gpufi_host.profile.kernel[i].id,
	                    gpufi_host.profile.kernel[i].instance,
	                    gpufi_host.profile.variable[i][gpufi_host.fault.varid].call_count);
	        else
	            break;
	        printf("\n");
	    }

    /*
    if(gpufi_host.debug.injected){
			printf("debug - %d %d %d %d\n", gpufi_host.debug.kernel,
				gpufi_host.debug.instance,
				gpufi_host.debug.loop,
				gpufi_host.debug.iteration);
		}
    */

    return 0;
}

#else

int GPUFI_HALT(char *x)
{
    return 0;
}


#endif

////////////////////////////////////////////////////
// DEVICE PART
////////////////////////////////////////////////////

#if GPUFI_FUNC_LIB

__device__ 
int GPUFI_EXIT(struct _gpufi_data_ *gpufi_dev, int errno)
{
	gpufi_dev->sdc = 1;
	//int *addr = (int *)0x0;
	//*addr = (int)0x80;
	//return 100 / errno;
	return 0;
}

#else

__device__
int GPUFI_STRCMP(char *src, char *dst)
{
	return 1;
}

__device__
void CudaStrcpy(char *dst, char *src, int max)
{
}

__device__ int GPUFI_EXIT(int errno)
{
	return 0;
}

__device__
void GPUFI_KERNEL_DEC(int *count)
{
}

__device__
void GPUFI_KERNEL_SET(int *count, int value)
{
}

#endif

#if GPUFI_FUNC_KERNEL

__device__
void GPUFI_KERNEL(struct _gpufi_data_ *gpufi_dev, int begin, int type, char *name)
{
    if(blockIdx.x != gpufi_dev->current.blid || threadIdx.x != gpufi_dev->current.thid)
        return;

        if(begin == GPUFI_KERNEL_BEGIN){
            /* 1: update current */
            if(gpufi_dev->profile.kernel_bitmap[type] == 0) /* kernel */
                gpufi_dev->profile.kernel_bitmap[type] = 1;
            gpufi_dev->profile.kernel_instance[type]++; /* instance */

            /* update current */
            gpufi_dev->current.kernel = type;
            gpufi_dev->current.instance = gpufi_dev->profile.kernel_instance[type] - 1;
            gpufi_dev->current.loop = -1;

            /* 2: profiling */
            gpufi_dev->current.profile_index++;

            gpufi_dev->profile.kernel[gpufi_dev->current.profile_index].state  = 1;
            gpufi_dev->profile.kernel[gpufi_dev->current.profile_index].id  = gpufi_dev->current.kernel;
            gpufi_dev->profile.kernel[gpufi_dev->current.profile_index].instance  = gpufi_dev->current.instance;
        }
}

#else

// DEVICE PART

__device__
void GPUFI_KERNEL(struct _gpufi_data_ *gpufi_dev, int begin, int type, char *name)
{
}

#endif


#if GPUFI_FUNC_LOOP

__device__
void GPUFI_KERNEL_LOOP(struct _gpufi_data_ *gpufi_dev, int begin)
{
    if(blockIdx.x != gpufi_dev->current.blid || threadIdx.x != gpufi_dev->current.thid)
        return;

/*
        gpufi_dev->current.loop++;
        gpufi_dev->current.iteration = -1;
        gpufi_dev->current.kernel = type;


//        if(gpufi_dev->fault.disable)
//            return;
            
        GPUFI_KERNEL_DEC(&gpufi_dev->fault.loop);
        GPUFI_KERNEL_SET(&gpufi_dev->fault.iteration, gpufi_dev->fault.iteration_count);
*/
}

__device__
void GPUFI_KERNEL_ITERATION(struct _gpufi_data_ *gpufi_dev)
{
    if(blockIdx.x != gpufi_dev->current.blid || threadIdx.x != gpufi_dev->current.thid)
        return;

/*
        gpufi_dev->current.iteration++;
        if(gpufi_dev->fault.disable)
            return;

        GPUFI_KERNEL_DEC(&gpufi_dev->fault.iteration);
*/
}

#else

__device__
void GPUFI_KERNEL_LOOP(struct _gpufi_data_ *gpufi_dev, int begin)
{
}

__device__
void GPUFI_KERNEL_ITERATION(struct _gpufi_data_ *gpufi_dev)
{
}

#endif

#if GPUFI_FUNC_VARIABLE

__device__
void GPUFI_KERNEL_VARIABLE(struct _gpufi_data_ *gpufi_dev, int varid, char *name, int *variable, int var_type)
{
    if(blockIdx.x != gpufi_dev->current.blid || threadIdx.x != gpufi_dev->current.thid)
        return;

    if(gpufi_dev->fault.disabled)
        return;

    if(varid == gpufi_dev->fault.varid && gpufi_dev->profile.kernel[gpufi_dev->current.profile_index].id == gpufi_dev->fault.kernel && gpufi_dev->profile.kernel[gpufi_dev->current.profile_index].instance == gpufi_dev->fault.instance && gpufi_dev->profile.variable[gpufi_dev->current.profile_index][varid].call_count == gpufi_dev->fault.call){
				    int temp;
            gpufi_dev->fault.disabled = 1;
            gpufi_dev->fault.injected++;
            if(var_type == GPUFI_DATATYPE_INTEGER){
                *variable ^= gpufi_dev->fault.mask;
            }
            else if(var_type == GPUFI_DATATYPE_FLOAT_POINTER ||
              var_type == GPUFI_DATATYPE_INTEGER_POINTER ||
              var_type == GPUFI_DATATYPE_UNKNOWN_POINTER){
					    temp = (int)*variable;
					    temp ^= gpufi_dev->fault.mask;
					    *variable = (int) temp;
            }
            else if(var_type == GPUFI_DATATYPE_FLOAT){
              *variable = __int_as_float(__float_as_int(*variable) ^ gpufi_dev->fault.mask);
            }
            else{
	            gpufi_dev->fault.injected--;
            }

            /* injected fault */
//            switch(gpufi_dev->fault.mask_type){
//            case MASK_XOR:
//                break;
//            }
    }
/*
#if DEBUG_INJECT_LOC
        gpufi_dev->injected.kernel = gpufi_dev->current.kernel;
        gpufi_dev->injected.instance = gpufi_dev->current.instance;
        gpufi_dev->injected.loop = gpufi_dev->current.loop;
        gpufi_dev->injected.iteration = gpufi_dev->current.iteration;
        gpufi_dev->injected.varid = varid;
#endif
*/
//    gpufi_dev->profile.variable[gpufi_dev->current.profile_index][varid].call_count++;
}

__device__
void GPUFI_KERNEL_VARIABLE_FORCE(struct _gpufi_data_ *gpufi_dev, int varid, char *name, int *variable)
{
#if 0
    if(blockIdx.x != gpufi_dev->current.blid || threadIdx.x != gpufi_dev->current.thid)
        return;

    if(gpufi_dev->fault.mode == GPUFI_PROFILE){
    }
    else{
        if(gpufi_dev->fault.kernel == gpufi_dev->current.kernel && 
            gpufi_dev->fault.instance == gpufi_dev->current.instance && 
            gpufi_dev->fault.loop == gpufi_dev->current.loop &&
	          gpufi_dev->fault.varid == varid){

            gpufi_dev->fault.injected++;

            /* injected fault */
            switch(gpufi_dev->fault.mask_type){
            case MASK_XOR:
                *variable ^= gpufi_dev->fault.mask;
                break;
            }
        }
    }
#endif
}

__device__
void GPUFI_KERNEL_VARIABLE_STREAM(struct _gpufi_data_ *gpufi_dev, int varid, char *name, int *variable, int length, int count)
{
#if 0
    int i;

    if(blockIdx.x != gpufi_dev->current.blid || threadIdx.x != gpufi_dev->current.thid)
        return;

    if(gpufi_dev->fault.mode == GPUFI_PROFILE){
//        gpufi_dev->profile.variable[gpufi_dev->current.kernel][gpufi_dev->current.instance][gpufi_dev->current.loop][varid]++;
    }
    else{
//        if(gpufi_dev->fault.disable)
//            return;
		for(i=0; i<count; i++){
//        if(!gpufi_dev->fault.injected){
            if(gpufi_dev->fault.kernel == gpufi_dev->current.kernel &&
               gpufi_dev->fault.instance == gpufi_dev->current.instance &&
               gpufi_dev->fault.loop == gpufi_dev->current.loop &&
               gpufi_dev->fault.iteration == gpufi_dev->current.iteration &&
	           gpufi_dev->fault.varid == varid
            ){
/*            if(0 == gpufi_dev->current.kernel &&
               1 == gpufi_dev->current.instance &&
               1 == gpufi_dev->current.loop &&
               1 == gpufi_dev->current.iteration &&
	           0 == varid){*/
                gpufi_dev->fault.injected++;

                /* injected fault */
                switch(gpufi_dev->fault.mask_type){
                case MASK_XOR:
                    variable[length*i/count] ^= gpufi_dev->fault.mask;
                    break;
                }
            }
//        }
	}
    }
#endif
}

#else

__device__
void GPUFI_KERNEL_VARIABLE(struct _gpufi_data_ *gpufi_dev, int varid, char *name, int *variable, int var_type)
{
}

__device__
void GPUFI_KERNEL_VARIABLE_FORCE(struct _gpufi_data_ *gpufi_dev, int varid, char *name, int *variable)
{
}

#endif

#endif