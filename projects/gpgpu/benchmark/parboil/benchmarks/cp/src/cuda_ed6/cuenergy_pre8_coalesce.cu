#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include "gpufi.h"
#include <stdio.h>

#include "cuenergy.h"

#if UNROLLX != 8
# error "UNROLLX must be 8"
#endif

#if BLOCKSIZEX != 16
# error "BLOCKSIZEX must be 16"
#endif

// Max constant buffer size is 64KB, minus whatever
// the CUDA runtime and compiler are using that we don't know about.
// At 16 bytes for atom, for this program 4070 atoms is about the max
// we can store in the constant buffer.
__constant__ float4 atominfo[MAXATOMS];

// This kernel calculates coulombic potential at each grid point and
// stores the results in the output array.

__global__ void cenergy(int numatoms, float gridspacing, float * energygrid) {

  int numatoms_dup = numatoms; //[ED]
  float gridspacing_dup = gridspacing; //[ED]
  float * energygrid_dup = energygrid; //[ED]
  
  unsigned int xindex  = __umul24(blockIdx.x, blockDim.x) * UNROLLX
                         + threadIdx.x;
  unsigned int xindex_dup  = __umul24(blockIdx.x, blockDim.x) * UNROLLX
                         + threadIdx.x;	//[ED]
                         
  unsigned int yindex  = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
  unsigned int yindex_dup  = __umul24(blockIdx.y, blockDim.y) + threadIdx.y; //[ED]

GPUFI_CHECK(yindex,yindex_dup);
GPUFI_CHECK(xindex,xindex_dup);
  unsigned int outaddr = (__umul24(gridDim.x, blockDim.x) * UNROLLX) * yindex
                         + xindex;
  unsigned int outaddr_dup = (__umul24(gridDim.x, blockDim.x) * UNROLLX) * yindex
                         + xindex;	//[ED]

GPUFI_CHECK(gridspacing,gridspacing_dup);
GPUFI_CHECK(yindex,yindex_dup);
  float coory = gridspacing * yindex;
  float coory_dup = gridspacing * yindex;	//[ED]

GPUFI_CHECK(gridspacing,gridspacing_dup);
GPUFI_CHECK(xindex,xindex_dup);
  float coorx = gridspacing * xindex;
  float coorx_dup = gridspacing * xindex;	//[ED]

  float energyvalx1=0.0f;
  float energyvalx1_dup=0.0f;
  float energyvalx2=0.0f;
  float energyvalx2_dup=0.0f;
  float energyvalx3=0.0f;
  float energyvalx3_dup=0.0f;
  float energyvalx4=0.0f;
  float energyvalx4_dup=0.0f;
  float energyvalx5=0.0f;
  float energyvalx5_dup=0.0f;
  float energyvalx6=0.0f;
  float energyvalx6_dup=0.0f;
  float energyvalx7=0.0f;
  float energyvalx7_dup=0.0f;
  float energyvalx8=0.0f;
  float energyvalx8_dup=0.0f;


GPUFI_CHECK(gridspacing,gridspacing_dup);
  float gridspacing_u = gridspacing * BLOCKSIZEX;
  float gridspacing_u_dup = gridspacing * BLOCKSIZEX;	//[ED]

  int atomid;
  int atomid_dup;


GPUFI_CHECK(numatoms,numatoms_dup);
  for (atomid=0, atomid_dup=0; atomid<numatoms && atomid == atomid_dup; atomid++, atomid_dup++) {
GPUFI_CHECK(coory,coory_dup);
GPUFI_CHECK(atomid,atomid_dup);
    float dy = coory - atominfo[atomid].y;
    float dy_dup = coory - atominfo[atomid].y;	//[ED]
GPUFI_CHECK(dy,dy_dup);
GPUFI_CHECK(atomid,atomid_dup);
    float dyz2 = (dy * dy) + atominfo[atomid].z;
    float dyz2_dup = (dy * dy) + atominfo[atomid].z;	//[ED]

GPUFI_CHECK(coorx,coorx_dup);
    float dx1 = coorx - atominfo[atomid].x;
    float dx1_dup = coorx - atominfo[atomid].x;
GPUFI_CHECK(dx1,dx1_dup);
GPUFI_CHECK(gridspacing_u,gridspacing_u_dup);
    float dx2 = dx1 + gridspacing_u;
    float dx2_dup = dx1 + gridspacing_u;
GPUFI_CHECK(dx2,dx2_dup);
GPUFI_CHECK(gridspacing_u,gridspacing_u_dup);
    float dx3 = dx2 + gridspacing_u;
    float dx3_dup = dx2 + gridspacing_u;
GPUFI_CHECK(dx3,dx3_dup);
GPUFI_CHECK(gridspacing_u,gridspacing_u_dup);
    float dx4 = dx3 + gridspacing_u;
    float dx4_dup = dx3 + gridspacing_u;
GPUFI_CHECK(dx4,dx4_dup);
GPUFI_CHECK(gridspacing_u,gridspacing_u_dup);
    float dx5 = dx4 + gridspacing_u;
    float dx5_dup = dx4 + gridspacing_u;
GPUFI_CHECK(dx5,dx5_dup);
GPUFI_CHECK(gridspacing_u,gridspacing_u_dup);
    float dx6 = dx5 + gridspacing_u;
    float dx6_dup = dx5 + gridspacing_u;
GPUFI_CHECK(dx6,dx6_dup);
GPUFI_CHECK(gridspacing_u,gridspacing_u_dup);
    float dx7 = dx6 + gridspacing_u;
    float dx7_dup = dx6 + gridspacing_u;
GPUFI_CHECK(dx7,dx7_dup);
GPUFI_CHECK(gridspacing_u,gridspacing_u_dup);
    float dx8 = dx7 + gridspacing_u;
    float dx8_dup = dx7 + gridspacing_u;


GPUFI_CHECK(dx1,dx1_dup);
GPUFI_CHECK(dyz2,dyz2_dup);
GPUFI_CHECK(atomid,atomid_dup);
    energyvalx1 += atominfo[atomid].w * (1.0f / sqrtf(dx1*dx1 + dyz2));
    energyvalx1_dup += atominfo[atomid].w * (1.0f / sqrtf(dx1*dx1 + dyz2));
GPUFI_CHECK(dx2,dx2_dup);
GPUFI_CHECK(dyz2,dyz2_dup);
GPUFI_CHECK(atomid,atomid_dup);
    energyvalx2 += atominfo[atomid].w * (1.0f / sqrtf(dx2*dx2 + dyz2));
    energyvalx2_dup += atominfo[atomid].w * (1.0f / sqrtf(dx2*dx2 + dyz2));
GPUFI_CHECK(dx3,dx3_dup);
GPUFI_CHECK(dyz2,dyz2_dup);
GPUFI_CHECK(atomid,atomid_dup);
    energyvalx3 += atominfo[atomid].w * (1.0f / sqrtf(dx3*dx3 + dyz2));
    energyvalx3_dup += atominfo[atomid].w * (1.0f / sqrtf(dx3*dx3 + dyz2));
GPUFI_CHECK(dx4,dx4_dup);
GPUFI_CHECK(dyz2,dyz2_dup);
GPUFI_CHECK(atomid,atomid_dup);
    energyvalx4 += atominfo[atomid].w * (1.0f / sqrtf(dx4*dx4 + dyz2));
    energyvalx4_dup += atominfo[atomid].w * (1.0f / sqrtf(dx4*dx4 + dyz2));
GPUFI_CHECK(dx5,dx5_dup);
GPUFI_CHECK(dyz2,dyz2_dup);
GPUFI_CHECK(atomid,atomid_dup);
    energyvalx5 += atominfo[atomid].w * (1.0f / sqrtf(dx5*dx5 + dyz2));
    energyvalx5_dup += atominfo[atomid].w * (1.0f / sqrtf(dx5*dx5 + dyz2));
GPUFI_CHECK(dx6,dx6_dup);
GPUFI_CHECK(dyz2,dyz2_dup);
GPUFI_CHECK(atomid,atomid_dup);
    energyvalx6 += atominfo[atomid].w * (1.0f / sqrtf(dx6*dx6 + dyz2));
    energyvalx6_dup += atominfo[atomid].w * (1.0f / sqrtf(dx6*dx6 + dyz2));
GPUFI_CHECK(dx7,dx7_dup);
GPUFI_CHECK(dyz2,dyz2_dup);
GPUFI_CHECK(atomid,atomid_dup);
    energyvalx7 += atominfo[atomid].w * (1.0f / sqrtf(dx7*dx7 + dyz2));
    energyvalx7_dup += atominfo[atomid].w * (1.0f / sqrtf(dx7*dx7 + dyz2));
GPUFI_CHECK(dx8,dx8_dup);
GPUFI_CHECK(dyz2,dyz2_dup);
GPUFI_CHECK(atomid,atomid_dup);
    energyvalx8 += atominfo[atomid].w * (1.0f / sqrtf(dx8*dx8 + dyz2));
    energyvalx8_dup += atominfo[atomid].w * (1.0f / sqrtf(dx8*dx8 + dyz2));
  }	

GPUFI_CHECK(energygrid,energygrid_dup);
GPUFI_CHECK(outaddr,outaddr_dup);
GPUFI_CHECK(energyvalx1,energyvalx1_dup);
float temp = energygrid[outaddr];
  energygrid[outaddr]   += energyvalx1;
  temp += energyvalx1;
GPUFI_CHECK(energygrid[outaddr],temp);

GPUFI_CHECK(energygrid,energygrid_dup);
GPUFI_CHECK(outaddr,outaddr_dup);
GPUFI_CHECK(energyvalx2,energyvalx2_dup);
temp = energygrid[outaddr+1*BLOCKSIZEX];
  energygrid[outaddr+1*BLOCKSIZEX] += energyvalx2;
  temp += energyvalx2;
GPUFI_CHECK(energygrid[outaddr+1*BLOCKSIZEX],temp);

GPUFI_CHECK(energygrid,energygrid_dup);
GPUFI_CHECK(outaddr,outaddr_dup);
GPUFI_CHECK(energyvalx3,energyvalx3_dup);
temp = energygrid[outaddr+2*BLOCKSIZEX];
  energygrid[outaddr+2*BLOCKSIZEX] += energyvalx3;
  temp += energyvalx3;
GPUFI_CHECK(energygrid[outaddr+2*BLOCKSIZEX],temp);

GPUFI_CHECK(energygrid,energygrid_dup);
GPUFI_CHECK(outaddr,outaddr_dup);
GPUFI_CHECK(energyvalx4,energyvalx4_dup);
temp = energygrid[outaddr+3*BLOCKSIZEX];
  energygrid[outaddr+3*BLOCKSIZEX] += energyvalx4;
  temp += energyvalx4;
GPUFI_CHECK(energygrid[outaddr+3*BLOCKSIZEX],temp);

GPUFI_CHECK(energygrid,energygrid_dup);
GPUFI_CHECK(outaddr,outaddr_dup);
GPUFI_CHECK(energyvalx5,energyvalx5_dup);
temp = energygrid[outaddr+4*BLOCKSIZEX];
  energygrid[outaddr+4*BLOCKSIZEX] += energyvalx5;
  temp += energyvalx5;
GPUFI_CHECK(energygrid[outaddr+4*BLOCKSIZEX],temp);

GPUFI_CHECK(energygrid,energygrid_dup);
GPUFI_CHECK(outaddr,outaddr_dup);
GPUFI_CHECK(energyvalx6,energyvalx6_dup);
temp = energygrid[outaddr+5*BLOCKSIZEX];
  energygrid[outaddr+5*BLOCKSIZEX] += energyvalx6;
  temp += energyvalx6;
GPUFI_CHECK(energygrid[outaddr+5*BLOCKSIZEX],temp);

GPUFI_CHECK(energygrid,energygrid_dup);
GPUFI_CHECK(outaddr,outaddr_dup);
GPUFI_CHECK(energyvalx7,energyvalx7_dup);
temp = energygrid[outaddr+6*BLOCKSIZEX];
  energygrid[outaddr+6*BLOCKSIZEX] += energyvalx7;
  temp += energyvalx7;
GPUFI_CHECK(energygrid[outaddr+6*BLOCKSIZEX],temp);

GPUFI_CHECK(energygrid,energygrid_dup);
GPUFI_CHECK(outaddr,outaddr_dup);
GPUFI_CHECK(energyvalx8,energyvalx8_dup);
temp = energygrid[outaddr+7*BLOCKSIZEX];
  energygrid[outaddr+7*BLOCKSIZEX] += energyvalx8;
  temp += energyvalx8;
GPUFI_CHECK(energygrid[outaddr+7*BLOCKSIZEX],temp);
}

// This function copies atoms from the CPU to the GPU and
// precalculates (z^2) for each atom.

int copyatomstoconstbuf(float *atoms, int count, float zplane) {
  if (count > MAXATOMS) {
    printf("Atom count exceeds constant buffer storage capacity\n");
    return -1;
  }

  float atompre[4*MAXATOMS];
  int i;
  for (i=0; i<count*4; i+=4) {
    atompre[i    ] = atoms[i    ];
    atompre[i + 1] = atoms[i + 1];
    float dz = zplane - atoms[i + 2];
    atompre[i + 2]  = dz*dz;
    atompre[i + 3] = atoms[i + 3];
  }

  hipMemcpyToSymbol(HIP_SYMBOL(atominfo), atompre, count * 4 * sizeof(float), 0);
  CUERR // check and clear any existing errors

  return 0;
}

