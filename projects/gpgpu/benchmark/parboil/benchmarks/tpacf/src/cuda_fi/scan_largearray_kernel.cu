#include "hip/hip_runtime.h"
/* Instrumented by FaultInjector */
#include "gpufi.h"

/*
**************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 **************************************************************************
*/
#include "gpufi.h"

#include <assert.h>

#include "model.h"

#include <math.h>

#include "gpufi.h"

float *  * g_scanBlockSums;
unsigned int g_numEltsAllocated = 0;
unsigned int g_numLevelsAllocated = 0;
__constant__ float dev_binb[(20+1)];
unsigned int NUM_SETS;
unsigned int NUM_ELEMENTS;
/* create the bin boundaries
 */
void initBinB(struct pb_TimerSet * timers)
{
    float * binb = ((float * )malloc(((20+1)*sizeof (float))));
    int k;
    for (k=0; k<(20+1); k ++ )
    {
        binb[k]=cos((((pow(10.0, (log10(1.0)+((k*1.0)/5)))/60.0)*3.141592653589793)/180.0));
    }
    pb_SwitchToTimer(timers, pb_TimerID_COPY);
    hipMemcpyToSymbol(HIP_SYMBOL(dev_binb), binb, ((20+1)*sizeof (float)));
    pb_SwitchToTimer(timers, pb_TimerID_COMPUTE);
    free(binb);
}

__global__ void gen_hists(hist_t * histograms, float * all_x_data, float * all_y_data, float * all_z_data, int NUM_SETS, int NUM_ELEMENTS,  struct _gpufi_data_ *gpufi_dev)
{
    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_BEGIN, 0, "gen_hists");
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 0, "histograms", (int*)&histograms, GPUFI_DATATYPE_UNKNOWN_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 1, "all_x_data", (int*)&all_x_data, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 2, "all_y_data", (int*)&all_y_data, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 3, "all_z_data", (int*)&all_z_data, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 4, "NUM_SETS", (int*)&NUM_SETS, GPUFI_DATATYPE_INTEGER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 5, "NUM_ELEMENTS", (int*)&NUM_ELEMENTS, GPUFI_DATATYPE_INTEGER);


    unsigned int bx = blockIdx.x;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 6, "bx", (int*)&bx, GPUFI_DATATYPE_INTEGER);


    unsigned int tid = threadIdx.x;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 7, "tid", (int*)&tid, GPUFI_DATATYPE_INTEGER);


    bool do_self = (bx<(NUM_SETS+1));
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 8, "do_self", (int*)&do_self, GPUFI_DATATYPE_UNKNOWN);


    float * data_x;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 9, "data_x", (int*)&data_x, GPUFI_DATATYPE_FLOAT_POINTER);


    float * data_y;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 10, "data_y", (int*)&data_y, GPUFI_DATATYPE_FLOAT_POINTER);


    float * data_z;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 11, "data_z", (int*)&data_z, GPUFI_DATATYPE_FLOAT_POINTER);


    float * random_x;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 12, "random_x", (int*)&random_x, GPUFI_DATATYPE_FLOAT_POINTER);


    float * random_y;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 13, "random_y", (int*)&random_y, GPUFI_DATATYPE_FLOAT_POINTER);


    float * random_z;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 14, "random_z", (int*)&random_z, GPUFI_DATATYPE_FLOAT_POINTER);


    __shared__ float data_x_s[256];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 15, "data_x_s", (int*)&data_x_s, GPUFI_DATATYPE_FLOAT);


    __shared__ float data_y_s[256];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 16, "data_y_s", (int*)&data_y_s, GPUFI_DATATYPE_FLOAT);


    __shared__ float data_z_s[256];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 17, "data_z_s", (int*)&data_z_s, GPUFI_DATATYPE_FLOAT);


    volatile __shared__ unsigned int warp_hists[20][((256/32)*16)];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 18, "warp_hists", (int*)&warp_hists, GPUFI_DATATYPE_INTEGER);


    /* 640B <1k
     */
    unsigned int w;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 19, "w", (int*)&w, GPUFI_DATATYPE_INTEGER);


    unsigned int i;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 20, "i", (int*)&i, GPUFI_DATATYPE_INTEGER);


    unsigned int warp_index;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 21, "warp_index", (int*)&warp_index, GPUFI_DATATYPE_INTEGER);


    unsigned int bin_index;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 22, "bin_index", (int*)&bin_index, GPUFI_DATATYPE_INTEGER);


    unsigned int offset;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 23, "offset", (int*)&offset, GPUFI_DATATYPE_INTEGER);


    unsigned int bin_base;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 24, "bin_base", (int*)&bin_base, GPUFI_DATATYPE_INTEGER);


    hist_t * hist_base;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 25, "hist_base", (int*)&hist_base, GPUFI_DATATYPE_UNKNOWN_POINTER);


    /* Fault Injector Instrumentation Begin */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_BEGIN);
    /* Fault Injector Instrumentation End */

    for (w=0; w<(20*((256/32)*16)); w+=256)
    {
        /* Fault Injector Instrumentation Begin */
        GPUFI_KERNEL_ITERATION(gpufi_dev);
        /* Fault Injector Instrumentation End */

        if (((w+tid)<(20*((256/32)*16))))
        {
            warp_hists[((w+tid)/((256/32)*16))][((w+tid)%((256/32)*16))]=0;
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 28, "warp_hists[((w+tid)/((256/32)*16))][((w+tid)%((256/32)*16))]", (int*)&warp_hists[((w+tid)/((256/32)*16))][((w+tid)%((256/32)*16))], GPUFI_DATATYPE_INTEGER);


        }
    }
    /* Automatically inserted by Cetus_Cuda */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_END);
    /* Fault Injector Instrumentation End *//* End of Cetus_Cuda code */

    GPUFI_KERNEL_VARIABLE(gpufi_dev, 27, "w", (int*)&w, GPUFI_DATATYPE_INTEGER);


    /* Get stuff into shared memory to kick off the loop.
     */
    if (( ! do_self))
    {
        data_x=all_x_data;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 29, "data_x", (int*)&data_x, GPUFI_DATATYPE_FLOAT_POINTER);


        data_y=all_y_data;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 30, "data_y", (int*)&data_y, GPUFI_DATATYPE_FLOAT_POINTER);


        data_z=all_z_data;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 31, "data_z", (int*)&data_z, GPUFI_DATATYPE_FLOAT_POINTER);


        random_x=(all_x_data+(NUM_ELEMENTS*(bx-NUM_SETS)));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 32, "random_x", (int*)&random_x, GPUFI_DATATYPE_FLOAT_POINTER);


        random_y=(all_y_data+(NUM_ELEMENTS*(bx-NUM_SETS)));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 33, "random_y", (int*)&random_y, GPUFI_DATATYPE_FLOAT_POINTER);


        random_z=(all_z_data+(NUM_ELEMENTS*(bx-NUM_SETS)));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 34, "random_z", (int*)&random_z, GPUFI_DATATYPE_FLOAT_POINTER);


    }
    else
    {
        random_x=(all_x_data+(NUM_ELEMENTS*bx));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 35, "random_x", (int*)&random_x, GPUFI_DATATYPE_FLOAT_POINTER);


        random_y=(all_y_data+(NUM_ELEMENTS*bx));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 36, "random_y", (int*)&random_y, GPUFI_DATATYPE_FLOAT_POINTER);


        random_z=(all_z_data+(NUM_ELEMENTS*bx));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 37, "random_z", (int*)&random_z, GPUFI_DATATYPE_FLOAT_POINTER);


        data_x=random_x;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 38, "data_x", (int*)&data_x, GPUFI_DATATYPE_FLOAT_POINTER);


        data_y=random_y;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 39, "data_y", (int*)&data_y, GPUFI_DATATYPE_FLOAT_POINTER);


        data_z=random_z;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 40, "data_z", (int*)&data_z, GPUFI_DATATYPE_FLOAT_POINTER);


    }
    /* Iterate over all data points
     */
    /* Fault Injector Instrumentation Begin */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_BEGIN);
    /* Fault Injector Instrumentation End */

    for (i=0; i<NUM_ELEMENTS; i+=256)
    {
        /* Fault Injector Instrumentation Begin */
        GPUFI_KERNEL_ITERATION(gpufi_dev);
        /* Fault Injector Instrumentation End */

        /* load current set of data into shared memory
         */
        /* (total of BLOCK_SIZE points loaded)
         */
        unsigned int j;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 43, "j", (int*)&j, GPUFI_DATATYPE_INTEGER);


        if (((tid+i)<NUM_ELEMENTS))
        {
            /* reading outside of bounds is a-okay
             */
            data_x_s[tid]=data_x[(tid+i)];
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 44, "data_x_s[tid]", (int*)&data_x_s[tid], GPUFI_DATATYPE_FLOAT);


            data_y_s[tid]=data_y[(tid+i)];
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 45, "data_y_s[tid]", (int*)&data_y_s[tid], GPUFI_DATATYPE_FLOAT);


            data_z_s[tid]=data_z[(tid+i)];
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 46, "data_z_s[tid]", (int*)&data_z_s[tid], GPUFI_DATATYPE_FLOAT);


        }
        __syncthreads();
        /* Iterate over all random points
         */
        for (j=(do_self ? (i+1) : 0); j<NUM_ELEMENTS; j+=256)
        {
            /* load current random point values
             */
            float random_x_s;
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 49, "random_x_s", (int*)&random_x_s, GPUFI_DATATYPE_FLOAT);


            float random_y_s;
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 50, "random_y_s", (int*)&random_y_s, GPUFI_DATATYPE_FLOAT);


            float random_z_s;
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 51, "random_z_s", (int*)&random_z_s, GPUFI_DATATYPE_FLOAT);


            unsigned int k;
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 52, "k", (int*)&k, GPUFI_DATATYPE_INTEGER);


            if (((tid+j)<NUM_ELEMENTS))
            {
                random_x_s=random_x[(tid+j)];
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 53, "random_x_s", (int*)&random_x_s, GPUFI_DATATYPE_FLOAT);


                random_y_s=random_y[(tid+j)];
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 54, "random_y_s", (int*)&random_y_s, GPUFI_DATATYPE_FLOAT);


                random_z_s=random_z[(tid+j)];
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 55, "random_z_s", (int*)&random_z_s, GPUFI_DATATYPE_FLOAT);


            }
            /* Iterate for all elements of current set of data points
             */
            /* (BLOCK_SIZE iterations per thread)
             */
            /* Each thread calcs against 1 random point within cur set of random
             */
            /* (so BLOCK_SIZE threads covers all random points within cur set)
             */
            for (k=0; (k<256)&&((k+i)<NUM_ELEMENTS); k+=1)
            {
                /* do actual calculations on the values:
                 */
                float distance = (((data_x_s[k]*random_x_s)+(data_y_s[k]*random_y_s))+(data_z_s[k]*random_z_s));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 58, "distance", (int*)&distance, GPUFI_DATATYPE_FLOAT);


                /* run binary search
                 */
                unsigned int bin_index;
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 59, "bin_index", (int*)&bin_index, GPUFI_DATATYPE_INTEGER);


                unsigned int min = 0;
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 60, "min", (int*)&min, GPUFI_DATATYPE_INTEGER);


                unsigned int max = 20;
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 61, "max", (int*)&max, GPUFI_DATATYPE_INTEGER);


                unsigned int k2;
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 62, "k2", (int*)&k2, GPUFI_DATATYPE_INTEGER);


                unsigned int warpnum;
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 63, "warpnum", (int*)&warpnum, GPUFI_DATATYPE_INTEGER);


                while ((max>(min+1)))
                {
                    k2=((min+max)/2);
                    GPUFI_KERNEL_VARIABLE(gpufi_dev, 64, "k2", (int*)&k2, GPUFI_DATATYPE_INTEGER);


                    if ((distance>=dev_binb[k2]))
                    {
                        max=k2;
                        GPUFI_KERNEL_VARIABLE(gpufi_dev, 65, "max", (int*)&max, GPUFI_DATATYPE_INTEGER);


                    }
                    else
                    {
                        min=k2;
                        GPUFI_KERNEL_VARIABLE(gpufi_dev, 66, "min", (int*)&min, GPUFI_DATATYPE_INTEGER);


                    }
                }
                warpnum=(tid/(32/16));
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 67, "warpnum", (int*)&warpnum, GPUFI_DATATYPE_INTEGER);


                if (((((distance<dev_binb[min])&&(distance>=dev_binb[max]))&&(( ! do_self)||((tid+j)>(i+k))))&&((tid+j)<NUM_ELEMENTS)))
                {
                    unsigned long myVal;
                    GPUFI_KERNEL_VARIABLE(gpufi_dev, 68, "myVal", (int*)&myVal, GPUFI_DATATYPE_UNKNOWN);


                    bin_index=(max-1);
                    GPUFI_KERNEL_VARIABLE(gpufi_dev, 69, "bin_index", (int*)&bin_index, GPUFI_DATATYPE_INTEGER);


                    /* Write into the mini histogram
                     */
                    /* do this until update is not overwritten:
                     */
                    do
                    {
                        /* read the current bin val
                         */
                        myVal=(warp_hists[bin_index][warpnum]&134217727);
                        GPUFI_KERNEL_VARIABLE(gpufi_dev, 70, "myVal", (int*)&myVal, GPUFI_DATATYPE_UNKNOWN);


                        /* increment and put thread tag in top bits
                         */
                        myVal=(((tid&31)<<27)|(myVal+1));
                        GPUFI_KERNEL_VARIABLE(gpufi_dev, 71, "myVal", (int*)&myVal, GPUFI_DATATYPE_UNKNOWN);


                        /* attempt to write the bin
                         */
                        warp_hists[bin_index][warpnum]=myVal;
                        GPUFI_KERNEL_VARIABLE(gpufi_dev, 72, "warp_hists[bin_index][warpnum]", (int*)&warp_hists[bin_index][warpnum], GPUFI_DATATYPE_INTEGER);


                        /* while updates overwritten, keep retrying
                         */
                    }
                    while((warp_hists[bin_index][warpnum]!=myVal));

                }
                __syncthreads();
            }
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 57, "k", (int*)&k, GPUFI_DATATYPE_INTEGER);


        }
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 48, "j", (int*)&j, GPUFI_DATATYPE_INTEGER);


    }
    /* Automatically inserted by Cetus_Cuda */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_END);
    /* Fault Injector Instrumentation End *//* End of Cetus_Cuda code */

    GPUFI_KERNEL_VARIABLE(gpufi_dev, 42, "i", (int*)&i, GPUFI_DATATYPE_INTEGER);


    /* coalesce the histograms in a block
     */
    warp_index=(tid&((((256/32)*16)>>1)-1));
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 73, "warp_index", (int*)&warp_index, GPUFI_DATATYPE_INTEGER);


    bin_index=(tid/(((256/32)*16)>>1));
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 74, "bin_index", (int*)&bin_index, GPUFI_DATATYPE_INTEGER);


    /* Fault Injector Instrumentation Begin */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_BEGIN);
    /* Fault Injector Instrumentation End */

    for (offset=(((256/32)*16)>>1); offset>0; offset>>=1)
    {
        /* Fault Injector Instrumentation Begin */
        GPUFI_KERNEL_ITERATION(gpufi_dev);
        /* Fault Injector Instrumentation End */

        for (bin_base=0; bin_base<20; bin_base+=(256/(((256/32)*16)>>1)))
        {
            __syncthreads();
            if (((warp_index<offset)&&((bin_base+bin_index)<20)))
            {
                unsigned long sum = (warp_hists[(bin_base+bin_index)][warp_index]+warp_hists[(bin_base+bin_index)][(warp_index+offset)]);
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 79, "sum", (int*)&sum, GPUFI_DATATYPE_UNKNOWN);


                warp_hists[(bin_base+bin_index)][warp_index]=sum;
                GPUFI_KERNEL_VARIABLE(gpufi_dev, 80, "warp_hists[(bin_base+bin_index)][warp_index]", (int*)&warp_hists[(bin_base+bin_index)][warp_index], GPUFI_DATATYPE_INTEGER);


            }
        }
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 78, "bin_base", (int*)&bin_base, GPUFI_DATATYPE_INTEGER);


    }
    /* Automatically inserted by Cetus_Cuda */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_END);
    /* Fault Injector Instrumentation End *//* End of Cetus_Cuda code */

    GPUFI_KERNEL_VARIABLE(gpufi_dev, 76, "offset", (int*)&offset, GPUFI_DATATYPE_INTEGER);


    __syncthreads();
    /* Put the results back in the real histogram
     */
    /* warp_hists[x][0] holds sum of all locations of bin x
     */
    hist_base=(histograms+(20*bx));
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 81, "hist_base", (int*)&hist_base, GPUFI_DATATYPE_UNKNOWN_POINTER);


    if ((tid<20))
    {
        hist_base[tid]=(warp_hists[tid][0]&134217727);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 82, "hist_base[tid]", (int*)&hist_base[tid], GPUFI_DATATYPE_UNKNOWN);


    }
    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_END, 0, "gen_hists");
}

void TPACF(hist_t * histograms, float * d_x_data, float * d_y_data, float * d_z_data)
{
    dim3 dimBlock;
    dim3 dimGrid;
    dimBlock.x=256;
    dimGrid.x=((NUM_SETS*2)+1);
    /* Fault Injection Instrumentation Begin */
    hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
    CUDA_ERRCK
    hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
    CUDA_ERRCK
    /* Fault Injection Instrumentation End */

    gen_hists<<<dimGrid, dimBlock>>> (histograms, d_x_data, d_y_data, d_z_data, NUM_SETS, NUM_ELEMENTS, gpufi_dev);
    /* Fault Injection Instrumentation Begin */
    hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
    CUDA_ERRCK
    hipFree(gpufi_dev);
    CUDA_ERRCK
    /* Fault Injection Instrumentation End */

}
