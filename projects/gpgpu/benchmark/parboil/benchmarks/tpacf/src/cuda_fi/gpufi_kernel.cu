#include "hip/hip_runtime.h"
#ifndef __GPUFI_KERNEL__
#define __GPUFI_KERNEL__

#include <stdio.h>

#define MAX2(x,y) ((x>y)?x:y)
#define MAX3(x,y,z) ((x>y)?MAX2(x,z):MAX2(y,z))

struct _gpufi_data_ gpufi_host = {0, };
__device__ struct _gpufi_data_ *gpufi_dev;

////////////////////////////////////////////////////
// CPU PART
////////////////////////////////////////////////////

/*
 * @about   Fetch a fault injection command (including fault type and time)
 *          from a file, namely fi_cmd.txt
 */
int GPUFI_INIT(int kernel_cnt, int variable_cnt)
{
    FILE    *fp;
    char    cmd[32];
	
    memset(&gpufi_host, 0, sizeof(struct _gpufi_data_));

    fp = fopen("fi_cmd.txt", "rt");
    if(!fp){
        printf("file open error\n");
        system("pwd");
        return -1;
    }

    fscanf(fp, "%s", cmd);
    printf("%s ", cmd);
		if(!strcmp(cmd, "fi")){
        fscanf(fp, "%d %d %d %d 0x%x %d %d\n",
                &gpufi_host.fault.kernel,
                &gpufi_host.fault.instance,
								&gpufi_host.fault.varid,
                &gpufi_host.fault.call,
                &gpufi_host.fault.mask, 
		      		  &gpufi_host.fault.blid,
    			    	&gpufi_host.fault.thid);

				gpufi_host.fault.injected = 0;
 				gpufi_host.fault.disabled = 1;

       	printf("fi_cmd %d %d %d %d 0x%x\n", gpufi_host.fault.kernel, gpufi_host.fault.instance,
    		gpufi_host.fault.varid, gpufi_host.fault.call, gpufi_host.fault.mask);
    }

    fclose(fp);

    return 0;
}

int GPUFI_HALT(char *fname)
{
    printf("* injected: %d\n", gpufi_host.fault.injected);
    fprintf(stderr, "* injected: %d\n", gpufi_host.fault.injected);
    
	  printf("fi %d %d %d %d 0x%x\n", gpufi_host.fault.kernel, gpufi_host.fault.instance,
			gpufi_host.fault.varid, gpufi_host.fault.call, gpufi_host.fault.mask);
	
    return 0;
}

////////////////////////////////////////////////////
// DEVICE PART
////////////////////////////////////////////////////

__device__ 
int GPUFI_EXIT(struct _gpufi_data_ *gpufi_dev, int errno)
{
	gpufi_dev->sdc = 1;
	//int *addr = (int *)0x0;
	//*addr = (int)0x80;
	//return 100 / errno;
	return 0;
}

__device__
void GPUFI_KERNEL(struct _gpufi_data_ *gpufi_dev, int begin, int type, char *name)
{
    if(blockIdx.x != gpufi_dev->fault.blid || threadIdx.x != gpufi_dev->fault.thid)
        return;

		gpufi_dev->fault.disabled = 1;
    if(begin == GPUFI_KERNEL_BEGIN){
    		if(gpufi_dev->fault.kernel == type){
            if(gpufi_dev->fault.instance == gpufi_dev->current.instance++){
    					gpufi_dev->fault.disabled = 0;
    				}
    		}
    }
}

__device__
void GPUFI_KERNEL_LOOP(struct _gpufi_data_ *gpufi_dev, int begin)
{
}

__device__
void GPUFI_KERNEL_ITERATION(struct _gpufi_data_ *gpufi_dev)
{
}

__device__
void GPUFI_KERNEL_VARIABLE(struct _gpufi_data_ *gpufi_dev, int varid, char *name, int *variable, int var_type)
{
    if(blockIdx.x != gpufi_dev->fault.blid || threadIdx.x != gpufi_dev->fault.thid)
        return;

    if(gpufi_dev->fault.disabled)
        return;

    if(varid == gpufi_dev->fault.varid){
    	if(gpufi_dev->fault.call == gpufi_dev->current.count++){
            gpufi_dev->fault.disabled = 1;
            gpufi_dev->fault.injected++;

            if(var_type == GPUFI_DATATYPE_INTEGER){
                *variable ^= gpufi_dev->fault.mask;
            }
            else if(var_type == GPUFI_DATATYPE_FLOAT_POINTER ||
              var_type == GPUFI_DATATYPE_INTEGER_POINTER ||
              var_type == GPUFI_DATATYPE_UNKNOWN_POINTER){
					    int temp;
					    temp = (int)*variable;
					    temp ^= gpufi_dev->fault.mask;
					    *variable = (int) temp;
            }
            else if(var_type == GPUFI_DATATYPE_FLOAT){
              *variable = __int_as_float(__float_as_int(*variable) ^ gpufi_dev->fault.mask);
            }
            else{
					    int temp;
					    temp = (int)*variable;
					    temp ^= gpufi_dev->fault.mask;
					    *variable = (int) temp;
            }
		  }
    }
}

#endif