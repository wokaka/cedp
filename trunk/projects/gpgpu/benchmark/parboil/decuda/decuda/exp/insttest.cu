#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <algorithm>

__global__ void my_kernel(uint32_t *data, uint32_t p0, uint32_t p1, uint32_t p2)
{
    //data[threadIdx.x] = __mul24(threadIdx.x,p0) + __mul24(threadIdx.y,0x1234);
    data[threadIdx.x] = threadIdx.x;
}
#if 0
int main()
{
    int width = 8;
    int size = width*4;

    uint32_t *data, *gdata;

    cudaMalloc((void**)&gdata, size);
    data = (uint32_t*)malloc(size);

    dim3 block_size(8,1,1);
    dim3 grid_size(1,1,1);
    int shared_size = 0;

    my_kernel<<<grid_size, block_size, shared_size>>>(gdata, 100);

    cudaMemcpy((void*)data, (void*)gdata, size, cudaMemcpyDeviceToHost);

    for(int x=0; x<width; ++x)
        printf("%08x ", data[x]);
    printf("\n");
    return 0;
}
#endif