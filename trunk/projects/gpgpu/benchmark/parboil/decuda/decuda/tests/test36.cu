
#include <hip/hip_runtime.h>
__constant__ int uu;
__constant__ short vv;
__constant__ int ww[10];
__device__ int xx;
__device__ int *xx2;

__global__ void f_test(unsigned int *data, unsigned char *data2)
{
	for(unsigned int x=0; x<10; ++x)
	{
		unsigned int y = data[x-2];
		data2[x] = data[x];	
		data2[x+2] = y>>16;
	}
	data[0] = uu;
	data[1] = vv;
	for(unsigned int x=0; x<10; ++x)
	{
            data[x] = ww[x];
	}
	xx = uu;
	xx2 = &xx;
}
