#include "hip/hip_runtime.h"
/*
**************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 **************************************************************************
*/
#include "gpufi.h" 
struct kValues
{
float Kx;
float Ky;
float Kz;
float PhiMag;
};

/*
Values in the k-space coordinate system are stored in constant memory
 * on the GPU
*/
__constant__ __device__ kValues ck[1024];
__global__ void ComputePhiMag_GPU(float * phiR, float * phiI, float * phiMag, int numK)
{
int indexK = ((blockIdx.x*512)+threadIdx.x);
if ((indexK<numK))
{
float real = phiR[indexK];
float imag = phiI[indexK];
phiMag[indexK]=((real*real)+(imag*imag));
}
}

__global__ void ComputeQ_GPU(int numK, int kGlobalIndex, float * x, float * y, float * z, float * Qr, float * Qi)
{
float sX;
float sY;
float sZ;
float sQr;
float sQi;
/* Determine the element of the X arrays computed by this thread */
int xIndex = ((blockIdx.x*256)+threadIdx.x);
/* Read block's X values from global mem to shared mem */
int kIndex = 0;
sX=x[xIndex];
sY=y[xIndex];
sZ=z[xIndex];
sQr=Qr[xIndex];
sQi=Qi[xIndex];
/* Loop over all elements of K in constant mem to compute a partial value */
/* for X. */
if ((numK%2))
{
float expArg = (6.283185307179586*(((ck[0].Kx*sX)+(ck[0].Ky*sY))+(ck[0].Kz*sZ)));
sQr+=(ck[0].PhiMag*cos(expArg));
sQi+=(ck[0].PhiMag*sin(expArg));
kIndex ++ ;
kGlobalIndex ++ ;
}
for (; (kIndex<1024)&&(kGlobalIndex<numK); ((kIndex+=2), (kGlobalIndex+=2)))
{
float expArg = (6.283185307179586*(((ck[kIndex].Kx*sX)+(ck[kIndex].Ky*sY))+(ck[kIndex].Kz*sZ)));
int kIndex1;
float expArg1;
sQr+=(ck[kIndex].PhiMag*cos(expArg));
sQi+=(ck[kIndex].PhiMag*sin(expArg));
kIndex1=(kIndex+1);
expArg1=(6.283185307179586*(((ck[kIndex1].Kx*sX)+(ck[kIndex1].Ky*sY))+(ck[kIndex1].Kz*sZ)));
sQr+=(ck[kIndex1].PhiMag*cos(expArg1));
sQi+=(ck[kIndex1].PhiMag*sin(expArg1));
}
Qr[xIndex]=sQr;
Qi[xIndex]=sQi;
}

void computePhiMag_GPU(int numK, float * phiR_d, float * phiI_d, float * phiMag_d)
{
int phiMagBlocks = (numK/512);
dim3 DimPhiMagBlock;
dim3 DimPhiMagGrid;
if ((numK%512))
{
phiMagBlocks ++ ;
}
DimPhiMagBlock.x=512;
DimPhiMagBlock.y=1;
DimPhiMagBlock.z=1;
DimPhiMagGrid.x=phiMagBlocks;
DimPhiMagGrid.y=1;
DimPhiMagGrid.z=1;
ComputePhiMag_GPU<<<DimPhiMagGrid,DimPhiMagBlock>>>(phiR_d, phiI_d, phiMag_d, numK);
}

void computeQ_GPU(int numK, int numX, float * x_d, float * y_d, float * z_d, kValues * kVals, float * Qr_d, float * Qi_d)
{
int QGrids = (numK/1024);
int QBlocks;
dim3 DimQBlock;
dim3 DimQGrid;
int QGrid;
if ((numK%1024))
{
QGrids ++ ;
}
QBlocks=(numX/256);
if ((numX%256))
{
QBlocks ++ ;
}
DimQBlock.x=256;
DimQBlock.y=1;
DimQBlock.z=1;
DimQGrid.x=QBlocks;
DimQGrid.y=1;
DimQGrid.z=1;
/* printf("Launch GPU kernel: %d x (%d, %d) x (%d, %d); %d\n", */
/* QGrids, DimQGrid.x, DimQGrid.y, DimQBlock.x, DimQBlock.y, */
/* KERNEL_Q_K_ELEMS_PER_GRID); */
for (QGrid=0; QGrid<QGrids; QGrid ++ )
{
/* Put the tile of K values into constant mem */
int QGridBase = (QGrid*1024);
kValues * kValsTile = (kVals+QGridBase);
int numElems = ((1024<(numK-QGridBase)) ? 1024 : (numK-QGridBase));
hipMemcpyToSymbol(HIP_SYMBOL(ck), kValsTile, (numElems*sizeof (kValues)), 0);
ComputeQ_GPU<<<DimQGrid, DimQBlock>>>(numK, QGridBase, x_d, y_d, z_d, Qr_d, Qi_d);
}
}

void createDataStructsCPU(int numK, int numX, float *  * phiMag, float *  * Qr, float *  * Qi)
{
( * phiMag)=((float * )memalign(16, (numK*sizeof (float))));
( * Qr)=((float * )memalign(16, (numX*sizeof (float))));
( * Qi)=((float * )memalign(16, (numX*sizeof (float))));
}

