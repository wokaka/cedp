#include "hip/hip_runtime.h"
/*
**************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 **************************************************************************
*/
#include "gpufi.h"

struct kValues
{
    float Kx;
    float Ky;
    float Kz;
    float PhiMag;
};

/*
Values in the k-space coordinate system are stored in constant memory
 * on the GPU
*/
__constant__ __device__ kValues ck[1024];
//__constant__ __device__ kValues *ck_dup = ck;

#define GPUFI_CHECK(n1,n2) { if(n1 != n2) GPUFI_EXIT(0);}

__global__ void ComputePhiMag_GPU(float * phiR, float * phiI, float * phiMag, int numK)
{
		int numK_dup = numK;
		//ed
		float *phiR_dup = phiR;
		float *phiI_dup = phiI;
		float *phiMag_dup = phiMag;
		
    int indexK = ((blockIdx.x*512)+threadIdx.x);
    int indexK_dup = ((blockIdx.x*512)+threadIdx.x);
    
    GPUFI_CHECK(indexK,indexK_dup);
    GPUFI_CHECK(numK,numK_dup);
    if ((indexK<numK))
    {
        // [ED]
        if(phiR != phiR_dup) GPUFI_EXIT(0);
		    GPUFI_CHECK(indexK,indexK_dup);
        float real = phiR[indexK];
        float real_dup = phiR[indexK];
        
        // [ED]
        if(phiI != phiI_dup) GPUFI_EXIT(0);
		    GPUFI_CHECK(indexK,indexK_dup);
        float imag = phiI[indexK];
        float imag_dup = phiI[indexK];
        
        // [ED]
        if(phiMag != phiMag_dup) GPUFI_EXIT(0);
		    GPUFI_CHECK(indexK,indexK_dup);
		    GPUFI_CHECK(real,real_dup);
		    GPUFI_CHECK(imag,imag_dup);
        phiMag[indexK]=((real*real)+(imag*imag));
		    GPUFI_CHECK(phiMag[indexK],((real*real)+(imag*imag)));
    }
}

__global__ void ComputeQ_GPU(int numK, int kGlobalIndex, float * x, float * y, float * z, float * Qr, float * Qi)
{
		int numK_dup = numK;
		int kGlobalIndex_dup = kGlobalIndex;
		// ed
		float *x_dup = x;
		float *y_dup = y;
		float *z_dup = z;
		float *Qr_dup = Qr;
		float *Qi_dup = Qi;
		
    float sX;
    float sY;
    float sZ;
    float sQr;
    float sQi;
    /* Determine the element of the X arrays computed by this thread
     */
    // [ED]
    //if(x != x_dup)
    //    GPUFI_EXIT(0);


    int xIndex = ((blockIdx.x*256)+threadIdx.x);
    int xIndex_dup = ((blockIdx.x*256)+threadIdx.x);
    /* Read block's X values from global mem to shared mem
     */
    int kIndex = 0;
    int kIndex_dup = 0;

    // [ED]
    if(x != x_dup) GPUFI_EXIT(0);
    GPUFI_CHECK(xIndex,xIndex_dup);
    sX=x[xIndex];
    float sX_dup = x[xIndex];

    // [ED]
    if(y != y_dup) GPUFI_EXIT(0);
    GPUFI_CHECK(xIndex,xIndex_dup);
    sY=y[xIndex];
    float sY_dup = y[xIndex];
    
    // [ED]
    if(z != z_dup) GPUFI_EXIT(0);
    GPUFI_CHECK(xIndex,xIndex_dup);
    sZ=z[xIndex];
    float sZ_dup = z[xIndex];
    
    // [ED]
    if(Qr != Qr_dup) GPUFI_EXIT(0);
    GPUFI_CHECK(xIndex,xIndex_dup);
    sQr=Qr[xIndex];
    float sQr_dup = Qr[xIndex];
    
    // [ED]
    if(Qi != Qi_dup) GPUFI_EXIT(0);
    GPUFI_CHECK(xIndex,xIndex_dup);
    sQi=Qi[xIndex];
    float sQi_dup = Qi[xIndex];
    
    /* Loop over all elements of K in constant mem to compute a partial value
     */
    /* for X.
     */
    GPUFI_CHECK(numK,numK_dup);
    if ((numK%2))
    {
		    // [ED]
		    //if(ck != ck_dup) GPUFI_EXIT(0);
		    GPUFI_CHECK(sX,sX_dup);
		    GPUFI_CHECK(sY,sY_dup);
		    GPUFI_CHECK(sZ,sZ_dup);
        float expArg = (6.283185307179586*(((ck[0].Kx*sX)+(ck[0].Ky*sY))+(ck[0].Kz*sZ)));
        float expArg_dup = (6.283185307179586*(((ck[0].Kx*sX)+(ck[0].Ky*sY))+(ck[0].Kz*sZ)));
		    //if(ck != ck_dup) GPUFI_EXIT(0);
		    GPUFI_CHECK(expArg,expArg_dup);
        sQr+=(ck[0].PhiMag*cos(expArg));
		    //if(ck != ck_dup) GPUFI_EXIT(0);
		    GPUFI_CHECK(expArg,expArg_dup);
        sQi+=(ck[0].PhiMag*sin(expArg));
		    GPUFI_CHECK(kIndex,kIndex_dup);
        kIndex ++ ;
        kIndex_dup++;
		    GPUFI_CHECK(kGlobalIndex,kGlobalIndex_dup);
        kGlobalIndex ++ ;
        kGlobalIndex_dup ++ ;
    }
    for (; (kIndex<1024&&kIndex==kIndex_dup)&&(kGlobalIndex<numK&&kGlobalIndex==kGlobalIndex_dup&&numK==numK_dup); ((kIndex+=2), (kGlobalIndex+=2), kIndex_dup+=2, kGlobalIndex_dup+=2))
    {
        
		    //if(ck != ck_dup) GPUFI_EXIT(0);
		    GPUFI_CHECK(kIndex,kIndex_dup);
		    GPUFI_CHECK(sX,sX_dup);
		    GPUFI_CHECK(sY,sY_dup);
		    GPUFI_CHECK(sZ,sZ_dup);
        float expArg = (6.283185307179586*(((ck[kIndex].Kx*sX)+(ck[kIndex].Ky*sY))+(ck[kIndex].Kz*sZ)));
        float expArg_dup = (6.283185307179586*(((ck[kIndex].Kx*sX)+(ck[kIndex].Ky*sY))+(ck[kIndex].Kz*sZ)));
        int kIndex1;
        int kIndex1_dup;
        float expArg1;
        float expArg1_dup;
		    //if(ck != ck_dup) GPUFI_EXIT(0);
		    GPUFI_CHECK(kIndex,kIndex_dup);
		    GPUFI_CHECK(expArg,expArg_dup);
        sQr+=(ck[kIndex].PhiMag*cos(expArg));
        sQr_dup+=(ck[kIndex].PhiMag*cos(expArg));
		    //if(ck != ck_dup) GPUFI_EXIT(0);
		    GPUFI_CHECK(kIndex,kIndex_dup);
		    GPUFI_CHECK(expArg,expArg_dup);
        sQi+=(ck[kIndex].PhiMag*sin(expArg));
		    GPUFI_CHECK(expArg,expArg_dup);
        sQi_dup+=(ck[kIndex].PhiMag*sin(expArg));
		    GPUFI_CHECK(kIndex,kIndex_dup);
        kIndex1_dup=(kIndex+1);
        kIndex1=(kIndex+1);
		    //if(ck != ck_dup) GPUFI_EXIT(0);
		    GPUFI_CHECK(kIndex1,kIndex1_dup);
		    GPUFI_CHECK(sX,sX_dup);
		    GPUFI_CHECK(sY,sY_dup);
		    GPUFI_CHECK(sZ,sZ_dup);
        expArg1=(6.283185307179586*(((ck[kIndex1].Kx*sX)+(ck[kIndex1].Ky*sY))+(ck[kIndex1].Kz*sZ)));
        expArg1_dup=(6.283185307179586*(((ck[kIndex1].Kx*sX)+(ck[kIndex1].Ky*sY))+(ck[kIndex1].Kz*sZ)));
		    //if(ck != ck_dup) GPUFI_EXIT(0);
		    GPUFI_CHECK(kIndex1,kIndex1_dup);
		    GPUFI_CHECK(expArg,expArg_dup);
        sQr+=(ck[kIndex1].PhiMag*cos(expArg1));
		    GPUFI_CHECK(expArg,expArg_dup);
        sQr_dup+=(ck[kIndex1].PhiMag*cos(expArg1));
		    //if(ck != ck_dup) GPUFI_EXIT(0);
		    GPUFI_CHECK(kIndex1,kIndex1_dup);
		    GPUFI_CHECK(expArg,expArg_dup);
        sQi+=(ck[kIndex1].PhiMag*sin(expArg1));
		    GPUFI_CHECK(expArg,expArg_dup);
        sQi_dup+=(ck[kIndex1].PhiMag*sin(expArg1));
    }
    // [ED]
    if(Qr != Qr_dup) GPUFI_EXIT(0);
    GPUFI_CHECK(xIndex,xIndex_dup);
    Qr[xIndex]=sQr;
    GPUFI_CHECK(Qr[xIndex],sQr);

    // [ED]
    if(Qi != Qi_dup) GPUFI_EXIT(0);
    GPUFI_CHECK(xIndex,xIndex_dup);
    Qi[xIndex]=sQi;
    GPUFI_CHECK(Qi[xIndex],sQi);
}

void computePhiMag_GPU(int numK, float * phiR_d, float * phiI_d, float * phiMag_d)
{
    int phiMagBlocks = (numK/512);
    dim3 DimPhiMagBlock;
    dim3 DimPhiMagGrid;
    if ((numK%512))
    {
        phiMagBlocks ++ ;
    }
    DimPhiMagBlock.x=512;
    DimPhiMagBlock.y=1;
    DimPhiMagBlock.z=1;
    DimPhiMagGrid.x=phiMagBlocks;
    DimPhiMagGrid.y=1;
    DimPhiMagGrid.z=1;
    ComputePhiMag_GPU<<<DimPhiMagGrid, DimPhiMagBlock>>> (phiR_d, phiI_d, phiMag_d, numK);
}

void computeQ_GPU(int numK, int numX, float * x_d, float * y_d, float * z_d, kValues * kVals, float * Qr_d, float * Qi_d)
{
    int QGrids = (numK/1024);
    int QBlocks;
    dim3 DimQBlock;
    dim3 DimQGrid;
    int QGrid;
    if ((numK%1024))
    {
        QGrids ++ ;
    }
    QBlocks=(numX/256);
    if ((numX%256))
    {
        QBlocks ++ ;
    }
    DimQBlock.x=256;
    DimQBlock.y=1;
    DimQBlock.z=1;
    DimQGrid.x=QBlocks;
    DimQGrid.y=1;
    DimQGrid.z=1;
    /* printf("Launch GPU kernel: %d x (%d, %d) x (%d, %d); %d\n",
     */
    /* QGrids, DimQGrid.x, DimQGrid.y, DimQBlock.x, DimQBlock.y,
     */
    /* KERNEL_Q_K_ELEMS_PER_GRID);
     */
    for (QGrid=0; QGrid<QGrids; QGrid ++ )
    {
        /* Put the tile of K values into constant mem
         */
        int QGridBase = (QGrid*1024);
        kValues * kValsTile = (kVals+QGridBase);
        int numElems = ((1024<(numK-QGridBase)) ? 1024 : (numK-QGridBase));
        hipMemcpyToSymbol(HIP_SYMBOL(ck), kValsTile, (numElems*sizeof (kValues)), 0);
        ComputeQ_GPU<<<DimQGrid, DimQBlock>>> (numK, QGridBase, x_d, y_d, z_d, Qr_d, Qi_d);
    }
}

void createDataStructsCPU(int numK, int numX, float *  * phiMag, float *  * Qr, float *  * Qi)
{
    ( * phiMag)=((float * )memalign(16, (numK*sizeof (float))));
    ( * Qr)=((float * )memalign(16, (numX*sizeof (float))));
    ( * Qi)=((float * )memalign(16, (numX*sizeof (float))));
}
