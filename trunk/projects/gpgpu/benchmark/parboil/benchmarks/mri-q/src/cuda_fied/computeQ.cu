#include "hip/hip_runtime.h"
/* Instrumented by FaultInjector */
#include "gpufi.h"

/*
**************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 **************************************************************************
*/
#include "gpufi.h"

struct kValues
{
    float Kx;
    float Ky;
    float Kz;
    float PhiMag;
};

/*
Values in the k-space coordinate system are stored in constant memory
 * on the GPU
*/
__constant__ kValues ck[1024];
__global__ void ComputePhiMag_GPU(float * phiR, float * phiI, float * phiMag, int numK,  struct _gpufi_data_ *gpufi_dev)
{
    // [ED]
    int numK_dup = numK;


    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_BEGIN, 0, "ComputePhiMag_GPU");
    int indexK = ((blockIdx.x*512)+threadIdx.x);
    // [ED]
    int indexK_dup = ((blockIdx.x*512)+threadIdx.x);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 0, "indexK", (int*)&indexK);


    // [ED]
    if(indexK != indexK_dup)
        GPUFI_EXIT(0);


    // [ED]
    if(numK != numK_dup)
        GPUFI_EXIT(0);


    if ((indexK<numK))
    {
        float real = phiR[indexK];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 1, "real", (int*)&real);


        float imag = phiI[indexK];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 2, "imag", (int*)&imag);


        phiMag[indexK]=((real*real)+(imag*imag));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 3, "phiMag[indexK]", (int*)&phiMag[indexK]);


    }
    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_END, 0, "ComputePhiMag_GPU");
}

__global__ void ComputeQ_GPU(int numK, int kGlobalIndex, float * x, float * y, float * z, float * Qr, float * Qi,  struct _gpufi_data_ *gpufi_dev)
{
    // [ED]
    int kGlobalIndex_dup = kGlobalIndex;


    // [ED]
    int numK_dup = numK;


    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_BEGIN, 1, "ComputeQ_GPU");
    float sX;
    float sY;
    float sZ;
    float sQr;
    float sQi;
    /* Determine the element of the X arrays computed by this thread
     */
    int xIndex = ((blockIdx.x*256)+threadIdx.x);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 4, "xIndex", (int*)&xIndex);


    /* Read block's X values from global mem to shared mem
     */
    int kIndex = 0;
    // [ED]
    int kIndex_dup = 0;


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 5, "kIndex", (int*)&kIndex);


    sX=x[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 6, "sX", (int*)&sX);


    sY=y[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 7, "sY", (int*)&sY);


    sZ=z[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 8, "sZ", (int*)&sZ);


    sQr=Qr[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 9, "sQr", (int*)&sQr);


    sQi=Qi[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 10, "sQi", (int*)&sQi);


    /* Loop over all elements of K in constant mem to compute a partial value
     */
    /* for X.
     */
    // [ED]
    if(numK != numK_dup)
        GPUFI_EXIT(0);


    if ((numK%2))
    {
        float expArg = (6.283185307179586*(((ck[0].Kx*sX)+(ck[0].Ky*sY))+(ck[0].Kz*sZ)));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 11, "expArg", (int*)&expArg);


        sQr+=(ck[0].PhiMag*cos(expArg));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 12, "sQr", (int*)&sQr);


        sQi+=(ck[0].PhiMag*sin(expArg));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 13, "sQi", (int*)&sQi);


        kIndex ++ ;
        kGlobalIndex ++ ;
    }
    /* Fault Injector Instrumentation Begin */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_BEGIN);
    /* Fault Injector Instrumentation End */

    // [ED]
    if(kGlobalIndex != kGlobalIndex_dup)
        GPUFI_EXIT(0);


    // [ED]
    if(kIndex != kIndex_dup)
        GPUFI_EXIT(0);


    // [ED]
    if(numK != numK_dup)
        GPUFI_EXIT(0);


    for (; (kIndex<1024)&&(kGlobalIndex<numK); ((kIndex+=2), (kGlobalIndex+=2)))
    {
        /* Fault Injector Instrumentation Begin */
        GPUFI_KERNEL_ITERATION(gpufi_dev);
        /* Fault Injector Instrumentation End */

        float expArg = (6.283185307179586*(((ck[kIndex].Kx*sX)+(ck[kIndex].Ky*sY))+(ck[kIndex].Kz*sZ)));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 14, "expArg", (int*)&expArg);


        int kIndex1;
        float expArg1;
        sQr+=(ck[kIndex].PhiMag*cos(expArg));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 15, "sQr", (int*)&sQr);


        sQi+=(ck[kIndex].PhiMag*sin(expArg));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 16, "sQi", (int*)&sQi);


        kIndex1=(kIndex+1);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 17, "kIndex1", (int*)&kIndex1);


        expArg1=(6.283185307179586*(((ck[kIndex1].Kx*sX)+(ck[kIndex1].Ky*sY))+(ck[kIndex1].Kz*sZ)));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 18, "expArg1", (int*)&expArg1);


        sQr+=(ck[kIndex1].PhiMag*cos(expArg1));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 19, "sQr", (int*)&sQr);


        sQi+=(ck[kIndex1].PhiMag*sin(expArg1));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 20, "sQi", (int*)&sQi);


    }
    /* Automatically inserted by Cetus_Cuda */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_END);
    /* Fault Injector Instrumentation End *//* End of Cetus_Cuda code */

    Qr[xIndex]=sQr;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 21, "Qr[xIndex]", (int*)&Qr[xIndex]);


    Qi[xIndex]=sQi;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 22, "Qi[xIndex]", (int*)&Qi[xIndex]);


    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_END, 1, "ComputeQ_GPU");
}

void computePhiMag_GPU(int numK, float * phiR_d, float * phiI_d, float * phiMag_d)
{
    int phiMagBlocks = (numK/512);
    dim3 DimPhiMagBlock;
    dim3 DimPhiMagGrid;
    if ((numK%512))
    {
        phiMagBlocks ++ ;
    }
    DimPhiMagBlock.x=512;
    DimPhiMagBlock.y=1;
    DimPhiMagBlock.z=1;
    DimPhiMagGrid.x=phiMagBlocks;
    DimPhiMagGrid.y=1;
    DimPhiMagGrid.z=1;
    /* Fault Injection Instrumentation Begin */
    hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
    CUDA_ERRCK
    hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
    CUDA_ERRCK
    /* Fault Injection Instrumentation End */

    ComputePhiMag_GPU<<<DimPhiMagGrid, DimPhiMagBlock>>> (phiR_d, phiI_d, phiMag_d, numK, gpufi_dev);
    /* Fault Injection Instrumentation Begin */
    hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
    CUDA_ERRCK
    hipFree(gpufi_dev);
    CUDA_ERRCK
    /* Fault Injection Instrumentation End */

}

void computeQ_GPU(int numK, int numX, float * x_d, float * y_d, float * z_d, kValues * kVals, float * Qr_d, float * Qi_d)
{
    int QGrids = (numK/1024);
    int QBlocks;
    dim3 DimQBlock;
    dim3 DimQGrid;
    int QGrid;
    if ((numK%1024))
    {
        QGrids ++ ;
    }
    QBlocks=(numX/256);
    if ((numX%256))
    {
        QBlocks ++ ;
    }
    DimQBlock.x=256;
    DimQBlock.y=1;
    DimQBlock.z=1;
    DimQGrid.x=QBlocks;
    DimQGrid.y=1;
    DimQGrid.z=1;
    /* printf("Launch GPU kernel: %d x (%d, %d) x (%d, %d); %d\n",
     */
    /* QGrids, DimQGrid.x, DimQGrid.y, DimQBlock.x, DimQBlock.y,
     */
    /* KERNEL_Q_K_ELEMS_PER_GRID);
     */
    for (QGrid=0; QGrid<QGrids; QGrid ++ )
    {
        /* Put the tile of K values into constant mem
         */
        int QGridBase = (QGrid*1024);
        kValues * kValsTile = (kVals+QGridBase);
        int numElems = ((1024<(numK-QGridBase)) ? 1024 : (numK-QGridBase));
        hipMemcpyToSymbol(HIP_SYMBOL(ck), kValsTile, (numElems*sizeof (kValues)), 0);
        /* Fault Injection Instrumentation Begin */
        hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
        CUDA_ERRCK
        hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

        ComputeQ_GPU<<<DimQGrid, DimQBlock>>> (numK, QGridBase, x_d, y_d, z_d, Qr_d, Qi_d, gpufi_dev);
        /* Fault Injection Instrumentation Begin */
        hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
        CUDA_ERRCK
        hipFree(gpufi_dev);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

    }
}

void createDataStructsCPU(int numK, int numX, float *  * phiMag, float *  * Qr, float *  * Qi)
{
    ( * phiMag)=((float * )memalign(16, (numK*sizeof (float))));
    ( * Qr)=((float * )memalign(16, (numX*sizeof (float))));
    ( * Qi)=((float * )memalign(16, (numX*sizeof (float))));
}
