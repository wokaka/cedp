#include "hip/hip_runtime.h"
/* Instrumented by FaultInjector */
#include "gpufi.h"

/*
**************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 **************************************************************************
*/
#include "gpufi.h"

struct kValues
{
    float Kx;
    float Ky;
    float Kz;
    float PhiMag;
};

/*
Values in the k-space coordinate system are stored in constant memory
 * on the GPU
*/
__constant__ __device__ kValues ck[1024];
__global__ void ComputePhiMag_GPU(float * phiR, float * phiI, float * phiMag, int numK,  struct _gpufi_data_ *gpufi_dev)
{
    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_BEGIN, 0, "ComputePhiMag_GPU");
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 0, "phiR", (int*)&phiR, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 1, "phiI", (int*)&phiI, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 2, "phiMag", (int*)&phiMag, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 3, "numK", (int*)&numK, GPUFI_DATATYPE_INTEGER);


    int indexK = ((blockIdx.x*512)+threadIdx.x);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 11, "indexK", (int*)&indexK, GPUFI_DATATYPE_INTEGER);


    if ((indexK<numK))
    {
        float real = phiR[indexK];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 12, "real", (int*)&real, GPUFI_DATATYPE_FLOAT);


        float imag = phiI[indexK];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 13, "imag", (int*)&imag, GPUFI_DATATYPE_FLOAT);


        phiMag[indexK]=((real*real)+(imag*imag));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 14, "phiMag[indexK]", (int*)&phiMag[indexK], GPUFI_DATATYPE_FLOAT);


    }
    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_END, 0, "ComputePhiMag_GPU");
}

__global__ void ComputeQ_GPU(int numK, int kGlobalIndex, float * x, float * y, float * z, float * Qr, float * Qi,  struct _gpufi_data_ *gpufi_dev)
{
    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_BEGIN, 1, "ComputeQ_GPU");
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 10, "Qi", (int*)&Qi, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 4, "numK", (int*)&numK, GPUFI_DATATYPE_INTEGER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 5, "kGlobalIndex", (int*)&kGlobalIndex, GPUFI_DATATYPE_INTEGER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 6, "x", (int*)&x, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 7, "y", (int*)&y, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 8, "z", (int*)&z, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 9, "Qr", (int*)&Qr, GPUFI_DATATYPE_FLOAT_POINTER);


    float sX;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 15, "sX", (int*)&sX, GPUFI_DATATYPE_FLOAT);


    float sY;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 16, "sY", (int*)&sY, GPUFI_DATATYPE_FLOAT);


    float sZ;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 17, "sZ", (int*)&sZ, GPUFI_DATATYPE_FLOAT);


    float sQr;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 18, "sQr", (int*)&sQr, GPUFI_DATATYPE_FLOAT);


    float sQi;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 19, "sQi", (int*)&sQi, GPUFI_DATATYPE_FLOAT);


    /* Determine the element of the X arrays computed by this thread
     */
    int xIndex = ((blockIdx.x*256)+threadIdx.x);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 20, "xIndex", (int*)&xIndex, GPUFI_DATATYPE_INTEGER);


    /* Read block's X values from global mem to shared mem
     */
    int kIndex = 0;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 21, "kIndex", (int*)&kIndex, GPUFI_DATATYPE_INTEGER);


    sX=x[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 22, "sX", (int*)&sX, GPUFI_DATATYPE_FLOAT);


    sY=y[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 23, "sY", (int*)&sY, GPUFI_DATATYPE_FLOAT);


    sZ=z[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 24, "sZ", (int*)&sZ, GPUFI_DATATYPE_FLOAT);


    sQr=Qr[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 25, "sQr", (int*)&sQr, GPUFI_DATATYPE_FLOAT);


    sQi=Qi[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 26, "sQi", (int*)&sQi, GPUFI_DATATYPE_FLOAT);


    /* Loop over all elements of K in constant mem to compute a partial value
     */
    /* for X.
     */
    if ((numK%2))
    {
        float expArg = (6.283185307179586*(((ck[0].Kx*sX)+(ck[0].Ky*sY))+(ck[0].Kz*sZ)));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 27, "expArg", (int*)&expArg, GPUFI_DATATYPE_FLOAT);


        sQr+=(ck[0].PhiMag*cos(expArg));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 28, "sQr", (int*)&sQr, GPUFI_DATATYPE_FLOAT);


        sQi+=(ck[0].PhiMag*sin(expArg));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 29, "sQi", (int*)&sQi, GPUFI_DATATYPE_FLOAT);


        kIndex ++ ;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 30, "kIndex", (int*)&kIndex, GPUFI_DATATYPE_INTEGER);


        kGlobalIndex ++ ;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 31, "kGlobalIndex", (int*)&kGlobalIndex, GPUFI_DATATYPE_INTEGER);


    }
    /* Fault Injector Instrumentation Begin */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_BEGIN);
    /* Fault Injector Instrumentation End */

    for (; (kIndex<1024)&&(kGlobalIndex<numK); ((kIndex+=2), (kGlobalIndex+=2)))
    {
        /* Fault Injector Instrumentation Begin */
        GPUFI_KERNEL_ITERATION(gpufi_dev);
        /* Fault Injector Instrumentation End */

        float expArg = (6.283185307179586*(((ck[kIndex].Kx*sX)+(ck[kIndex].Ky*sY))+(ck[kIndex].Kz*sZ)));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 34, "expArg", (int*)&expArg, GPUFI_DATATYPE_FLOAT);


        int kIndex1;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 35, "kIndex1", (int*)&kIndex1, GPUFI_DATATYPE_INTEGER);


        float expArg1;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 36, "expArg1", (int*)&expArg1, GPUFI_DATATYPE_FLOAT);


        sQr+=(ck[kIndex].PhiMag*cos(expArg));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 37, "sQr", (int*)&sQr, GPUFI_DATATYPE_FLOAT);


        sQi+=(ck[kIndex].PhiMag*sin(expArg));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 38, "sQi", (int*)&sQi, GPUFI_DATATYPE_FLOAT);


        kIndex1=(kIndex+1);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 39, "kIndex1", (int*)&kIndex1, GPUFI_DATATYPE_INTEGER);


        expArg1=(6.283185307179586*(((ck[kIndex1].Kx*sX)+(ck[kIndex1].Ky*sY))+(ck[kIndex1].Kz*sZ)));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 40, "expArg1", (int*)&expArg1, GPUFI_DATATYPE_FLOAT);


        sQr+=(ck[kIndex1].PhiMag*cos(expArg1));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 41, "sQr", (int*)&sQr, GPUFI_DATATYPE_FLOAT);


        sQi+=(ck[kIndex1].PhiMag*sin(expArg1));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 42, "sQi", (int*)&sQi, GPUFI_DATATYPE_FLOAT);


    }
    /* Automatically inserted by Cetus_Cuda */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_END);
    /* Fault Injector Instrumentation End *//* End of Cetus_Cuda code */

    GPUFI_KERNEL_VARIABLE(gpufi_dev, 32, "kIndex", (int*)&kIndex, GPUFI_DATATYPE_INTEGER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 33, "kGlobalIndex", (int*)&kGlobalIndex, GPUFI_DATATYPE_INTEGER);


    Qr[xIndex]=sQr;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 43, "Qr[xIndex]", (int*)&Qr[xIndex], GPUFI_DATATYPE_FLOAT);


    Qi[xIndex]=sQi;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 44, "Qi[xIndex]", (int*)&Qi[xIndex], GPUFI_DATATYPE_FLOAT);


    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_END, 1, "ComputeQ_GPU");
}

void computePhiMag_GPU(int numK, float * phiR_d, float * phiI_d, float * phiMag_d)
{
    int phiMagBlocks = (numK/512);
    dim3 DimPhiMagBlock;
    dim3 DimPhiMagGrid;
    if ((numK%512))
    {
        phiMagBlocks ++ ;
    }
    DimPhiMagBlock.x=512;
    DimPhiMagBlock.y=1;
    DimPhiMagBlock.z=1;
    DimPhiMagGrid.x=phiMagBlocks;
    DimPhiMagGrid.y=1;
    DimPhiMagGrid.z=1;
    /* Fault Injection Instrumentation Begin */
    hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
    CUDA_ERRCK
    hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
    CUDA_ERRCK
    /* Fault Injection Instrumentation End */

    ComputePhiMag_GPU<<<DimPhiMagGrid, DimPhiMagBlock>>> (phiR_d, phiI_d, phiMag_d, numK, gpufi_dev);
    /* Fault Injection Instrumentation Begin */
    hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
    CUDA_ERRCK
    hipFree(gpufi_dev);
    CUDA_ERRCK
    /* Fault Injection Instrumentation End */

}

void computeQ_GPU(int numK, int numX, float * x_d, float * y_d, float * z_d, kValues * kVals, float * Qr_d, float * Qi_d)
{
    int QGrids = (numK/1024);
    int QBlocks;
    dim3 DimQBlock;
    dim3 DimQGrid;
    int QGrid;
    if ((numK%1024))
    {
        QGrids ++ ;
    }
    QBlocks=(numX/256);
    if ((numX%256))
    {
        QBlocks ++ ;
    }
    DimQBlock.x=256;
    DimQBlock.y=1;
    DimQBlock.z=1;
    DimQGrid.x=QBlocks;
    DimQGrid.y=1;
    DimQGrid.z=1;
    /* printf("Launch GPU kernel: %d x (%d, %d) x (%d, %d); %d\n",
     */
    /* QGrids, DimQGrid.x, DimQGrid.y, DimQBlock.x, DimQBlock.y,
     */
    /* KERNEL_Q_K_ELEMS_PER_GRID);
     */
    for (QGrid=0; QGrid<QGrids; QGrid ++ )
    {
        /* Put the tile of K values into constant mem
         */
        int QGridBase = (QGrid*1024);
        kValues * kValsTile = (kVals+QGridBase);
        int numElems = ((1024<(numK-QGridBase)) ? 1024 : (numK-QGridBase));
        hipMemcpyToSymbol(HIP_SYMBOL(ck), kValsTile, (numElems*sizeof (kValues)), 0);
        /* Fault Injection Instrumentation Begin */
        hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
        CUDA_ERRCK
        hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

        ComputeQ_GPU<<<DimQGrid, DimQBlock>>> (numK, QGridBase, x_d, y_d, z_d, Qr_d, Qi_d, gpufi_dev);
        /* Fault Injection Instrumentation Begin */
        hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
        CUDA_ERRCK
        hipFree(gpufi_dev);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

    }
}

void createDataStructsCPU(int numK, int numX, float *  * phiMag, float *  * Qr, float *  * Qi)
{
    ( * phiMag)=((float * )memalign(16, (numK*sizeof (float))));
    ( * Qr)=((float * )memalign(16, (numX*sizeof (float))));
    ( * Qi)=((float * )memalign(16, (numX*sizeof (float))));
}
