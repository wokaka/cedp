#include "hip/hip_runtime.h"
/* Instrumented by FaultInjector */
#include "gpufi.h"

/*
**************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 **************************************************************************
*/
#include "gpufi.h"

/* #include <stdio.h>
 */
#include "cuenergy.h"

/* typedef double float4;
 */
/* Max constant buffer size is 64KB, minus whatever
 */
/* the CUDA runtime and compiler are using that we don't know about.
 */
/* At 16 bytes for atom, for this program 4070 atoms is about the max
 */
/* we can store in the constant buffer.
 */
__constant__ float4 atominfo[4000];
/* This kernel calculates coulombic potential at each grid point and
 */
/* stores the results in the output array.
 */
__global__ void cenergy(int numatoms, float gridspacing, float * energygrid,  struct _gpufi_data_ *gpufi_dev)
{
    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_BEGIN, 0, "cenergy");
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 0, "numatoms", (int*)&numatoms, GPUFI_DATATYPE_INTEGER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 1, "gridspacing", (int*)&gridspacing, GPUFI_DATATYPE_FLOAT);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 2, "energygrid", (int*)&energygrid, GPUFI_DATATYPE_FLOAT_POINTER);


    unsigned int xindex = ((__umul24(blockIdx.x, blockDim.x)*8)+threadIdx.x);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 3, "xindex", (int*)&xindex, GPUFI_DATATYPE_INTEGER);


    unsigned int yindex = (__umul24(blockIdx.y, blockDim.y)+threadIdx.y);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 4, "yindex", (int*)&yindex, GPUFI_DATATYPE_INTEGER);


    unsigned int outaddr = (((__umul24(gridDim.x, blockDim.x)*8)*yindex)+xindex);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 5, "outaddr", (int*)&outaddr, GPUFI_DATATYPE_INTEGER);


    float coory = (gridspacing*yindex);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 6, "coory", (int*)&coory, GPUFI_DATATYPE_FLOAT);


    float coorx = (gridspacing*xindex);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 7, "coorx", (int*)&coorx, GPUFI_DATATYPE_FLOAT);


    float energyvalx1 = 0.0;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 8, "energyvalx1", (int*)&energyvalx1, GPUFI_DATATYPE_FLOAT);


    float energyvalx2 = 0.0;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 9, "energyvalx2", (int*)&energyvalx2, GPUFI_DATATYPE_FLOAT);


    float energyvalx3 = 0.0;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 10, "energyvalx3", (int*)&energyvalx3, GPUFI_DATATYPE_FLOAT);


    float energyvalx4 = 0.0;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 11, "energyvalx4", (int*)&energyvalx4, GPUFI_DATATYPE_FLOAT);


    float energyvalx5 = 0.0;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 12, "energyvalx5", (int*)&energyvalx5, GPUFI_DATATYPE_FLOAT);


    float energyvalx6 = 0.0;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 13, "energyvalx6", (int*)&energyvalx6, GPUFI_DATATYPE_FLOAT);


    float energyvalx7 = 0.0;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 14, "energyvalx7", (int*)&energyvalx7, GPUFI_DATATYPE_FLOAT);


    float energyvalx8 = 0.0;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 15, "energyvalx8", (int*)&energyvalx8, GPUFI_DATATYPE_FLOAT);


    float gridspacing_u = (gridspacing*16);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 16, "gridspacing_u", (int*)&gridspacing_u, GPUFI_DATATYPE_FLOAT);


    int atomid;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 17, "atomid", (int*)&atomid, GPUFI_DATATYPE_INTEGER);


    /* Fault Injector Instrumentation Begin */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_BEGIN);
    /* Fault Injector Instrumentation End */

    for (atomid=0; atomid<numatoms; atomid ++ )
    {
        /* Fault Injector Instrumentation Begin */
        GPUFI_KERNEL_ITERATION(gpufi_dev);
        /* Fault Injector Instrumentation End */

        float dy = (coory-atominfo[atomid].y);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 20, "dy", (int*)&dy, GPUFI_DATATYPE_FLOAT);


        float dyz2 = ((dy*dy)+atominfo[atomid].z);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 21, "dyz2", (int*)&dyz2, GPUFI_DATATYPE_FLOAT);


        float dx1 = (coorx-atominfo[atomid].x);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 22, "dx1", (int*)&dx1, GPUFI_DATATYPE_FLOAT);


        float dx2 = (dx1+gridspacing_u);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 23, "dx2", (int*)&dx2, GPUFI_DATATYPE_FLOAT);


        float dx3 = (dx2+gridspacing_u);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 24, "dx3", (int*)&dx3, GPUFI_DATATYPE_FLOAT);


        float dx4 = (dx3+gridspacing_u);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 25, "dx4", (int*)&dx4, GPUFI_DATATYPE_FLOAT);


        float dx5 = (dx4+gridspacing_u);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 26, "dx5", (int*)&dx5, GPUFI_DATATYPE_FLOAT);


        float dx6 = (dx5+gridspacing_u);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 27, "dx6", (int*)&dx6, GPUFI_DATATYPE_FLOAT);


        float dx7 = (dx6+gridspacing_u);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 28, "dx7", (int*)&dx7, GPUFI_DATATYPE_FLOAT);


        float dx8 = (dx7+gridspacing_u);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 29, "dx8", (int*)&dx8, GPUFI_DATATYPE_FLOAT);


        energyvalx1+=(atominfo[atomid].w*(1.0/sqrtf(((dx1*dx1)+dyz2))));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 30, "energyvalx1", (int*)&energyvalx1, GPUFI_DATATYPE_FLOAT);


        energyvalx2+=(atominfo[atomid].w*(1.0/sqrtf(((dx2*dx2)+dyz2))));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 31, "energyvalx2", (int*)&energyvalx2, GPUFI_DATATYPE_FLOAT);


        energyvalx3+=(atominfo[atomid].w*(1.0/sqrtf(((dx3*dx3)+dyz2))));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 32, "energyvalx3", (int*)&energyvalx3, GPUFI_DATATYPE_FLOAT);


        energyvalx4+=(atominfo[atomid].w*(1.0/sqrtf(((dx4*dx4)+dyz2))));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 33, "energyvalx4", (int*)&energyvalx4, GPUFI_DATATYPE_FLOAT);


        energyvalx5+=(atominfo[atomid].w*(1.0/sqrtf(((dx5*dx5)+dyz2))));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 34, "energyvalx5", (int*)&energyvalx5, GPUFI_DATATYPE_FLOAT);


        energyvalx6+=(atominfo[atomid].w*(1.0/sqrtf(((dx6*dx6)+dyz2))));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 35, "energyvalx6", (int*)&energyvalx6, GPUFI_DATATYPE_FLOAT);


        energyvalx7+=(atominfo[atomid].w*(1.0/sqrtf(((dx7*dx7)+dyz2))));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 36, "energyvalx7", (int*)&energyvalx7, GPUFI_DATATYPE_FLOAT);


        energyvalx8+=(atominfo[atomid].w*(1.0/sqrtf(((dx8*dx8)+dyz2))));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 37, "energyvalx8", (int*)&energyvalx8, GPUFI_DATATYPE_FLOAT);


    }
    /* Automatically inserted by Cetus_Cuda */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_END);
    /* Fault Injector Instrumentation End *//* End of Cetus_Cuda code */

    GPUFI_KERNEL_VARIABLE(gpufi_dev, 19, "atomid", (int*)&atomid, GPUFI_DATATYPE_INTEGER);


    energygrid[outaddr]+=energyvalx1;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 38, "energygrid[outaddr]", (int*)&energygrid[outaddr], GPUFI_DATATYPE_FLOAT);


    energygrid[(outaddr+(1*16))]+=energyvalx2;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 39, "energygrid[(outaddr+(1*16))]", (int*)&energygrid[(outaddr+(1*16))], GPUFI_DATATYPE_FLOAT);


    energygrid[(outaddr+(2*16))]+=energyvalx3;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 40, "energygrid[(outaddr+(2*16))]", (int*)&energygrid[(outaddr+(2*16))], GPUFI_DATATYPE_FLOAT);


    energygrid[(outaddr+(3*16))]+=energyvalx4;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 41, "energygrid[(outaddr+(3*16))]", (int*)&energygrid[(outaddr+(3*16))], GPUFI_DATATYPE_FLOAT);


    energygrid[(outaddr+(4*16))]+=energyvalx5;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 42, "energygrid[(outaddr+(4*16))]", (int*)&energygrid[(outaddr+(4*16))], GPUFI_DATATYPE_FLOAT);


    energygrid[(outaddr+(5*16))]+=energyvalx6;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 43, "energygrid[(outaddr+(5*16))]", (int*)&energygrid[(outaddr+(5*16))], GPUFI_DATATYPE_FLOAT);


    energygrid[(outaddr+(6*16))]+=energyvalx7;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 44, "energygrid[(outaddr+(6*16))]", (int*)&energygrid[(outaddr+(6*16))], GPUFI_DATATYPE_FLOAT);


    energygrid[(outaddr+(7*16))]+=energyvalx8;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 45, "energygrid[(outaddr+(7*16))]", (int*)&energygrid[(outaddr+(7*16))], GPUFI_DATATYPE_FLOAT);


    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_END, 0, "cenergy");
}

/* This function copies atoms from the CPU to the GPU and
 */
/* precalculates (z^2) for each atom.
 */
int copyatomstoconstbuf(float * atoms, int count, float zplane)
{
    float atompre[(4*4000)];
    int i;
    if ((count>4000))
    {
        printf("Atom count exceeds constant buffer storage capacity\n");
        return ( - 1);
    }
    for (i=0; i<(count*4); i+=4)
    {
        float dz;
        atompre[i]=atoms[i];
        atompre[(i+1)]=atoms[(i+1)];
        dz=(zplane-atoms[(i+2)]);
        atompre[(i+2)]=(dz*dz);
        atompre[(i+3)]=atoms[(i+3)];
    }
    hipMemcpyToSymbol(HIP_SYMBOL(atominfo), atompre, ((count*4)*sizeof (float)), 0);
    if (1)
    {
        ;
    }
    /* check and clear any existing errors
     */
    return 0;
}
