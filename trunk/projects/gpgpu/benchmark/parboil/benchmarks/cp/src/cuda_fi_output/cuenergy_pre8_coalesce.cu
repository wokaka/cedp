#include "hip/hip_runtime.h"
/*
**************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 **************************************************************************
*/
#include <stdio.h> 
#include "cuenergy.h" 
/* Max constant buffer size is 64KB, minus whatever */
/* the CUDA runtime and compiler are using that we don't know about. */
/* At 16 bytes for atom, for this program 4070 atoms is about the max */
/* we can store in the constant buffer. */
__constant__ float4 atominfo[4000];
/* This kernel calculates coulombic potential at each grid point and */
/* stores the results in the output array. */
__global__ void cenergy(int numatoms, float gridspacing, float * energygrid)
{
unsigned int xindex = ((__umul24(blockIdx.x, blockDim.x)*8)+threadIdx.x);
unsigned int yindex = (__umul24(blockIdx.y, blockDim.y)+threadIdx.y);
unsigned int outaddr = (((__umul24(gridDim.x, blockDim.x)*8)*yindex)+xindex);
float coory = (gridspacing*yindex);
float coorx = (gridspacing*xindex);
float energyvalx1 = 0.0;
float energyvalx2 = 0.0;
float energyvalx3 = 0.0;
float energyvalx4 = 0.0;
float energyvalx5 = 0.0;
float energyvalx6 = 0.0;
float energyvalx7 = 0.0;
float energyvalx8 = 0.0;
float gridspacing_u = (gridspacing*16);
int atomid;
for (atomid=0; atomid<numatoms; atomid ++ )
{
float dy = (coory-atominfo[atomid].y);
float dyz2 = ((dy*dy)+atominfo[atomid].z);
float dx1 = (coorx-atominfo[atomid].x);
float dx2 = (dx1+gridspacing_u);
float dx3 = (dx2+gridspacing_u);
float dx4 = (dx3+gridspacing_u);
float dx5 = (dx4+gridspacing_u);
float dx6 = (dx5+gridspacing_u);
float dx7 = (dx6+gridspacing_u);
float dx8 = (dx7+gridspacing_u);
energyvalx1+=(atominfo[atomid].w*(1.0/sqrtf(((dx1*dx1)+dyz2))));
energyvalx2+=(atominfo[atomid].w*(1.0/sqrtf(((dx2*dx2)+dyz2))));
energyvalx3+=(atominfo[atomid].w*(1.0/sqrtf(((dx3*dx3)+dyz2))));
energyvalx4+=(atominfo[atomid].w*(1.0/sqrtf(((dx4*dx4)+dyz2))));
energyvalx5+=(atominfo[atomid].w*(1.0/sqrtf(((dx5*dx5)+dyz2))));
energyvalx6+=(atominfo[atomid].w*(1.0/sqrtf(((dx6*dx6)+dyz2))));
energyvalx7+=(atominfo[atomid].w*(1.0/sqrtf(((dx7*dx7)+dyz2))));
energyvalx8+=(atominfo[atomid].w*(1.0/sqrtf(((dx8*dx8)+dyz2))));
}
energygrid[outaddr]+=energyvalx1;
energygrid[(outaddr+(1*16))]+=energyvalx2;
energygrid[(outaddr+(2*16))]+=energyvalx3;
energygrid[(outaddr+(3*16))]+=energyvalx4;
energygrid[(outaddr+(4*16))]+=energyvalx5;
energygrid[(outaddr+(5*16))]+=energyvalx6;
energygrid[(outaddr+(6*16))]+=energyvalx7;
energygrid[(outaddr+(7*16))]+=energyvalx8;
}

/* This function copies atoms from the CPU to the GPU and */
/* precalculates (z^2) for each atom. */
int copyatomstoconstbuf(float * atoms, int count, float zplane)
{
float atompre[(4*4000)];
int i;
if ((count>4000))
{
printf("Atom count exceeds constant buffer storage capacity\n");
return ( - 1);
}
for (i=0; i<(count*4); i+=4)
{
float dz;
atompre[i]=atoms[i];
atompre[(i+1)]=atoms[(i+1)];
dz=(zplane-atoms[(i+2)]);
atompre[(i+2)]=(dz*dz);
atompre[(i+3)]=atoms[(i+3)];
}
hipMemcpyToSymbol(HIP_SYMBOL(atominfo), atompre, ((count*4)*sizeof (float)), 0);
if (1)
{
;
}
/* check and clear any existing errors */
return 0;
}

