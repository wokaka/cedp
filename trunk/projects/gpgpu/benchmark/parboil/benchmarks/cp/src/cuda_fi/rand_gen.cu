#include "hip/hip_runtime.h"
/* Instrumented by FaultInjector */
#include "gpufi.h"

/*
**************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 **************************************************************************
*/
#include "randomc.h"

#include "gpufi.h"

/*
 
   The following two functions implement the Mersenne Twister random 
   number generator.  The copyright notice/disclaimer, etc are related
   to this code.
 
   Copyright (C) 1997 - 2002, Makoto Matsumoto and Takuji Nishimura,
   All rights reserved.                          
 
   Redistribution and use in source and binary forms, with or without
   modification, are permitted provided that the following conditions
   are met:
 
     1. Redistributions of source code must retain the above copyright
        notice, this list of conditions and the following disclaimer.
 
     2. Redistributions in binary form must reproduce the above copyright
        notice, this list of conditions and the following disclaimer in the
        documentation and/or other materials provided with the distribution.
 
     3. The names of its contributors may not be used to endorse or promote 
        products derived from this software without specific prior written 
        permission.
 
   THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
   "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
   LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
   A PARTICULAR PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
   CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
   EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
   PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
   PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
   LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
   NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
   SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 
 
*/
void RandomInit(uint32 seed)
{
    int i;
    /* re-seed generator
     */
    if ((threadIdx.x==0))
    {
        mt[0]=(seed&0XFFFFFFFF);
        for (i=1; i<624; i ++ )
        {
            mt[i]=((1812433253*(mt[(i-1)]^(mt[(i-1)]>>30)))+i);
        }
    }
    __syncthreads();
}

void BRandom(  )
{
    /* generate 32 random bits
     */
    uint32 y;
    int thdx;
    /* block size is 256
     */
    /* step 1: 0-226, MERS_N-MERS_M=227
     */
    if ((threadIdx.x<(624-397)))
    {
        y=((mt[threadIdx.x]&(MASK<<31))|(mt[(threadIdx.x+1)]&((1<<31)-1)));
        y=((mt[(threadIdx.x+397)]^(y>>1))^((y&1) ? MERS_A : 0));
    }
    __syncthreads();
    if ((threadIdx.x<(624-397)))
    {
        mt[threadIdx.x]=y;
    }
    __syncthreads();
    /* step 2: 227-453
     */
    thdx=(threadIdx.x+(624-397));
    if ((threadIdx.x<(624-397)))
    {
        y=((mt[thdx]&(MASK<<31))|(mt[(thdx+1)]&((1<<31)-1)));
        y=((mt[threadIdx.x]^(y>>1))^((y&1) ? MERS_A : 0));
    }
    __syncthreads();
    if ((threadIdx.x<(624-397)))
    {
        mt[thdx]=y;
    }
    __syncthreads();
    /* step 3: 454-622
     */
    thdx+=(624-397);
    if ((thdx<(624-1)))
    {
        y=((mt[thdx]&(MASK<<31))|(mt[(thdx+1)]&((1<<31)-1)));
        y=((mt[(threadIdx.x+(624-397))]^(y>>1))^((y&1) ? MERS_A : 0));
    }
    __syncthreads();
    if ((thdx<(624-1)))
    {
        mt[thdx]=y;
    }
    __syncthreads();
    /* step 4: 623
     */
    if ((threadIdx.x==0))
    {
        y=((mt[(624-1)]&(MASK<<31))|(mt[0]&((1<<31)-1)));
        mt[(624-1)]=((mt[(397-1)]^(y>>1))^((y&1) ? MERS_A : 0));
    }
    __syncthreads();
    /* Tempering (May be omitted):
     */
    y^=(y>>11);
    y^=((y<<7)&MERS_B);
    y^=((y<<15)&MERS_C);
    y^=(y>>18);
}
