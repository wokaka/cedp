#include "hip/hip_runtime.h"
/* Instrumented by FaultInjector */
#include "gpufi.h"

#include <stdio.h>

#include <string.h>

#include "parboil.h"

#include "shell.h"

#include "crys_kernel.cu"

#include "gpufi.h"

uint4 * d_Block_Work;
uint2 * d_FinalReduce;
float * d_Output, * d_ReductionSum;
struct hipArray * d_Coors, * d_Sprms, * d_Wghts;
float4 * Coors;
float2 * Sprms;
uint4 * Block_Work;
int totNumBlocks;
int MaxBlocks;
struct Atom * ComputeAtom, * BasisAtom;
struct Shell * ComputeShell;
float Coor[64];
float Alpha[64];
float Coeff[64];
float Wghts[2];
float * ReductionSum;
int totReductionElements;
uint2 * FinalReduce;
struct double3
{
    double x, y, z;
};

void AllocateDataOnDevice(int , int , int , int , int , int );
void RunKernel(int , struct pb_TimerSet * , struct pb_Parameters * );
void CalcOnHost(int );
/* & */
struct Atom * ReadBasisAtoms(int * , struct pb_Parameters * );
/* & */
int TotalNumOfShells(char * , int , int * );
void PopulateShells(char * , int );
void PopulateHostData(int , int , int );
int NumOfIntegrals(int );
void DistributeBlockWork(int );
void FreeAllData(struct pb_TimerSet * );
void PopulateWeights(  );
double root1(double X);
int main(int argc, char * argv[])
{
    struct pb_TimerSet timers;
    struct pb_Parameters * params;
    int numBasisAtoms;
    int totNumAtoms;
    int totNumShells;
    int totBasisShells = 0;
    int i, j;
    int numIntegrals;
    int d_output_mem;
    int d_work_mem;
    int reduction_mem;
    int final_mem;
    int d_total_mem;
    pb_InitializeTimerSet(( & timers));
    GPUFI_INIT(2, 86);


    params=pb_ReadParameters(( & argc), argv);
    totNumBlocks=0;
    MaxBlocks=0;
    pb_SwitchToTimer(( & timers), pb_TimerID_IO);
    BasisAtom=ReadBasisAtoms(( & numBasisAtoms), params);
    pb_SwitchToTimer(( & timers), pb_TimerID_COMPUTE);
    totNumShells=TotalNumOfShells(params->inpFiles[0], numBasisAtoms, ( & totNumAtoms));
    totReductionElements=((((totNumShells*(totNumShells+1))*(totNumShells+2))*(totNumShells+3))/24);
    ComputeAtom=((struct Atom * )malloc((totNumAtoms*sizeof (struct Atom))));
    ComputeShell=((struct Shell * )malloc((totNumShells*sizeof (struct Shell))));
    pb_SwitchToTimer(( & timers), pb_TimerID_IO);
    PopulateShells(params->inpFiles[0], numBasisAtoms);
    pb_SwitchToTimer(( & timers), pb_TimerID_COMPUTE);
    /* 	all shells are ready now
     */
    /* 	prepare host data
     */
    for (i=0; i<numBasisAtoms; i ++ )
    {
        for (j=0; j<BasisAtom[i].numShells; j ++ )
        {
            totBasisShells+=BasisAtom[i].AtomShell[j].numPrimitives;
        }
    }
    Coors=((float4 * )malloc((totNumAtoms*sizeof (float4))));
    Sprms=((float2 * )malloc((totBasisShells*sizeof (float2))));
    PopulateHostData(totNumAtoms, totNumShells, numBasisAtoms);
    /* 	distribute the work now
     */
    FinalReduce=((uint2 * )malloc((totReductionElements*sizeof (uint2))));
    numIntegrals=NumOfIntegrals(totNumShells);
    pb_SwitchToTimer(( & timers), pb_TimerID_IO);
    printf("Total # of integrals to compute: %d\n", numIntegrals);
    printf("Total # of blocks allocated: %d\n", totNumBlocks);
    printf("Final array size: %d\n", totReductionElements);
    pb_SwitchToTimer(( & timers), pb_TimerID_COMPUTE);
    Block_Work=((uint4 * )malloc((totNumBlocks*sizeof (uint4))));
    DistributeBlockWork(totNumShells);
    d_output_mem=(totNumBlocks*sizeof (float));
    d_work_mem=(totNumBlocks*sizeof (uint4));
    reduction_mem=(totReductionElements*sizeof (float));
    final_mem=(totReductionElements*sizeof (uint2));
    ReductionSum=((float * )malloc(reduction_mem));
    /* 	prepare device data
     */
    pb_SwitchToTimer(( & timers), pb_TimerID_COPY);
    AllocateDataOnDevice(d_output_mem, d_work_mem, reduction_mem, final_mem, totNumAtoms, totBasisShells);
    pb_SwitchToTimer(( & timers), pb_TimerID_COMPUTE);
    d_total_mem=(((d_output_mem+d_work_mem)+reduction_mem)+final_mem);
    printf("%.2lf MB allocated\n", (((double)d_total_mem)/1048576));
    printf("maxblocks = %d\n", MaxBlocks);
    /* 	okay, now ready to do something useful
     */
    RunKernel(numIntegrals, ( & timers), params);
    /* 	loading data back to the host
     */
    pb_SwitchToTimer(( & timers), pb_TimerID_COPY);
    hipMemcpy(ReductionSum, d_ReductionSum, reduction_mem, hipMemcpyDeviceToHost);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 146);
            exit(( - 1));
        }
    }
    pb_SwitchToTimer(( & timers), pb_TimerID_COMPUTE);
    FreeAllData(( & timers));
    if (params->outFile)
    {
        FILE * file;
        pb_SwitchToTimer(( & timers), pb_TimerID_IO);
        file=fopen(params->outFile, "w");
        printf("totReductionElements %d\n", totReductionElements);
        /* only printing the first 20000
         */
        for (i=0; i<(reduction_mem/sizeof (float)); i ++ )
        {
            fprintf(file, "%d\t%e\n", i, ReductionSum[i]);
        }
        fclose(file);
    }
    pb_SwitchToTimer(( & timers), pb_TimerID_NONE);
    pb_PrintTimerSet(( & timers));
    pb_FreeParameters(params);
    GPUFI_HALT("fi_profile.txt");
    return 0;
    GPUFI_HALT("fi_profile.txt");
}

void RunKernel(int numIntegrals, struct pb_TimerSet * timers, struct pb_Parameters * params)
{
    int runs = ((int)ceil(((1.0*totNumBlocks)/65535)));
    int RemainingBlocks;
    int StartBlock = 0;
    int run;
    int RemainReduction;
    int Offset = 0;
    printf("%d computation cycles will be performed...\n", runs);
    RemainingBlocks=totNumBlocks;
    for (run=0; run<runs; run ++ )
    {
        int numBlocks = min(65535, RemainingBlocks);
        struct dim3 grid;
        struct dim3 block;
        grid.x=numBlocks;
        grid.y=1;
        grid.z=1;
        block.x=64;
        block.y=1;
        block.z=1;
        pb_SwitchToTimer(timers, pb_TimerID_GPU);
        /* Fault Injection Instrumentation Begin */
        hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
        CUDA_ERRCK
        hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

        ComputeX<<<grid, block>>> (d_Block_Work, d_Output, StartBlock, gpufi_dev);
        /* Fault Injection Instrumentation Begin */
        hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
        CUDA_ERRCK
        hipFree(gpufi_dev);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

        {
            hipError_t err;
            if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 192);
                exit(( - 1));
            }
        }
        if (params->synchronizeGpu)
        {
            hipDeviceSynchronize();
        }
        pb_SwitchToTimer(timers, pb_TimerID_COMPUTE);
        RemainingBlocks-=65535;
        StartBlock+=numBlocks;
    }
    runs=((int)ceil(((1.0*totReductionElements)/65535)));
    printf("done.\n\n%d reduction cycles will be performed...\n", runs);
    RemainReduction=totReductionElements;
    for (run=0; run<runs; run ++ )
    {
        int numBlocks = min(65535, RemainReduction);
        struct dim3 grid;
        struct dim3 block;
        grid.x=numBlocks;
        grid.y=1;
        grid.z=1;
        block.x=MaxBlocks;
        block.y=1;
        block.z=1;
        pb_SwitchToTimer(timers, pb_TimerID_GPU);
        /* Fault Injection Instrumentation Begin */
        hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
        CUDA_ERRCK
        hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

        DoReduction<<<grid, block>>> (d_ReductionSum, d_Output, d_FinalReduce, MaxBlocks, Offset, gpufi_dev);
        /* Fault Injection Instrumentation Begin */
        hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
        CUDA_ERRCK
        hipFree(gpufi_dev);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

        {
            hipError_t err;
            if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 219);
                exit(( - 1));
            }
        }
        if (params->synchronizeGpu)
        {
            hipDeviceSynchronize();
        }
        pb_SwitchToTimer(timers, pb_TimerID_COMPUTE);
        RemainReduction-=65535;
        Offset+=numBlocks;
    }
}

void AllocateDataOnDevice(int d_output_mem, int d_work_mem, int reduction_mem, int final_mem, int numCoors, int numSprms)
{
    hipMalloc(((void *  * )( & d_ReductionSum)), reduction_mem);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 233);
            exit(( - 1));
        }
    }
    hipMalloc(((void *  * )( & d_Output)), d_output_mem);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 235);
            exit(( - 1));
        }
    }
    hipMalloc(((void *  * )( & d_Block_Work)), d_work_mem);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 237);
            exit(( - 1));
        }
    }
    hipMalloc(((void *  * )( & d_FinalReduce)), final_mem);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 239);
            exit(( - 1));
        }
    }
    hipMallocArray(( & d_Coors), ( & texCoors.channelDesc), numCoors, 1);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 242);
            exit(( - 1));
        }
    }
    hipMallocArray(( & d_Sprms), ( & texSprms.channelDesc), numSprms, 1);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 245);
            exit(( - 1));
        }
    }
    hipMallocArray(( & d_Wghts), ( & texWghts.channelDesc), (1<<2), 2);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 248);
            exit(( - 1));
        }
    }
    hipMemcpy(d_Block_Work, Block_Work, d_work_mem, hipMemcpyHostToDevice);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 252);
            exit(( - 1));
        }
    }
    hipMemcpy(d_FinalReduce, FinalReduce, final_mem, hipMemcpyHostToDevice);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 255);
            exit(( - 1));
        }
    }
    hipMemcpyToArray(d_Coors, 0, 0, ((void * )Coors), (numCoors*sizeof (float4)), hipMemcpyHostToDevice);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 260);
            exit(( - 1));
        }
    }
    hipMemcpyToArray(d_Sprms, 0, 0, ((void * )Sprms), (numSprms*sizeof (float2)), hipMemcpyHostToDevice);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 264);
            exit(( - 1));
        }
    }
    hipMemcpyToArray(d_Wghts, 0, 0, ((void * )Wghts), (2*sizeof (float)), hipMemcpyHostToDevice);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 268);
            exit(( - 1));
        }
    }
    texWghts.filterMode=hipFilterModeLinear;
    hipBindTextureToArray(texCoors, d_Coors, texCoors.channelDesc);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 273);
            exit(( - 1));
        }
    }
    hipBindTextureToArray(texSprms, d_Sprms, texSprms.channelDesc);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 275);
            exit(( - 1));
        }
    }
    hipBindTextureToArray(texWghts, d_Wghts, texWghts.channelDesc);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 277);
            exit(( - 1));
        }
    }
}

void FreeAllData(struct pb_TimerSet * timers)
{
    pb_SwitchToTimer(timers, pb_TimerID_COPY);
    hipFree(((void * )d_FinalReduce));
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 284);
            exit(( - 1));
        }
    }
    hipFree(((void * )d_Block_Work));
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 286);
            exit(( - 1));
        }
    }
    hipFree(((void * )d_Output));
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 288);
            exit(( - 1));
        }
    }
    hipFree(((void * )d_ReductionSum));
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 290);
            exit(( - 1));
        }
    }
    hipFreeArray(d_Coors);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 292);
            exit(( - 1));
        }
    }
    hipFreeArray(d_Wghts);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 294);
            exit(( - 1));
        }
    }
    hipFreeArray(d_Sprms);
    {
        hipError_t err;
        if (((err=hipGetLastError())!=hipSuccess))
        {
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), 296);
            exit(( - 1));
        }
    }
    pb_SwitchToTimer(timers, pb_TimerID_COMPUTE);
    free(((void * )Block_Work));
    free(((void * )FinalReduce));
    free(((void * )ComputeAtom));
    free(((void * )BasisAtom));
    free(((void * )ComputeShell));
}

/* & */
struct Atom * ReadBasisAtoms(int * numBasisAtoms, struct pb_Parameters * params)
{
    FILE * basis = fopen(params->inpFiles[1], "r");
    int numAtoms = 0, numShells = 0;
    struct Atom * BasisAtom;
    int atom, shell;
    if (( ! basis))
    {
        printf("Unable to open file %s\n", params->inpFiles[1]);
        exit(0);
    }
    fscanf(basis, "%*s %*s %d", ( & numAtoms));
    fscanf(basis, "%*s %*s %d", ( & numShells));
    printf("\n>>>>>>> STARTED BASIS SET OUTPUT <<<<<<<\n");
    printf("\n# OF KNOWN ATOMS:  %d\n", numAtoms);
    printf("# OF KNOWN SHELLS: %d\n\n", numShells);
    ( * numBasisAtoms)=numAtoms;
    BasisAtom=((struct Atom * )malloc((numAtoms*sizeof (struct Atom))));
    for (atom=0; atom<numAtoms; atom ++ )
    {
        char type[4];
        char buff[4];
        fscanf(basis, "%*s %s", type);
        fscanf(basis, "%*s %d", ( & numShells));
        BasisAtom[atom].numShells=numShells;
        strcpy(BasisAtom[atom].Type, type);
        printf("\nAtom %s (%d shells)\n", BasisAtom[atom].Type, BasisAtom[atom].numShells);
        for (shell=0; shell<numShells; shell ++ )
        {
            int numPrimitives = 0;
            int prim;
            fscanf(basis, "%*s %*d %*s %d", ( & numPrimitives));
            BasisAtom[atom].AtomShell[shell].numPrimitives=numPrimitives;
            sprintf(buff, "%d", (shell+1));
            strcpy(BasisAtom[atom].AtomShell[shell].Type, type);
            strcpy((BasisAtom[atom].AtomShell[shell].Type+1), buff);
            printf("\tShell %s: %d primitives\n", BasisAtom[atom].AtomShell[shell].Type, BasisAtom[atom].AtomShell[shell].numPrimitives);
            for (prim=0; prim<numPrimitives; prim ++ )
            {
                fscanf(basis, "%*s %*s %*s %f %f", ( & BasisAtom[atom].AtomShell[shell].Alpha[prim]), ( & BasisAtom[atom].AtomShell[shell].Coeff[prim]));
                printf("\t\tprimitive %d: %10.2f    %5.2f\n", (prim+1), BasisAtom[atom].AtomShell[shell].Alpha[prim], BasisAtom[atom].AtomShell[shell].Coeff[prim]);
            }
            printf("\n");
        }
    }
    printf(">>>>>>>> DONE BASIS SET OUTPUT <<<<<<<<\n\n\n");
    fclose(basis);
    return BasisAtom;
}

/* & */
int TotalNumOfShells(char * fname, int numBasisAtoms, int * totNumAtoms)
{
    FILE * inp = fopen(fname, "r");
    int numShells = 0;
    int atom, batom;
    if (( ! inp))
    {
        printf("Unable to open %s\n", fname);
        exit(0);
    }
    fscanf(inp, "%*s %d", totNumAtoms);
    for (atom=0; atom<( * totNumAtoms); atom ++ )
    {
        char type[8];
        int notfound = 1;
        fscanf(inp, "%s %*s %*s %*s", type);
        for (batom=0; batom<numBasisAtoms; batom ++ )
        {
            if (( ! strcmp(BasisAtom[batom].Type, type)))
            {
                numShells+=BasisAtom[batom].numShells;
                notfound=0;
                break;
            }
        }
        if (notfound)
        {
            printf("Unable to find atom \'%s\' in the basis set\n", type);
            exit(0);
        }
    }
    fclose(inp);
    return numShells;
}

void PopulateShells(char * fname, int numBasisAtoms)
{
    FILE * inp = fopen(fname, "r");
    int numAtoms = 0, currentShell = 0;
    int atom, batom, shell, prim;
    if (( ! inp))
    {
        printf("Unable to open %s\n", fname);
        exit(0);
    }
    fscanf(inp, "%*s %d", ( & numAtoms));
    for (atom=0; atom<numAtoms; atom ++ )
    {
        int currentInList = 0;
        fscanf(inp, "%s %f %f %f", ( & ComputeAtom[atom].Type), ( & ComputeAtom[atom].X), ( & ComputeAtom[atom].Y), ( & ComputeAtom[atom].Z));
        for (batom=0; batom<numBasisAtoms; batom ++ )
        {
            /* 	this part populates inList
             */
            if (( ! strcmp(BasisAtom[batom].Type, ComputeAtom[atom].Type)))
            {
                for (shell=0; shell<BasisAtom[batom].numShells; shell ++ )
                {
                    ComputeShell[currentShell]=BasisAtom[batom].AtomShell[shell];
                    ComputeShell[currentShell].myAtom=atom;
                    /* 	this part populates inList
                     */
                    for (prim=0; prim<BasisAtom[batom].AtomShell[shell].numPrimitives; prim ++ )
                    {
                        ComputeShell[currentShell].inList[prim]=(currentInList ++ );
                    }
                    currentShell ++ ;
                }
                break;
            }
            else
            {
                for (shell=0; shell<BasisAtom[batom].numShells; shell ++ )
                {
                    currentInList+=BasisAtom[batom].AtomShell[shell].numPrimitives;
                }
            }
        }
    }
    fclose(inp);
}

void PopulateHostData(int totNumAtoms, int totNumShells, int numBasisAtoms)
{
    int atom;
    int currentPos = 0, batom, shell, prim;
    PopulateWeights();
    for (atom=0; atom<totNumAtoms; atom ++ )
    {
        Coors[atom].x=ComputeAtom[atom].X;
        Coors[atom].y=ComputeAtom[atom].Y;
        Coors[atom].z=ComputeAtom[atom].Z;
    }
    for (batom=0; batom<numBasisAtoms; batom ++ )
    {
        for (shell=0; shell<BasisAtom[batom].numShells; shell ++ )
        {
            for (prim=0; prim<BasisAtom[batom].AtomShell[shell].numPrimitives; prim ++ )
            {
                Sprms[currentPos].x=BasisAtom[batom].AtomShell[shell].Alpha[prim];
                Sprms[currentPos].y=BasisAtom[batom].AtomShell[shell].Coeff[prim];
                currentPos ++ ;
            }
        }
    }
}

int NumOfIntegrals(int totNumShells)
{
    int numIntegrals = 0;
    int firstRedElement = 0;
    int redElement = 0;
    int shell1, shell2, shell3, shell4;
    for (shell1=0; shell1<totNumShells; shell1 ++ )
    {
        for (shell2=shell1; shell2<totNumShells; shell2 ++ )
        {
            for (shell3=shell2; shell3<totNumShells; shell3 ++ )
            {
                for (shell4=shell3; shell4<totNumShells; shell4 ++ )
                {
                    int integrals = (((ComputeShell[shell1].numPrimitives*ComputeShell[shell2].numPrimitives)*ComputeShell[shell3].numPrimitives)*ComputeShell[shell4].numPrimitives);
                    int blocks;
                    numIntegrals+=integrals;
                    blocks=((int)ceil(((1.0*integrals)/64)));
                    totNumBlocks+=blocks;
                    if ((blocks>MaxBlocks))
                    {
                        MaxBlocks=blocks;
                    }
                    FinalReduce[redElement].x=firstRedElement;
                    FinalReduce[redElement].y=blocks;
                    firstRedElement+=blocks;
                    redElement ++ ;
                }
            }
        }
    }
    return numIntegrals;
}

void DistributeBlockWork(int totNumShells)
{
    int numElements = 0;
    int StartBlock = 0;
    int block;
    int shell1, shell2, shell3, shell4, basic;
    for (shell1=0; shell1<totNumShells; shell1 ++ )
    {
        for (shell2=shell1; shell2<totNumShells; shell2 ++ )
        {
            for (shell3=shell2; shell3<totNumShells; shell3 ++ )
            {
                for (shell4=shell3; shell4<totNumShells; shell4 ++ )
                {
                    int integrals = (((ComputeShell[shell1].numPrimitives*ComputeShell[shell2].numPrimitives)*ComputeShell[shell3].numPrimitives)*ComputeShell[shell4].numPrimitives);
                    int blocks = ((int)ceil(((1.0*integrals)/64)));
                    StartBlock=numElements;
                    for (block=0; block<blocks; block ++ )
                    {
                        int a4 = ComputeShell[shell4].numPrimitives;
                        int a3 = ComputeShell[shell3].numPrimitives;
                        int a2 = ComputeShell[shell2].numPrimitives;
                        int a1 = ComputeShell[shell1].numPrimitives;
                        int offset4 = ComputeShell[shell4].inList[0];
                        int offset3 = ComputeShell[shell3].inList[0];
                        int offset2 = ComputeShell[shell2].inList[0];
                        int offset1 = ComputeShell[shell1].inList[0];
                        Block_Work[numElements].y=((((ComputeShell[shell1].myAtom<<24)|(ComputeShell[shell2].myAtom<<16))|(ComputeShell[shell3].myAtom<<8))|ComputeShell[shell4].myAtom);
                        Block_Work[numElements].z=((((offset1<<24)|(offset2<<16))|(offset3<<8))|offset4);
                        Block_Work[numElements].x=((((a1<<12)|(a2<<8))|(a3<<4))|a4);
                        Block_Work[numElements].w=StartBlock;
                        numElements ++ ;
                    }
                }
            }
        }
    }
}

double root1(double X)
{
    double PIE4;
    double WW1 = 0.0;
    double F1, E, Y, inv;
    PIE4=0.785398163397448;
    if ((X<3.0E-7))
    {
        WW1=(1.0-(0.333333333*X));
    }
    else
    {
        if ((X<1.0))
        {
            F1=((((((((((((((((((( - 8.36313918003957E-8)*X)+1.21222603512827E-6)*X)-1.15662609053481E-5)*X)+9.25197374512647E-5)*X)-6.40994113129432E-4)*X)+0.00378787044215009)*X)-0.0185185172458485)*X)+0.0714285713298222)*X)-0.199999999997023)*X)+0.333333333333318);
            WW1=(((X+X)*F1)+exp(( - X)));
        }
        else
        {
            if ((X<3.0))
            {
                Y=(X-2.0);
                F1=((((((((((((((((((((((( - 1.61702782425558E-10)*Y)+1.96215250865776E-9)*Y)-2.14234468198419E-8)*Y)+2.17216556336318E-7)*Y)-1.98850171329371E-6)*Y)+1.62429321438911E-5)*Y)-1.16740298039895E-4)*Y)+7.24888732052332E-4)*Y)-0.00379490003707156)*Y)+0.0161723488664661)*Y)-0.0529428148329736)*Y)+0.115702180856167);
                WW1=(((X+X)*F1)+exp(( - X)));
            }
            else
            {
                if ((X<5.0))
                {
                    Y=(X-4.0);
                    F1=((((((((((((((((((((((( - 2.62453564772299E-11)*Y)+3.24031041623823E-10)*Y)-3.614965656163E-9)*Y)+3.760256799971E-8)*Y)-3.553558319675E-7)*Y)+3.022556449731E-6)*Y)-2.290098979647E-5)*Y)+1.526537461148E-4)*Y)-8.81947375894379E-4)*Y)+0.00433207949514611)*Y)-0.0175257821619926)*Y)+0.0528406320615584);
                    WW1=(((X+X)*F1)+exp(( - X)));
                }
                else
                {
                    if ((X<10.0))
                    {
                        E=exp(( - X));
                        inv=(1/X);
                        WW1=((((((((((((((0.46897511375022*inv)-0.69955602298985)*inv)+0.53689283271887)*inv)-0.32883030418398)*inv)+0.24645596956002)*inv)-0.49984072848436)*inv)-3.1501078774085E-6)*E)+sqrt((PIE4*inv)));
                    }
                    else
                    {
                        if ((X<15.0))
                        {
                            E=exp(( - X));
                            inv=(1/X);
                            WW1=((((((((( - 0.18784686463512)*inv)+0.22991849164985)*inv)-0.49893752514047)*inv)-2.1916512131607E-5)*E)+sqrt((PIE4*inv)));
                        }
                        else
                        {
                            if ((X<33.0))
                            {
                                E=exp(( - X));
                                inv=(1/X);
                                WW1=((((((0.1962326414943*inv)-0.4969524146449)*inv)-6.0156581186481E-5)*E)+sqrt((PIE4*inv)));
                            }
                            else
                            {
                                inv=(1/X);
                                WW1=sqrt((PIE4*inv));
                            }
                        }
                    }
                }
            }
        }
    }
    return WW1;
}

void PopulateWeights(  )
{
    int i;
    for (i=0; i<2; i ++ )
    {
        float X = ((float)(((1.0*i)*10)/(2-1)));
        Wghts[i]=root1(X);
    }
}
