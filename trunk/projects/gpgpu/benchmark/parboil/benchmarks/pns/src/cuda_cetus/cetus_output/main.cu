/* Automatically inserted by Cetus_Cuda */
#include "gpufi.h"
/* End of Cetus_Cuda code */

/*
**************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 **************************************************************************
*/
#include <stdlib.h> 
#include <stdio.h> 
#include <string.h> 
#include <math.h> 
#include "parboil.h" 
#include "rand_gen.cu" 
#include "petri_kernel.cu" 
#include "gpufi.h" 
static int N, s, t, N2, NSQUARE2;
uint32 host_mt[624];
void * AllocateDeviceMemory(int size);
void CopyFromDeviceMemory(void * h_p, void * d_p, int size);
void CopyFromHostMemory(void * d_p, void * h_p, int size);
void FreeDeviceMemory(void * mem);
void PetrinetOnDevice(struct pb_TimerSet * timers, struct pb_Parameters * params);
void compute_statistics(  );
float results[4];
float * h_vars;
int * h_maxs;
int main(int argc, char *  * argv)
{
struct pb_TimerSet timers;
struct pb_Parameters * params;
/*
Automatically inserted by Cetus_Cuda
*/
GPUFI_FETCH_COMMAND();
/*
End of Cetus_Cuda code
*/
pb_InitializeTimerSet(( & timers));
params=pb_ReadParameters(( & argc), argv);
if ((argc<4))
{
printf("Usage: petri n s t\n""n: the place-transition grid is 2nX2n\n""s: the maximum steps in a trajectory\n""t: number of trajectories\n");
/*
Automatically inserted by Cetus_Cuda
*/
GPUFI_END("fi_profile.txt");
/*
End of Cetus_Cuda code
*/
return ( - 1);
}
pb_SwitchToTimer(( & timers), pb_TimerID_COMPUTE);
N=atoi(argv[1]);
if ((N<1))
{
/*
Automatically inserted by Cetus_Cuda
*/
GPUFI_END("fi_profile.txt");
/*
End of Cetus_Cuda code
*/
return ( - 1);
}
s=atoi(argv[2]);
if ((s<1))
{
/*
Automatically inserted by Cetus_Cuda
*/
GPUFI_END("fi_profile.txt");
/*
End of Cetus_Cuda code
*/
return ( - 1);
}
t=atoi(argv[3]);
if ((t<1))
{
/*
Automatically inserted by Cetus_Cuda
*/
GPUFI_END("fi_profile.txt");
/*
End of Cetus_Cuda code
*/
return ( - 1);
}
N2=(N+N);
NSQUARE2=(N*N2);
h_vars=((float * )malloc((t*sizeof (float))));
h_maxs=((int * )malloc((t*sizeof (int))));
/* compute the simulation on the GPU */
PetrinetOnDevice(( & timers), params);
pb_SwitchToTimer(( & timers), pb_TimerID_COMPUTE);
compute_statistics();
free(h_vars);
free(h_maxs);
pb_SwitchToTimer(( & timers), pb_TimerID_IO);
printf("petri N=%d s=%d t=%d\n", N, s, t);
printf("mean_vars: %f    var_vars: %f\n", results[0], results[1]);
printf("mean_maxs: %f    var_maxs: %f\n", results[2], results[3]);
if (params->outFile)
{
FILE * out;
out=fopen(params->outFile, "w");
fprintf(out, "%f %f %f %f\n", results[0], results[1], results[2], results[3]);
fclose(out);
}
pb_SwitchToTimer(( & timers), pb_TimerID_NONE);
pb_PrintTimerSet(( & timers));
pb_FreeParameters(params);
/*
Automatically inserted by Cetus_Cuda
*/
GPUFI_END("fi_profile.txt");
/*
End of Cetus_Cuda code
*/
return 0;
}

void compute_statistics(  )
{
float sum = 0;
float sum_vars = 0;
float sum_max = 0;
float sum_max_vars = 0;
int i;
for (i=0; i<t; i ++ )
{
sum+=h_vars[i];
sum_vars+=(h_vars[i]*h_vars[i]);
sum_max+=h_maxs[i];
sum_max_vars+=(h_maxs[i]*h_maxs[i]);
}
results[0]=(sum/t);
results[1]=((sum_vars/t)-(results[0]*results[0]));
results[2]=(sum_max/t);
results[3]=((sum_max_vars/t)-(results[2]*results[2]));
}

void PetrinetOnDevice(struct pb_TimerSet * timers, struct pb_Parameters * params)
{
/* Allocate memory */
int i;
int unit_size = (((NSQUARE2*(sizeof (int)+sizeof (char)))+sizeof (float))+sizeof (int));
int block_num = (750000000/unit_size);
int * p_hmaxs;
float * p_hvars;
int * g_places;
float * g_vars;
int * g_maxs;
dim3 grid;
dim3 threads;
dim3 grid1;
pb_SwitchToTimer(timers, pb_TimerID_COPY);
g_places=((int * )AllocateDeviceMemory((((unit_size-sizeof (float))-sizeof (int))*block_num)));
{
hipError_t err = hipGetLastError();
if (err)
{
fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
}
}
g_vars=((float * )AllocateDeviceMemory((block_num*sizeof (float))));
{
hipError_t err = hipGetLastError();
if (err)
{
fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
}
}
g_maxs=((int * )AllocateDeviceMemory((block_num*sizeof (int))));
{
hipError_t err = hipGetLastError();
if (err)
{
fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
}
}
/* Setup the execution configuration */
grid.x=block_num;
/* number of blocks */
grid.y=1;
grid.z=1;
grid.x=256;
/* each block has 256 threads */
grid.y=1;
grid.z=1;
p_hmaxs=h_maxs;
p_hvars=h_vars;
/* Launch the device computation threads! */
for (i=0; i<(t-block_num); i+=block_num)
{
if (params->synchronizeGpu)
{
hipDeviceSynchronize();
}
pb_SwitchToTimer(timers, pb_TimerID_GPU);
/* Automatically inserted by Cetus_Cuda */
hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
CUDA_ERRCK
hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
CUDA_ERRCK
/* End of Cetus_Cuda code */

PetrinetKernel<<<grid, threads>>> (g_places, g_vars, g_maxs, N, s, (5489*(i+1)), gpufi_dev);
/* Automatically inserted by Cetus_Cuda */
hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
CUDA_ERRCK
hipFree(gpufi_dev);
CUDA_ERRCK
/* End of Cetus_Cuda code */

{
hipError_t err = hipGetLastError();
if (err)
{
fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
}
}
if (params->synchronizeGpu)
{
hipDeviceSynchronize();
}
pb_SwitchToTimer(timers, pb_TimerID_COPY);
CopyFromDeviceMemory(p_hmaxs, g_maxs, (block_num*sizeof (int)));
{
hipError_t err = hipGetLastError();
if (err)
{
fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
}
}
CopyFromDeviceMemory(p_hvars, g_vars, (block_num*sizeof (float)));
{
hipError_t err = hipGetLastError();
if (err)
{
fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
}
}
if (params->synchronizeGpu)
{
hipDeviceSynchronize();
}
pb_SwitchToTimer(timers, pb_TimerID_COMPUTE);
p_hmaxs+=block_num;
p_hvars+=block_num;
}
grid1.x=(t-i);
grid1.y=1;
grid1.z=1;
pb_SwitchToTimer(timers, pb_TimerID_GPU);
/* Automatically inserted by Cetus_Cuda */
hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
CUDA_ERRCK
hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
CUDA_ERRCK
/* End of Cetus_Cuda code */

PetrinetKernel<<<grid1, threads>>> (g_places, g_vars, g_maxs, N, s, time(NULL), gpufi_dev);
/* Automatically inserted by Cetus_Cuda */
hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
CUDA_ERRCK
hipFree(gpufi_dev);
CUDA_ERRCK
/* End of Cetus_Cuda code */

{
hipError_t err = hipGetLastError();
if (err)
{
fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
}
}
/* Read result from the device */
if (params->synchronizeGpu)
{
hipDeviceSynchronize();
}
pb_SwitchToTimer(timers, pb_TimerID_COPY);
CopyFromDeviceMemory(p_hmaxs, g_maxs, ((t-i)*sizeof (int)));
{
hipError_t err = hipGetLastError();
if (err)
{
fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
}
}
CopyFromDeviceMemory(p_hvars, g_vars, ((t-i)*sizeof (float)));
{
hipError_t err = hipGetLastError();
if (err)
{
fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
}
}
/* Free device matrices */
FreeDeviceMemory(g_places);
{
hipError_t err = hipGetLastError();
if (err)
{
fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
}
}
FreeDeviceMemory(g_vars);
{
hipError_t err = hipGetLastError();
if (err)
{
fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
}
}
FreeDeviceMemory(g_maxs);
{
hipError_t err = hipGetLastError();
if (err)
{
fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
}
}
}

/* Allocate a device matrix of same size as M. */
void * AllocateDeviceMemory(int size)
{
int * mem;
hipMalloc(((void *  * )( & mem)), size);
return mem;
}

/* Copy device memory to host memory */
void CopyFromDeviceMemory(void * h_p, void * d_p, int size)
{
hipMemcpy(h_p, d_p, size, hipMemcpyDeviceToHost);
}

/* Copy device memory from host memory */
void CopyFromHostMemory(void * d_p, void * h_p, int size)
{
hipMemcpy(d_p, h_p, size, hipMemcpyHostToDevice);
}

/* Free a device matrix. */
void FreeDeviceMemory(void * mem)
{
if ((mem!=NULL))
{
hipFree(mem);
}
}

