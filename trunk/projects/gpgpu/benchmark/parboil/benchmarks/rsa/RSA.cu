
#include <stdio.h>
#include "string.h"
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include "time.h"
#include <cutil.h>
#include <sys/time.h>
#include <malloc.h>
#include <RSA_kernel.cu>



using namespace std;

unsigned int MM(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Mpi, /*unsigned int *Mj, unsigned int *Mi,*/ unsigned int *MiInverse, unsigned int (*Mij)[MODULI_SET_SIZE], /*unsigned int *Mpj,*/ unsigned int *Mpjr, unsigned int *MpjInverse, unsigned int (*Mpji)[MODULI_SET_SIZE], unsigned int *MpInvR, unsigned int *MInv, unsigned int *Result);
//unsigned int MMlast(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Mpi, unsigned int *Mj, unsigned int *Mi, unsigned int *Mir, unsigned int *MiInverse, unsigned int (*Mij)[MODULI_SET_SIZE], unsigned int *MInvR, unsigned int *Mpj, unsigned int *Mpjr, unsigned int *MpjInverse, unsigned int (*Mpji)[MODULI_SET_SIZE], unsigned int *MpInvR, unsigned int *MInv, unsigned int *Result);
unsigned int MMlast(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Mpi, unsigned int *Mj, /*unsigned int *Mi, unsigned int *Mir,*/ unsigned int *MiInverse, unsigned int (*Mij)[MODULI_SET_SIZE], /*unsigned int *MInvR, unsigned int *Mpj,*/ unsigned int *Mpjr, unsigned int *MpjInverse, unsigned int (*Mpji)[MODULI_SET_SIZE], unsigned int *MpInvR, unsigned int *MInv, /*unsigned int (*MI)[MODULI_SET_SIZE],*/ unsigned int *Result);
unsigned int FirstStep(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Q);
unsigned int ThirdStep(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Qp, unsigned int *Rp, unsigned int *MInv);
unsigned int ApproxExt( unsigned int *Q, unsigned int *Qp, unsigned int *modulus, /*unsigned int *Mi,*/ unsigned int *MiInverse, unsigned int (*Mij)[MODULI_SET_SIZE]);
unsigned int ExactExt( unsigned int *Rp, unsigned int *R, unsigned int *modulus, unsigned int *Mpi, /*unsigned int *Mpj,*/ unsigned int *Mpjr, unsigned int *MpjInverse, unsigned int (*Mpji)[MODULI_SET_SIZE], unsigned int *MpInvR);
unsigned int ExactExtLast( unsigned int *Q, unsigned int *Qp, unsigned int *modulus, unsigned int *Mj, /*unsigned int *Mi, unsigned int *Mir,*/ unsigned int *MiInverse, unsigned int (*Mij)[MODULI_SET_SIZE]/*, unsigned int *MInvR*/);
unsigned int MRS( unsigned int *Q, unsigned int *Qp, unsigned int *modulus, unsigned int (*MI)[MODULI_SET_SIZE]);
unsigned int ModularInverse(unsigned int N, unsigned int D);
//unsigned int Exp(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *MsqModN, unsigned int *modulus, unsigned int *Mpi, unsigned int *Mj, unsigned int *Mi, unsigned int *Mir, unsigned int *MiInverse, unsigned int (*Mij)[MODULI_SET_SIZE], unsigned int *MInvR, unsigned int *Mpj, unsigned int *Mpjr, unsigned int *MpjInverse, unsigned int (*Mpji)[MODULI_SET_SIZE], unsigned int *MpInvR, unsigned int *MInv, unsigned int *Result);
unsigned int Exp(unsigned int *A, /*unsigned int *B,*/ unsigned int *N, unsigned int *MsqModN, unsigned int *modulus, unsigned int *Mpi, unsigned int *Mj, /*unsigned int *Mi, unsigned int *Mir,*/ unsigned int *MiInverse, unsigned int (*Mij)[MODULI_SET_SIZE], /*unsigned int *MInvR, unsigned int *Mpj,*/ unsigned int *Mpjr, unsigned int *MpjInverse, unsigned int (*Mpji)[MODULI_SET_SIZE], unsigned int *MpInvR, unsigned int *MInv, /*unsigned int (*MI)[MODULI_SET_SIZE],*/ unsigned int *Result);
unsigned int gcd(unsigned int A, unsigned int B, long long *X, long long *Y);


int main()
{
	//unsigned int in_modulus[] = {3, 7, 13, 19, 29, 8, 5, 11, 17, 23, 31}; // Modulii used for conversion to RNS
	//unsigned int M = 150423, MP = 666655; // Constants specific to the above modulus set
	unsigned int in_A[NUMBER_OF_MESSAGES][MODULI_SET_SIZE * 2 + 1], in_B[MODULI_SET_SIZE * 2 + 1], in_N[MODULI_SET_SIZE * 2 + 1];
	unsigned int /*Mi[MODULI_SET_SIZE],*/ MiInverse[MODULI_SET_SIZE], Mij[MODULI_SET_SIZE+1][MODULI_SET_SIZE]; //Variables for precomutation of approximate base conversion
	unsigned int /*Mpj[MODULI_SET_SIZE],*/ Mpjr[MODULI_SET_SIZE], MpjInverse[MODULI_SET_SIZE], Mpji[MODULI_SET_SIZE][MODULI_SET_SIZE], MpInvR, Mpi[MODULI_SET_SIZE]; //Variables for precomutation of exact base conversion
	unsigned int /*Mir[MODULI_SET_SIZE], MInvR,*/ Mj[MODULI_SET_SIZE+1]; // Variables to perform exact base conversion for the first base conversion in the final Montgomery pass
	unsigned int MInv[MODULI_SET_SIZE+1]; //Variables for precomutation of third step
	//unsigned int MsquareN = 12580; // Precomputed result needed for input to final pass of the Montgomery multiplication
	unsigned int MsqModN[MODULI_SET_SIZE*2+1]; // Variable for input to the final pass of the Montgomery multiplication
	//unsigned int MI[MODULI_SET_SIZE][MODULI_SET_SIZE]; // Variable for MRS base conversion

	//unsigned int Result[MODULI_SET_SIZE*2+1]; // Results (on both bases) from a Montgomery pass
	unsigned int Final_Result[NUMBER_OF_MESSAGES][MODULI_SET_SIZE*2+1]; // Results (on both bases) from the final Montgomery pass
	unsigned int Device_Final_Result[NUMBER_OF_MESSAGES][MODULI_SET_SIZE*2+1]; // Results (on both bases) from the final Montgomery pass

	int i, j;
	//Timer
	float cpu_time = 0;
	float gpu_time = 0;
	unsigned int timer;
	CUT_SAFE_CALL(cutCreateTimer(&timer));



	printf("Starting .....");


	/////////Read RNS look-up table from file////////////////////

	unsigned int input[MODULI_SET_SIZE * 2 + 1][256];
			FILE *fp=fopen("RNS.txt", "r");
			if (fp)
			{
				while(feof(fp)== 0)
				{
					for (i=0; i <= (MODULI_SET_SIZE*2); i++ )
					{
						for(j=0; j<=255; j++)
						{
							fscanf(fp,"%u",&input[i][j]);
						}
					}
				}
			printf("\nLook-up table read complete.");
			fclose(fp);
			}
			else
		printf("\nFile Read Error!\n");


	/////////Read modulus set from file////////////////////

	unsigned int input1[MODULI_SET_SIZE * 2 + 1];
		FILE *fp1=fopen("ModFile.txt", "r");
		if (fp1)
		{
			while(feof(fp1)== 0)
			{
				for (i=0; i <= (MODULI_SET_SIZE*2); i++)
				{
					fscanf(fp1,"%u",&input1[i]);
				}
			}
		printf("\nModulus file read complete.");
		fclose(fp1);
		}
		else
			printf("\nFile Read Error!\n");



	/////////////Read 1024-bit input numbers from file/////////////////////////////

	// Read A from file
		FILE *aFile;
		char str[4];
		unsigned int sub_hex;
			unsigned int buffer_A[NUMBER_OF_MESSAGES][256], a_counter[NUMBER_OF_MESSAGES];
		i = 0;
		j = 0;
		aFile = fopen ("AFile.txt","r");
		if (aFile)
		{
			while((feof(aFile)== 0)&&(j<=NUMBER_OF_MESSAGES-1))
			{
				while((str[0] = getc(aFile)) != '\n')
				{
					sscanf(str,"%x", &sub_hex);
					buffer_A[j][i] = sub_hex;
					i++;
				}

				a_counter[j] = i-1;
				i=0;
				j++;
			}
		printf("\nA read complete.");
		fclose(aFile);
		}
		else
		printf("\nFile Read Error!\n");


	// Read B from file
	FILE *bFile;
	unsigned int buffer_B[256], b_counter;
	i = 0;
	bFile = fopen ("BFile.txt","r");
	if (bFile)
	{
		while ((str[0] = getc(bFile)) != EOF)
		{
			sscanf(str,"%x", &sub_hex);
			buffer_B[i] = sub_hex;
			i++;
			}

	}
	else
		printf("\nFile Read Error!\n");

	b_counter = i-1;
	printf("\nB read complete.");
	fclose(bFile);

	// Read N from file
	FILE *nFile;
	unsigned int buffer_N[256], n_counter;
	i = 0;
	nFile = fopen ("NFile.txt","r");
	if (nFile)
	{
		while ((str[0] = getc(nFile)) != EOF)
		{
			sscanf(str,"%x", &sub_hex);
			buffer_N[i] = sub_hex;
			i++;
			}

	}
	else
		printf("\nFile Read Error!\n");

	n_counter = i-1;
	printf("\nN read complete.");
	fclose(nFile);



	///////////Convert inputs to RNS////////////////////////

	// Initialize inputs

for (i = 0; i <= MODULI_SET_SIZE*2; i++)
	{
		//in_A[i] = 0;
		in_B[i] = 0;
		in_N[i] = 0;
	}

	for (i = 0; i < NUMBER_OF_MESSAGES; i++)
	{
		for (j = 0; j <= MODULI_SET_SIZE*2; j++)
		{
			in_A[i][j] = 0;
		}
	}


	// Conversion of A

	long long temp1, temp2, temp3;

		for (int p = 0; p < NUMBER_OF_MESSAGES; p++)
		{
			for (i = 0, j = a_counter[p]; j>=0; i++, j--)
			{
				for (unsigned int k = 0; k <= MODULI_SET_SIZE*2; k++)
				{
					temp1 = (long long) buffer_A[p][j] * (long long)input[k][i];
					temp2 = (long long) in_A[p][k] + temp1;
					temp3 = temp2 % input1[k];
					in_A[p][k] = temp3;
				}
			}
	}


	// Conversion of B

	for (i = 0, j = b_counter; j>=0; i++, j--)
	{
		for (unsigned int k = 0; k <= MODULI_SET_SIZE*2; k++)
		{
			temp1 = (long long) buffer_B[j] * (long long)input[k][i];
			temp2 = (long long) in_B[k] + temp1;
			temp3 = temp2 % input1[k];
			in_B[k] = temp3;
		}
	}



	// Conversion of N

	for (i = 0, j = n_counter; j>=0; i++, j--)
	{
		for (unsigned int k = 0; k <= MODULI_SET_SIZE*2; k++)
		{
			temp1 = (long long) buffer_N[j] * (long long)input[k][i];
			temp2 = (long long) in_N[k] + temp1;
			temp3 = temp2 % input1[k];
			in_N[k] = temp3;
		}
	}

	printf("\nInitial RNS conversion complete.");
	/*printf("\n\nA[] = ");
	for (i = 0; i<= MODULI_SET_SIZE*2; i++)
	{
		printf("%d ", in_A[i]);
	}

	printf("\n\nB[] = ");
	for (i = 0; i<= MODULI_SET_SIZE*2; i++)
	{
		printf("%d ", in_B[i]);
	}

	printf("\n\nN[] = ");
	for (i = 0; i<= MODULI_SET_SIZE*2; i++)
	{
		printf("%d ", in_N[i]);
	}*/

	////////////////////// Perform precomputations /////////////////////////////////////

	/////Precomputations for approximate base conversion/////

	// Precompute Mi and |Mi^(-1)| mi

	/*for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	{
		Mi[i] = M / in_modulus[i]; // Compute Mi
		MiInverse[i] = ModularInverse(Mi[i], in_modulus[i]) % in_modulus[i]; // |Mi^(-1)| mi
	}*/

	FILE *fp2=fopen("MiInverse.txt", "r");
		if (fp2)
		{
			while(feof(fp2)== 0)
			{
				for (i=0; i <= (MODULI_SET_SIZE-1); i++)
				{
					fscanf(fp2,"%u",&MiInverse[i]);
				}
			}
		printf("\nMiInverse file read complete.");
		fclose(fp2);
		}
		else
		printf("\nFile Read Error!\n");





	// Precompute  |Mi| mj
	/*for (j = 0; j <= MODULI_SET_SIZE ; j++) // j = 6 because the redundant modulus also needs to be calculated here
	{
		for (i = 0; i<=MODULI_SET_SIZE-1; i++)
		{
			Mij[j][i] = Mi[i] % in_modulus[j+MODULI_SET_SIZE];
		}
	}*/

	FILE *fp3=fopen("Mij.txt", "r");
			if (fp3)
			{
				while(feof(fp3)== 0)
				{
					for (i=0; i <= (MODULI_SET_SIZE); i++ )
					{
						for(j=0; j<=(MODULI_SET_SIZE - 1); j++)
						{
							fscanf(fp3,"%u",&Mij[i][j]);
						}
					}
				}
			printf("\nMij file read complete.");
			fclose(fp3);
			}
			else
			printf("\nFile Read Error!\n");



	/////Precomputations for exact base conversion/////

	// Precompute Mpj, |Mpj|mr  and |Mpj^(-1)| mj

	/*for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	{
		Mpj[i] = MP / in_modulus[i+MODULI_SET_SIZE+1]; // Compute Mpj
		Mpjr[i] = Mpj[i] % in_modulus[MODULI_SET_SIZE];  // Compute |Mpj|mr
		MpjInverse[i] = ModularInverse(Mpj[i], in_modulus[i+MODULI_SET_SIZE+1]) % in_modulus[i+MODULI_SET_SIZE+1]; // |Mpj^(-1)| mj
	}*/

	FILE *fp4=fopen("Mpjr.txt", "r");
		if (fp4)
		{
			while(feof(fp4)== 0)
			{
				for (i=0; i <= (MODULI_SET_SIZE-1); i++)
				{
					fscanf(fp4,"%u",&Mpjr[i]);
				}
			}
		printf("\nMpjr file read complete.");
		fclose(fp4);
		}
		else
			printf("\nFile Read Error!\n");



		FILE *fp5=fopen("MpjInverse.txt", "r");
		if (fp5)
		{
			while(feof(fp5)== 0)
			{
				for (i=0; i <= (MODULI_SET_SIZE-1); i++)
				{
					fscanf(fp5,"%u",&MpjInverse[i]);
				}
			}
		printf("\nMpjInverse file read complete.");
		fclose(fp5);
		}
		else
			printf("\nFile Read Error!\n");





	//Precompute |Mp(^-1)|mr

	/*MpInvR = ModularInverse(MP, in_modulus[MODULI_SET_SIZE]);*/

	FILE *fp6=fopen("MpInvR.txt", "r");
		if (fp6)
		{
			while(feof(fp6)== 0)
			{
				fscanf(fp6,"%u",&MpInvR);
			}
		printf("\nMpInvR file read complete.");
		fclose(fp6);
		}
		else
			printf("\nFile Read Error!\n");




	// Precompute  |Mpj| mi
	/*for (i = 0; i <= MODULI_SET_SIZE-1 ; i++)
	{
		for (j = 0; j<=MODULI_SET_SIZE-1; j++)
		{
			Mpji[i][j] = Mpj[j] % in_modulus[i];
		}
	} */

	FILE *fp7=fopen("Mpji.txt", "r");
		if (fp7)
		{
			while(feof(fp7)== 0)
			{
				for (i=0; i <= (MODULI_SET_SIZE-1); i++ )
				{
					for(j=0; j<=(MODULI_SET_SIZE - 1); j++)
					{
						fscanf(fp7,"%u",&Mpji[i][j]);
					}
				}
			}
		printf("\nMpji file read complete.");
		fclose(fp7);
		}
		else
			printf("\nFile Read Error!\n");



/*
	///Extra precomputation needed to perform exact base conversion for the first base conversion in the final Montgomery pass
	// Precompute |Mi|mr

	//for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	//{
	//	Mir[i] = Mi[i] % in_modulus[MODULI_SET_SIZE];
	//}

	ifstream indata8; // indata is like cin
    unsigned int num8;

	indata8.open("Mir.txt"); // opens the file
	if(!indata8) { // file couldn't be opened
      cerr << "Error: file could not be opened" << endl;
      exit(1);
   }

	indata8 >> num8;
	for (i=0; (i <= (MODULI_SET_SIZE - 1))&&(!indata8.eof()); i++ )
	{
			Mir[i] = num8;
			indata8 >> num8; // sets EOF flag if no value found
	}

	indata8.close();
	printf("\nMir file read complete.");




	//Precompute |M(^-1)|mr

	//MInvR = ModularInverse(M, in_modulus[MODULI_SET_SIZE]);

	ifstream indata9; // indata is like cin
    unsigned int num9;

	indata9.open("MInvR.txt"); // opens the file
	if(!indata9) { // file couldn't be opened
      cerr << "Error: file could not be opened" << endl;
      exit(1);
   }

	indata9 >> num9;
	while(!indata9.eof())
	{
			MInvR = num9;
			indata9 >> num9; // sets EOF flag if no value found
	}

	indata9.close();
	printf("\nMInvR file read complete.");

*/


	/////Precomputations for third step/////


	//Precompute M^(-1)

	/*for (i = 0; i <= MODULI_SET_SIZE; i++)
	{
		MInv[i] = ModularInverse(M, in_modulus[i+MODULI_SET_SIZE]);
	}*/

	FILE *fp10=fopen("MInv.txt", "r");
			if (fp10)
			{
				while(feof(fp10)== 0)
				{
					for (i=0; i <= (MODULI_SET_SIZE); i++)
					{
						fscanf(fp10,"%u",&MInv[i]);
					}
				}
			printf("\nMInv file read complete.");
			fclose(fp10);
			}
			else
		printf("\nFile Read Error!\n");



	/*//////////////////////Perform first pass of Montgomery multiplication ///////////////

	MM(in_A, in_B, in_N, in_modulus, &M, &MP, Mi, MiInverse, Mij, Mpj, Mpjr, MpjInverse, Mpji, &MpInvR, MInv, Result);*/




	////////////////////// Precompute M^2 mod N  ////////////////////////////////////////

	/*for (i =0; i <= MODULI_SET_SIZE*2; i++)
	{
		MsqModN[i] = MsquareN % in_modulus[i];
	}*/

	FILE *fp11=fopen("MsqModN.txt", "r");
		if (fp11)
		{
			while(feof(fp11)== 0)
			{
				for (i=0; i <= (MODULI_SET_SIZE*2); i++)
				{
					fscanf(fp11,"%u",&MsqModN[i]);
				}
			}
		printf("\nMsqModN file read complete.");
		fclose(fp11);
		}
		else
		printf("\nFile Read Error!\n");

	////////////////////// Precompute MP mod mi  ////////////////////////////////////////

		FILE *fp12=fopen("Mpi.txt", "r");
				if (fp12)
				{
					while(feof(fp12)== 0)
					{
						for (i=0; i <= (MODULI_SET_SIZE-1); i++)
						{
							fscanf(fp12,"%u",&Mpi[i]);
						}
					}
				printf("\nMpi file read complete.");
				fclose(fp12);
				}
				else
					printf("\nFile Read Error!\n");



	////////////////////// Precompute M mod mj  ////////////////////////////////////////

	FILE *fp13=fopen("Mj.txt", "r");
		if (fp13)
		{
			while(feof(fp13)== 0)
			{
				for (i=0; i <= (MODULI_SET_SIZE); i++)
				{
					fscanf(fp13,"%u",&Mj[i]);
				}
			}
		printf("\nMj file read complete.");
		fclose(fp13);
		}
		else
			printf("\nFile Read Error!\n");


/*
	/////////////////////// Precompute MI /////////////////////////


ifstream indata14; // indata is like cin
    unsigned int num14;

	indata14.open("MI.txt"); // opens the file
	if(!indata14) { // file couldn't be opened
      cerr << "Error: file could not be opened" << endl;
      exit(1);
   }

	indata14 >> num14;
	for (i = 0; (i <= (MODULI_SET_SIZE - 1))&&(!indata14.eof()); i++ )
	{
		for(j = 0; (j <= MODULI_SET_SIZE - 1 )&&(!indata14.eof()); j++)
		{
			MI[i][j]=num14;
			indata14 >> num14; // sets EOF flag if no value found
		}
	}

	indata14.close();
	printf("\nMI file read complete.");

*/


	/*//////////////////////Perform final pass of Montgomery multiplication ///////////////

	printf(" \n\n\n\nSecoond pass starting .....");

	MMlast(MsqModN, Result, in_N, in_modulus, &M, &MP, Mi, Mir, MiInverse, Mij, &MInvR, Mpj, Mpjr, MpjInverse, Mpji, &MpInvR, MInv, Final_Result);*/

	//////////////////////Perform Montgomery Exponentiation ///////////////

	printf(" \n\n\n\nCPU starting .....");


	for (i = 0; i < NUMBER_OF_MESSAGES; i++)
		{
			printf("\n\nCipher# %d", i+1);

			//Timer
			cutResetTimer(timer);
			cutStartTimer(timer);


			Exp(&in_A[i][0], /*in_B,*/ in_N, MsqModN, input1, Mpi, Mj, /*Mi, Mir,*/ MiInverse, Mij, /*&MInvR, Mpj,*/ Mpjr, MpjInverse, Mpji, &MpInvR, MInv, /*MI,*/ &Final_Result[i][0]);

			//Timer
			cutStopTimer(timer);
			cpu_time += cutGetTimerValue(timer)/1000;

			printf("\nCPU Processing Time = %f sec", cutGetTimerValue(timer)/1000);
		}

	printf("\n\nCPU completed.");
	printf("\n----------------------------------------");
	printf("\nNumber of Mesages = %d", NUMBER_OF_MESSAGES);
	printf("\nTotal CPU Processing Time = %f sec", cpu_time);
	printf("\n----------------------------------------");


	////////////////////////////////////GPU ////////////////////////////////////

	printf("\n\nGPU phase:");
	// Precomputed variables
	unsigned int e[] = {1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1}; // e = 65537
	unsigned int One[2*MODULI_SET_SIZE+1];
	for (i = 0; i <= 2*MODULI_SET_SIZE; i++) One[i] = 1;




	// pointers for precomputed variables
	unsigned int (*prt_A)[MODULI_SET_SIZE * 2 + 1], *prt_N, *prt_MsqModN, *prt_input1, *prt_Mpi, *prt_Mj, /*Mi, Mir,*/ *prt_MiInverse;
	unsigned int (*prt_Mij)[MODULI_SET_SIZE], /*&MInvR, Mpj,*/ *prt_Mpjr, *prt_MpjInverse, (*prt_Mpji)[MODULI_SET_SIZE], *prt_MpInvR, *prt_MInv /*MI,*/ ;
	unsigned int (*prt_Device_Final_Result)[MODULI_SET_SIZE * 2 + 1];
	unsigned int *prt_e, *prt_One;

	// assign precomputed variables to pointers
	prt_A = in_A;
	prt_N = in_N;
	prt_MsqModN = MsqModN;
	prt_input1 = input1;
	prt_Mpi = Mpi;
	prt_Mj = Mj;
	prt_MiInverse = MiInverse;
	prt_Mij = Mij;
	prt_Mpjr = Mpjr;
	prt_MpjInverse = MpjInverse;
	prt_Mpji = Mpji;
	prt_MpInvR = &MpInvR;
	prt_MInv = MInv;
	prt_Device_Final_Result = Device_Final_Result;
	prt_e = e;
	prt_One = One;



	//sizes of variables
	unsigned int size1 = sizeof(unsigned int);
	unsigned int size2 = (MODULI_SET_SIZE) * sizeof(unsigned int);
	unsigned int size3 = (MODULI_SET_SIZE+1) * sizeof(unsigned int);
	unsigned int size4 = (MODULI_SET_SIZE * 2 + 1) * sizeof(unsigned int);
	unsigned int size5 = (MODULI_SET_SIZE * MODULI_SET_SIZE) * sizeof(unsigned int);
	unsigned int size6 = ((MODULI_SET_SIZE+1)* MODULI_SET_SIZE) * sizeof(unsigned int);


	// allocate memory in GPU
	unsigned int (*d_A)[MODULI_SET_SIZE * 2 + 1];
	hipMalloc((void**)&d_A, NUMBER_OF_MESSAGES*size4);
	hipMemcpy(d_A, prt_A, NUMBER_OF_MESSAGES*size4, hipMemcpyHostToDevice);
	printf("\nA allocated");
	fflush(NULL);

	unsigned int* d_N;
	hipMalloc((void**)&d_N, size4);
	hipMemcpy(d_N, prt_N, size4, hipMemcpyHostToDevice);
	printf("\nN allocated");
	fflush(NULL);

	//unsigned int* d_MsqModN;
	//hipMalloc((void**)&d_MsqModN, size4);
	//hipMemcpy(d_MsqModN, prt_MsqModN, size4, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_MsqModN), prt_MsqModN, size4);
	printf("\nMsqModN allocated");
	fflush(NULL);

	unsigned int* d_input1;
	hipMalloc((void**)&d_input1, size4);
	hipMemcpy(d_input1, prt_input1, size4, hipMemcpyHostToDevice);
	printf("\ninput1 allocated");
	fflush(NULL);

	//unsigned int* d_Mpi;
	//hipMalloc((void**)&d_Mpi, size2);
	//hipMemcpy(d_Mpi, prt_Mpi, size2, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_Mpi), prt_Mpi, size2);
	printf("\nMpi allocated");
	fflush(NULL);

	//unsigned int* d_Mj;
	//hipMalloc((void**)&d_Mj, size3);
	//hipMemcpy(d_Mj, prt_Mj, size3, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_Mj), prt_Mj, size3);
	printf("\nMj allocated");
	fflush(NULL);

	//unsigned int* d_MiInverse;
	//hipMalloc((void**)&d_MiInverse, size2);
	//hipMemcpy(d_MiInverse, prt_MiInverse, size2, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_MiInverse), prt_MiInverse, size2);
	printf("\nMiInverse allocated");
	fflush(NULL);

	//unsigned int (*d_Mij)[MODULI_SET_SIZE];
	//hipMalloc((void**)&d_Mij, size6);
	//hipMemcpy(d_Mij, prt_Mij, size6, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_Mij), prt_Mij, size6);
	printf("\nMij allocated");
	fflush(NULL);


	//unsigned int* d_Mpjr;
	//hipMalloc((void**)&d_Mpjr, size2);
	//hipMemcpy(d_Mpjr, prt_Mpjr, size2, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_Mpjr), prt_Mpjr, size2);
	printf("\nMpjr allocated");
	fflush(NULL);

	//unsigned int* d_MpjInverse;
	//hipMalloc((void**)&d_MpjInverse, size2);
	//hipMemcpy(d_MpjInverse, prt_MpjInverse, size2, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_MpjInverse), prt_MpjInverse, size2);
	printf("\nMpjInverse allocated");
	fflush(NULL);

	//unsigned int (*d_Mpji)[MODULI_SET_SIZE];
	//hipMalloc((void**)&d_Mpji, size5);
	//hipMemcpy(d_Mpji, prt_Mpji, size5, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_Mpji), prt_Mpji, size5);
	printf("\nMpji allocated");
	fflush(NULL);

	//unsigned int* d_MpInvR;
	//hipMalloc((void**)&d_MpInvR, size1);
	//hipMemcpy(d_MpInvR, prt_MpInvR, size1, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_MpInvR), prt_MpInvR, size1);
	printf("\nMpInvR allocated");
	fflush(NULL);

	//unsigned int* d_MInv;
	//hipMalloc((void**)&d_MInv, size3);
	//hipMemcpy(d_MInv, prt_MInv, size3, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_MInv), prt_MInv, size3);
	printf("\nMInv allocated");
	fflush(NULL);

	unsigned int (*d_Device_Final_Result)[MODULI_SET_SIZE * 2 + 1];
	hipMalloc((void**)&d_Device_Final_Result, NUMBER_OF_MESSAGES*size4);
	hipMemcpy(d_Device_Final_Result, prt_Device_Final_Result, NUMBER_OF_MESSAGES*size4, hipMemcpyHostToDevice);
	printf("\nFinal_Result allocated");
	fflush(NULL);

	hipMemcpyToSymbol(HIP_SYMBOL(d_e), prt_e, SIZE_OF_e*sizeof(unsigned int));
	printf("\ne allocated");
	fflush(NULL);

	hipMemcpyToSymbol(HIP_SYMBOL(d_One), prt_One, size4);
	printf("\nOne allocated");
	fflush(NULL);



	// GPU configuration
	//dim3 DimQBlock(NUMBER_OF_THREADS, 1);
	//dim3 DimQGrid(1, 1);
	dim3 DimQBlock(NUMBER_OF_THREADS, 1);
	dim3 DimQGrid(NUMBER_OF_MESSAGES, 1);


	printf("\n\nGPU started......");

	cutResetTimer(timer);
	cutStartTimer(timer);

//	dExp <<< DimQGrid, DimQBlock >>> (d_A, /*in_B,*/ d_N, d_MsqModN, d_input1, d_Mpi, d_Mj, /*Mi, Mir,*/ d_MiInverse, d_Mij, /*&MInvR, Mpj,*/ d_Mpjr, d_MpjInverse, d_Mpji, d_MpInvR, d_MInv, /*MI,*/ d_Device_Final_Result);
	dExp <<< DimQGrid, DimQBlock >>> (d_A, d_N, d_input1, d_Device_Final_Result);

	//check if kernel invocation generated an error
	CUT_CHECK_ERROR("Kernel execution failed");


    hipDeviceSynchronize();

    cutStopTimer(timer);
    gpu_time = cutGetTimerValue(timer)/1000;


    printf ("\n\nGPU completed.....");

	hipMemcpy(prt_Device_Final_Result, d_Device_Final_Result, NUMBER_OF_MESSAGES*size4, hipMemcpyDeviceToHost);



	/*
	for( j = 0; j < NUMBER_OF_MESSAGES; j++)
	{
		printf("\n\nCipher# %d",j);
		printf("\n----------------------------------------------------------------\n");

		for (i = 0; i <= 2*MODULI_SET_SIZE; i++) printf("%u, ", Device_Final_Result[j][i]);

		printf("\n----------------------------------------------------------------");

	}
	*/


	printf("\n----------------------------------------");
	printf("\nTotal GPU Processing Time = %f sec", gpu_time);
	printf("\nSpeedup = %f", cpu_time/gpu_time);
	printf("\n----------------------------------------");


	for (i = 0; i < NUMBER_OF_MESSAGES; i++)
	{
		for(j = 0; j <= 2*MODULI_SET_SIZE; j++)
		{

			if (Final_Result[i][j] != Device_Final_Result[i][j])
			{
				printf("\nERROR !! Results from CPU and GPU don't Match.\n");
				exit(0);
			}
		}

	}

	printf("\n\nResults from CPU and GPU match.\n");


	hipFree(d_A);
	hipFree(d_N);
	hipFree(d_input1);
	hipFree(d_Device_Final_Result);
	hipFree(d_MsqModN);
	hipFree(d_Mpi);
	hipFree(d_Mj);
	hipFree(d_MiInverse);
	hipFree(d_Mij);
	hipFree(d_Mpjr);
	hipFree(d_MpjInverse);
	hipFree(d_Mpji);
	hipFree(&d_MpInvR);
	hipFree(d_MInv);


	getchar();
	return 0;
}




//////////////////////////Montgomery multiplication///////////////////////

unsigned int MM(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Mpi, /*unsigned int *Mj, unsigned int *Mi,*/ unsigned int *MiInverse, unsigned int (*Mij)[MODULI_SET_SIZE], /*unsigned int *Mpj,*/ unsigned int *Mpjr, unsigned int *MpjInverse, unsigned int (*Mpji)[MODULI_SET_SIZE], unsigned int *MpInvR, unsigned int *MInv, unsigned int *Result)
{	int i;
	unsigned int Q[MODULI_SET_SIZE]; // Variable for step 1 // Q = 6 as needed for the final Montgomery pass
	unsigned int Qp[MODULI_SET_SIZE+1]; // Varibale for step 2
	unsigned int Rp[MODULI_SET_SIZE+1]; // Varibale for step 3
	unsigned int R[MODULI_SET_SIZE]; // Varibale for step 4


	/////////////////Step 1: Q <- ( -A * B) * N^(-1) /////////////////////////

	FirstStep(A, B, N, modulus, Q);


	//////////////////Step 2: First base conversion using CRT /////////////////////

	ApproxExt( Q, Qp, modulus, /*Mi,*/ MiInverse, Mij);


	/////////////////Step 3: Rp <- (A * B + Qp * N) * M^(-1) /////////////////////////

	ThirdStep(A, B, N, modulus, Qp, Rp, MInv);


	//////////////////Step 4: Second base conversion using Shenoy and Kumaresan /////////////////////

	ExactExt( Rp, R, modulus, Mpi, /*Mpj,*/ Mpjr, MpjInverse, Mpji, &*MpInvR);



	// Copy results back

	for (i = 0; i<= MODULI_SET_SIZE-1; i++)
	{
		Result[i] = R[i];
	}

	for (i = 0; i<= MODULI_SET_SIZE; i++)
	{
		Result[i+MODULI_SET_SIZE] = Rp[i];
	}

	return 0;

}


//////////////////////////Final Montgomery multiplication///////////////////////

unsigned int MMlast(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Mpi, unsigned int *Mj, /*unsigned int *Mi, unsigned int *Mir,*/ unsigned int *MiInverse, unsigned int (*Mij)[MODULI_SET_SIZE], /*unsigned int *MInvR, unsigned int *Mpj,*/ unsigned int *Mpjr, unsigned int *MpjInverse, unsigned int (*Mpji)[MODULI_SET_SIZE], unsigned int *MpInvR, unsigned int *MInv, /*unsigned int (*MI)[MODULI_SET_SIZE],*/ unsigned int *Result)
{	int i;
	unsigned int Q[MODULI_SET_SIZE]; // Variable for step 1
	unsigned int Qp[MODULI_SET_SIZE+1]; // Varibale for step 2
	unsigned int Rp[MODULI_SET_SIZE+1]; // Varibale for step 3
	unsigned int R[MODULI_SET_SIZE]; // Varibale for step 4


	/////////////////Step 1: Q <- ( -A * B) * N^(-1) /////////////////////////

	FirstStep(A, B, N, modulus, Q);


	//////////////////Step 2: First base conversion using CRT /////////////////////

	//ApproxExt( Q, Qp, modulus, &*M, Mi, MiInverse, Mij);
	ExactExtLast( Q, Qp, modulus, Mj, /*Mi, Mir,*/ MiInverse, Mij/*, &*MInvR*/);
	//MRS( Q, Qp, modulus, MI);

	/////////////////Step 3: Rp <- (A * B + Qp * N) * M^(-1) /////////////////////////

	ThirdStep(A, B, N, modulus, Qp, Rp, MInv);


	//////////////////Step 4: Second base conversion using Shenoy and Kumaresan /////////////////////

	ExactExt( Rp, R, modulus, Mpi, /*Mpj,*/ Mpjr, MpjInverse, Mpji, &*MpInvR);



	// Copy results back

	for (i = 0; i<= MODULI_SET_SIZE-1; i++)
	{
		Result[i] = R[i];
	}

	for (i = 0; i<= MODULI_SET_SIZE; i++)
	{
		Result[i+MODULI_SET_SIZE] = Rp[i];
	}

return 0;
}




/////////////////Step 1: Q <- ( -A * B) * N^(-1) /////////////////////////

unsigned int FirstStep(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Q)
{
	long long temp;
	int i;

	for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	{
		Q[i] = 0; // Initialize Q
	}

	for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	{
		/*temp = -((long long)A[i] * (long long)B[i]) * (long long)ModularInverse( N[i], modulus[i]);
		Q[i] = (temp % (long long)modulus[i]) + (long long)modulus[i];*/ // Source :http://stackoverflow.com/questions/989943/weird-objective-c-mod-behavior
												 // Since temp is always negative here, use the first comment/answer in the source website.
		temp = ((unsigned long long)A[i] * (unsigned long long)B[i]) % modulus[i];
		temp = -((temp * (unsigned long long)ModularInverse( N[i], modulus[i])) % modulus[i]);
		Q[i] = (temp % modulus[i]) + modulus[i];
	}
	/*printf("\n\n\nStep1 [Q <- ( -A * B) * N^(-1)] complete.");
	printf("\n\nQ[] = ");
	for (i = 0; i<=MODULI_SET_SIZE-1; i++)
	{
		printf("%u ", Q[i]);
	}*/

	/*ofstream myfile3 ("Temp2.txt");
  	if (myfile3.is_open())
	  {


		for (i = 0; i<=(MODULI_SET_SIZE-1); i++)
		{
			myfile3 << Q[i] << " ";
		}

		 myfile3.close();
	  }
 	 else cout << "Unable to open file";*/


	return 0;
}



/////////////////Step 3: Rp <- (A * B + Qp * N) * M^(-1) /////////////////////////

unsigned int ThirdStep(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Qp, unsigned int *Rp, unsigned int *MInv)
{
	int i;
	unsigned int temp, temp1, temp2;

	// Compute Rp
	for (i = 0; i <= MODULI_SET_SIZE; i++)
	{
		//Rp[i] = ((A[i+MODULI_SET_SIZE] * B[i+MODULI_SET_SIZE] + Qp[i] * N[i+MODULI_SET_SIZE]) * MInv[i]) % modulus[i+MODULI_SET_SIZE];
		temp = ((unsigned long long)A[i+MODULI_SET_SIZE] * (unsigned long long)B[i+MODULI_SET_SIZE]) % modulus[i+MODULI_SET_SIZE];
		temp1 = ((unsigned long long)Qp[i] * (unsigned long long)N[i+MODULI_SET_SIZE]) % modulus[i+MODULI_SET_SIZE];
		temp2 = ((unsigned long long)temp + (unsigned long long)temp1) % modulus[i+MODULI_SET_SIZE];
		Rp[i] = ((unsigned long long)temp2 * (unsigned long long)MInv[i]) % modulus[i+MODULI_SET_SIZE];
	}

	/*printf("\n\n\nStep3 [Rp <- (A * B + Qp * N) * M^(-1)] complete.");
	printf("\n\nRp[] = ");
	for (i = 0; i<=MODULI_SET_SIZE; i++)
	{
		printf("%u ", Rp[i]);
	}*/

	/*ofstream myfile5 ("Temp4.txt");
  	if (myfile5.is_open())
	  {


		for (i = 0; i<=(MODULI_SET_SIZE); i++)
		{
			myfile5 << Rp[i] << " ";
		}

		 myfile5.close();
	  }
 	 else cout << "Unable to open file";*/


	return 0;

}



//////////////////Approximate base conversion using CRT /////////////////////

unsigned int ApproxExt( unsigned int *Q, unsigned int *Qp, unsigned int *modulus, /*unsigned int *Mi,*/ unsigned int *MiInverse, unsigned int (*Mij)[MODULI_SET_SIZE])
{
	unsigned int sigma[MODULI_SET_SIZE], temp;
	int i, j;


	// Compute sigma
	for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	{
		sigma[i] = ((long long)Q[i]*(long long)MiInverse[i]) % modulus[i];
	}


	// Compute Q^

	for (i = 0; i <= MODULI_SET_SIZE; i++)
	{
		Qp[i] = 0; // Initialize Q^
	}

	for (j = 0; j <= MODULI_SET_SIZE; j++)
	{
		for (i = 0; i <= MODULI_SET_SIZE-1; i++)
		{
			//Qp[j] = ((long long)Qp[j] + (long long)Mij[j][i] * (long long)sigma[i]) % (long long)modulus[j+MODULI_SET_SIZE];
			  temp = ((long long)Mij[j][i] * (long long)sigma[i]) % (long long)modulus[j+MODULI_SET_SIZE];
			  Qp[j] = ((long long)temp + (long long)Qp[j]) % (long long)modulus[j+MODULI_SET_SIZE];
		}
	}

	printf("\n\n\nStep2 [first base conversion] complete.");
	printf("\n\nQp[] = ");
	for (i = 0; i<=MODULI_SET_SIZE; i++)
	{
		printf("%u ", Qp[i]);
	}

	return 0;
}

//////////////////Exact base conversion using MRS /////////////////////

unsigned int MRS( unsigned int *Q, unsigned int *Qp, unsigned int *modulus, unsigned int (*MI)[MODULI_SET_SIZE])
{

	unsigned int xp[MODULI_SET_SIZE];
	unsigned long long temp = 1;
	int i, j;


	// compute the MRS coefficients
	xp[0] = Q[0] % modulus[0];

	for (i = 1; i<= MODULI_SET_SIZE-1; i++)
	{
		for(j = 0; j <= i-1; j++)
		{
			if (j <=0)
			{
				//temp1 = (Q[i]-xp[j])*MI[j][i];
				temp = ((modulus[i]+((long long)Q[i]-(long long)xp[j]))% modulus[i])%modulus[i];
				temp = (temp*(long long)MI[j][i])% modulus[i];
			}
			else
			{
				//temp1 = (temp1 - xp[j])*MI[j][i];
				temp = ((modulus[i]+(temp - (long long)xp[j]))% modulus[i])%modulus[i];
				temp = (temp*(long long)MI[j][i])% modulus[i];

			}
		}
		//xp[i] = temp % modulus[i];
		xp[i] = temp;
	}



	unsigned long long tRNS[MODULI_SET_SIZE+1], temp1[MODULI_SET_SIZE+1], temp2[MODULI_SET_SIZE+1];
	for (i = 0; i <= MODULI_SET_SIZE; i++)
	{
		tRNS[i] = xp[0];
		temp1[i] = 1;
	}

	temp = 1;
	for(i = 1; i <= MODULI_SET_SIZE-1; i++)
	{
		for (j = 0; j <= MODULI_SET_SIZE; j++)
		{
			temp1[j] = (temp1[j] * (long long)modulus[i-1])%modulus[MODULI_SET_SIZE+j];
			temp2[j] = ((long long)xp[i] * temp1[j])%modulus[MODULI_SET_SIZE+j];
			tRNS[j] = (tRNS[j]+temp2[j])%modulus[MODULI_SET_SIZE+j];
		}
		//X = X + xp[i] * temp1;
	}

	for (i = 0; i <= MODULI_SET_SIZE; i++)
	{
		Qp[i] = tRNS[i];
	}

	printf("\n\n\nStep2 [first base conversion] complete.");
	printf("\n\nQp[] = ");
	for (i = 0; i<=MODULI_SET_SIZE; i++)
	{
		printf("%u ", Qp[i]);
	}

	/*ofstream myfile4 ("Temp3.txt");
  	if (myfile4.is_open())
	  {


		for (i = 0; i<=(MODULI_SET_SIZE); i++)
		{
			myfile4 << Qp[i] << " ";
		}

		 myfile4.close();
	  }
 	 else cout << "Unable to open file";*/




	return 0;
}





//////////////////Exact base conversion using Shenoy and Kumaresan /////////////////////

unsigned int ExactExt( unsigned int *Rp, unsigned int *R, unsigned int *modulus, unsigned int *Mpi, /*unsigned int *Mpj,*/ unsigned int *Mpjr, unsigned int *MpjInverse, unsigned int (*Mpji)[MODULI_SET_SIZE], unsigned int *MpInvR)
{
	unsigned int Xi[MODULI_SET_SIZE],Beta, temp, temp1, temp2;
	long long  temp3;
	int i, j;


	// Compute xi

	for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	{
		Xi[i] = ((unsigned long long)Rp[i+1] * (unsigned long long)MpjInverse[i]) % modulus[i+MODULI_SET_SIZE+1];
	}


	//Compute Beta

	temp = 0;

	for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	{
		//temp = (temp + Mpjr[i] * Xi[i]) % modulus[MODULI_SET_SIZE];
		temp1 = ((unsigned long long)Mpjr[i] * (unsigned long long)Xi[i]) % modulus[MODULI_SET_SIZE];
		temp = ((unsigned long long)temp + (unsigned long long)temp1 ) % modulus[MODULI_SET_SIZE];

	}

	//temp = (modulus[MODULI_SET_SIZE] + (temp - Rp[0]) % modulus[MODULI_SET_SIZE]) % modulus[MODULI_SET_SIZE]; // Source :http://stackoverflow.com/questions/989943/weird-objective-c-mod-behavior
	temp3 = ((long long)temp - (long long)Rp[0]);
	//temp = temp % modulus[MODULI_SET_SIZE];
	temp3 = (modulus[MODULI_SET_SIZE] + temp3 % modulus[MODULI_SET_SIZE]) % modulus[MODULI_SET_SIZE];
	Beta = ((unsigned long long)temp3 * (unsigned long long)(*MpInvR)) % modulus[MODULI_SET_SIZE];							// Since temp will sometimes be positive ans sometimes negative,


	// Compute R
	for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	{
		R[i] = 0; // Initialize R
	}

	for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	{
		for (j = 0; j <= MODULI_SET_SIZE-1; j++)
		{
			//R[i] = (R[i] + Mpji[i][j] * Xi[j]) % modulus[i];
			temp2 = ((unsigned long long)Mpji[i][j] * (unsigned long long)Xi[j]) % modulus[i];
			R[i] = ((unsigned long long)R[i] + (unsigned long long)temp2)% modulus[i];
		}
		temp3 = R[i] - ((long long)Beta * (long long)(Mpi[i])) % modulus[i];
		R[i] = (modulus[i] + (temp3 % modulus[i])) % modulus[i]; // Source :http://stackoverflow.com/questions/989943/weird-objective-c-mod-behavior
	}															// Since temp will sometimes be positive ans sometimes negative,
																// use the third comment/answer in the source website.


	/*printf("\n\n\nStep4 [second base conversion] complete.");
	printf("\n\nR[] = ");
	for (i = 0; i<=MODULI_SET_SIZE-1; i++)
	{
		printf("%u ", R[i]);
	}*/


	/*ofstream myfile6 ("Temp5.txt");
  	if (myfile6.is_open())
	  {


		for (i = 0; i<=(MODULI_SET_SIZE-1); i++)
		{
			myfile6 << R[i] << " ";
		}

		 myfile6.close();
	  }
 	 else cout << "Unable to open file";*/




	return 0;


}


//////////////////Exact base conversion using Kawamura, Harrison and Shenoy and Kumaresan /////////////////////

unsigned int ExactExtLast( unsigned int *Q, unsigned int *Qp, unsigned int *modulus, unsigned int *Mj, /*unsigned int *Mi, unsigned int *Mir,*/ unsigned int *MiInverse, unsigned int (*Mij)[MODULI_SET_SIZE]/*, unsigned int *MInvR*/)
{
	unsigned int Xi[MODULI_SET_SIZE],Beta, temp2;
	float temp;
	int i, j;
	long long temp1;



	// Compute xi

	for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	{
		Xi[i] = ((unsigned long long)Q[i]*(unsigned long long)MiInverse[i]) % modulus[i];
	}


	//Compute Beta.
	//This part is done according to Kawamura et al (2000) page 5 equation 5 and idea from Harrison et al (2009) page 12.
	// Here Beta is essentially k as in Kawamura et al and Harrison et al.
	// This part needs further consideration and observation to see if the accumulation of results of the division below
	// actually works for large numbers of not. If not, alternative ways of doiing it would be (1) follow Bajard's base extension without conversion,
	// (2) Kawamura's approximate truncation method, or (3) Mixed Radix System

	temp = 0;

	for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	{
		temp = temp + (float)Xi[i] / (float)modulus[i];
	}

	Beta = temp;




	// Compute Qp
	// After the computation below, the validity of Qp[0] is questionalble; it needs to be seen what happens to this value for large numbers.
	for (i = 0; i <= MODULI_SET_SIZE; i++)
	{
		Qp[i] = 0; // Initialize R
	}

	for (j = 0; j <= MODULI_SET_SIZE; j++)
	{
		for (i = 0; i <= MODULI_SET_SIZE-1; i++)
		{
			//Qp[j] = ((unsigned long long)Qp[j] + (unsigned long long)Mij[j][i] * (unsigned long long)Xi[i]) % (unsigned long long)modulus[j+MODULI_SET_SIZE];
				temp2 = ((unsigned long long)Mij[j][i] * (unsigned long long)Xi[i]) % (unsigned long long)modulus[j+MODULI_SET_SIZE];
				Qp[j] = ((unsigned long long)Qp[j]+(unsigned long long)temp2) % (unsigned long long)modulus[j+MODULI_SET_SIZE];
		}
		temp1 = (long long)Qp[j] - (Beta * (long long)(Mj[j])) % (long long)modulus[j+MODULI_SET_SIZE];
		/*temp1 = (long long)Beta * (long long)(Mj[j]);
		temp1 = temp1 % (long long)modulus[j+MODULI_SET_SIZE];
		temp1 = (long long)Qp[j] - temp1;*/
		Qp[j] = (modulus[j+MODULI_SET_SIZE] + (temp1 % modulus[j+MODULI_SET_SIZE])) % modulus[j+MODULI_SET_SIZE]; // Source :http://stackoverflow.com/questions/989943/weird-objective-c-mod-behavior
	}															 // Since temp will sometimes be positive ans sometimes negative,
																 // use the third comment/answer in the source website.


	/*printf("\n\n\nStep2 [first base conversion] complete.");
	printf("\n\nQp[] = ");
	for (i = 0; i<=MODULI_SET_SIZE; i++)
	{
		printf("%u ", Qp[i]);
	}*/

	return 0;
}



/*//////////Compute modular inverse///////////////////////////////////////////////
Code from http://en.wikipedia.org/wiki/Modular_multiplicative_inverse
and http://en.wikipedia.org/wiki/Extended_Euclidean_algorithm
*/

unsigned int ModularInverse(unsigned int N, unsigned int D)
{	long long temp_gcd, x, y;

	temp_gcd = gcd(N, D, &x, &y);

	if (temp_gcd == 1)
		return ((x+D)%D); // takes care of negative values of x
	else
		return 0;
}


/*//////////Compute gcd///////////////////////////////////////////////
Code from http://en.wikipedia.org/wiki/Modular_multiplicative_inverse
and http://en.wikipedia.org/wiki/Extended_Euclidean_algorithm
*/

unsigned int gcd(unsigned int A, unsigned int B, long long *X, long long *Y)
{
    long long x, y, u, v, m, n, a, b, q, r;

    /* B = A(0) + B(1) */
    x = 0; y = 1;

    /* A = A(1) + B(0) */
    u = 1; v = 0;

    for (a = A, b = B; 0 != a; b = a, a = r, x = u, y = v, u = m, v = n) {
        /* b = aq + r and 0 <= r < a */
        q = b / a;
        r = b % a;

        /* r = Ax + By - aq = Ax + By - (Au + Bv)q = A(x - uq) + B(y - vq) */
        m = x - (u * q);
        n = y - (v * q);
    }

    /* Ax + By = gcd(A, B) */
    *X = x; *Y = y;

    return b;
}


//////// Compute X^e mod N ////////////////////////////////////////
// source: Kawamura's paper page 532
// This function works for both MM and MMlast (commented out with //) inside the cide below. However, tha last Montgomery call must be MMlast.

unsigned int Exp(unsigned int *A, /*unsigned int *B,*/ unsigned int *N, unsigned int *MsqModN, unsigned int *modulus, unsigned int *Mpi, unsigned int *Mj, /*unsigned int *Mi, unsigned int *Mir,*/ unsigned int *MiInverse, unsigned int (*Mij)[MODULI_SET_SIZE], /*unsigned int *MInvR, unsigned int *Mpj,*/ unsigned int *Mpjr, unsigned int *MpjInverse, unsigned int (*Mpji)[MODULI_SET_SIZE], unsigned int *MpInvR, unsigned int *MInv, /*unsigned int (*MI)[MODULI_SET_SIZE],*/ unsigned int *Result)
{
	unsigned int XPrime[2*MODULI_SET_SIZE+1], Y[2*MODULI_SET_SIZE+1];
	unsigned int e[] = {1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1}; // e = 65537
	unsigned int One[2*MODULI_SET_SIZE+1];
	int i, j;

	for (i = 0; i <= 2*MODULI_SET_SIZE; i++)
	{
		One[i] = 1;
	}


	//Step 1:
	//MM(A, MsqModN, N, modulus, Mpi, /*Mj, Mi,*/ MiInverse, Mij, /*Mpj,*/ Mpjr, MpjInverse, Mpji, &*MpInvR, MInv, XPrime);
	MMlast(A, MsqModN, N, modulus, Mpi, Mj, /*Mi, Mir,*/ MiInverse, Mij, /*&*MInvR, Mpj,*/ Mpjr, MpjInverse, Mpji, &*MpInvR, MInv, /*MI,*/ XPrime);

	//Step 2:
	for (i = 0; i <= 2*MODULI_SET_SIZE; i++)
	{
		Y[i] = XPrime[i];
	}



	/*ofstream myfile2 ("Temp.txt");
  	if (myfile2.is_open())
	  {


		for (i = 0; i<=(MODULI_SET_SIZE-1); i++)
		{
			myfile2 << Y[i] << " ";
		}

		 myfile2.close();
	  }
 	 else cout << "Unable to open file";*/



	//Step 3:
	for (j = 15; j >= 0; j--)
	{
		//Step 4:
		//MM(Y, Y, N, modulus, Mpi, /*Mj, Mi,*/ MiInverse, Mij, /*Mpj,*/ Mpjr, MpjInverse, Mpji, &*MpInvR, MInv, Result);
		MMlast(Y, Y, N, modulus, Mpi, Mj, /*Mi, Mir,*/ MiInverse, Mij, /*&*MInvR, Mpj,*/ Mpjr, MpjInverse, Mpji, &*MpInvR, MInv, /*MI,*/ Result);

		for (i = 0; i <= 2*MODULI_SET_SIZE; i++)
		{
			Y[i] = Result[i];
		}

	/*ofstream myfile2 ("Temp1.txt");
  	if (myfile2.is_open())
	  {


		for (i = 0; i<=(MODULI_SET_SIZE-1); i++)
		{
			myfile2 << Y[i] << " ";
		}

		 myfile2.close();
	  }
 	 else cout << "Unable to open file";*/




		//Step 5:
		if (e[j] == 1)
		{
			//MM(Result, XPrime, N, modulus, Mpi, /*Mj, Mi,*/ MiInverse, Mij, /*Mpj,*/ Mpjr, MpjInverse, Mpji, &*MpInvR, MInv, TEMP);
			MMlast(Result, XPrime, N, modulus, Mpi, Mj, /*Mi, Mir,*/ MiInverse, Mij, /*&*MInvR, Mpj,*/ Mpjr, MpjInverse, Mpji, &*MpInvR, MInv, /*MI,*/ Y);


		}


	}

	//Step 7:
	MMlast(Y, One, N, modulus, Mpi, Mj, /*Mi, Mir,*/ MiInverse, Mij, /*&*MInvR, Mpj,*/ Mpjr, MpjInverse, Mpji, &*MpInvR, MInv, /*MI,*/ Result);

	/*
	printf("\n----------------------------------------------------------------\n");
	for (i = 0; i <= 2*MODULI_SET_SIZE; i++) printf("%u, ", Result[i]);
	printf("\n----------------------------------------------------------------");
	*/

	return 0;

}