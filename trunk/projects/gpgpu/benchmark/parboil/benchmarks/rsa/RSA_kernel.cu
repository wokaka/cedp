
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include "math.h"

#define MODULI_SET_SIZE 33
#define NUMBER_OF_MESSAGES 6800
#define NUMBER_OF_THREADS 34
#define SIZE_OF_e 17


__constant__ unsigned int d_MiInverse[MODULI_SET_SIZE], d_Mij[MODULI_SET_SIZE+1][MODULI_SET_SIZE];
__constant__ unsigned int d_Mpjr[MODULI_SET_SIZE], d_MpjInverse[MODULI_SET_SIZE], d_Mpji[MODULI_SET_SIZE][MODULI_SET_SIZE], d_MpInvR, d_Mpi[MODULI_SET_SIZE];
__constant__ unsigned int d_Mj[MODULI_SET_SIZE+1];
__constant__ unsigned int d_MInv[MODULI_SET_SIZE+1];
__constant__ unsigned int d_MsqModN[MODULI_SET_SIZE*2+1];
__constant__ unsigned int d_e[SIZE_OF_e];
__constant__ unsigned int d_One[MODULI_SET_SIZE*2+1];


//__device__ unsigned int XPrime[NUMBER_OF_MESSAGES][2*MODULI_SET_SIZE+1], Y[NUMBER_OF_MESSAGES][2*MODULI_SET_SIZE+1];
//__device__ unsigned int TR[NUMBER_OF_MESSAGES][MODULI_SET_SIZE+1], TRR[NUMBER_OF_MESSAGES][MODULI_SET_SIZE+1];




__global__ void dExp(unsigned int (*A)[2*MODULI_SET_SIZE+1], unsigned int *N, unsigned int *modulus, unsigned int (*Result)[2*MODULI_SET_SIZE+1]);

__device__ unsigned int d_MMlast(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Result);
__device__ unsigned int d_FirstStep(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Q);
__device__ unsigned int d_ThirdStep(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Qp, unsigned int *Rp);
__device__ unsigned int d_ExactExt( unsigned int *Rp, unsigned int *R, unsigned int *modulus);
__device__ unsigned int d_ExactExtLast( unsigned int *Q, unsigned int *Qp, unsigned int *modulus);
__device__ unsigned int d_ModularInverse(unsigned int N, unsigned int D);
__device__ unsigned int d_gcd(unsigned int A, unsigned int B, long long *X, long long *Y);




//////// Compute X^e mod N ////////////////////////////////////////
// source: Kawamura's paper page 532
// This function works for both MM and MMlast (commented out with //) inside the cide below. However, tha last Montgomery call must be MMlast.

__global__ void dExp(unsigned int (*A)[2*MODULI_SET_SIZE+1], unsigned int *N, unsigned int *modulus, unsigned int (*Result)[2*MODULI_SET_SIZE+1])
{
	__shared__ unsigned int XPrime[2*MODULI_SET_SIZE+1], Y[2*MODULI_SET_SIZE+1];
	//unsigned int e[] = {1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1}; // e = 65537
	//unsigned int One[2*MODULI_SET_SIZE+1];
	int i, j;
/*
__shared__ unsigned int Q[MODULI_SET_SIZE]; // Variable for step 1 // Q = 6 as needed for the final Montgomery pass
__shared__ unsigned int Qp[MODULI_SET_SIZE+1]; // Varibale for step 2
__shared__ unsigned int Rp[MODULI_SET_SIZE+1]; // Varibale for step 3
__shared__ unsigned int R[MODULI_SET_SIZE]; // Varibale for step 4
*/
	//unsigned int TR[NUMBER_OF_MESSAGES][MODULI_SET_SIZE+1], TRR[NUMBER_OF_MESSAGES][MODULI_SET_SIZE+1];

	//int tx = threadIdx.x;
	int bx = blockIdx.x;

	//for (i = 0; i <= 2*MODULI_SET_SIZE; i++)
	//{
	//	One[i] = 1;
	//}


	//Step 1:
	//MM(A, MsqModN, N, modulus, Mpi, /*Mj, Mi,*/ MiInverse, Mij, /*Mpj,*/ Mpjr, MpjInverse, Mpji, &*MpInvR, MInv, XPrime);
	d_MMlast(A[bx], d_MsqModN, N, modulus, XPrime);
		/*
		d_FirstStep(A[bx], d_MsqModN, N, modulus, Q);
		d_ExactExtLast(Q, Qp, modulus);
		d_ThirdStep(A[bx], d_MsqModN, N, modulus, Qp, Rp);
		d_ExactExt(Rp, R, modulus);

		for (i = 0; i<= MODULI_SET_SIZE-1; i++)	XPrime[i] = R[i];
		for (i = 0; i<= MODULI_SET_SIZE; i++) XPrime[i+MODULI_SET_SIZE] = Rp[i];
		*/
	//Step 2:
	for (i = 0; i <= 2*MODULI_SET_SIZE; i++)
	{
		Y[i] = XPrime[i];
	}

	//Step 3:
	for (j = SIZE_OF_e - 2; j >= 0; j--)
	{
		d_MMlast(Y, Y, N, modulus, Result[bx]);
		/*
		d_FirstStep(Y, Y, N, modulus, Q);
		d_ExactExtLast(Q, Qp, modulus);
		d_ThirdStep(Y, Y, N, modulus, Qp, Rp);
		d_ExactExt(Rp, R, modulus);

		for (i = 0; i<= MODULI_SET_SIZE-1; i++)	Result[bx][i] = R[i];
		for (i = 0; i<= MODULI_SET_SIZE; i++) Result[bx][i+MODULI_SET_SIZE] = Rp[i];
		*/
		for (i = 0; i <= 2*MODULI_SET_SIZE; i++)
		{
			Y[i] = Result[bx][i];
		}


		//Step 5:
		if (d_e[j] == 1)
		{
			d_MMlast(Result[bx], XPrime, N, modulus, Y);
			/*
			d_FirstStep(Result[bx], XPrime, N, modulus, Q);
			d_ExactExtLast(Q, Qp, modulus);
			d_ThirdStep(Result[bx], XPrime, N, modulus, Qp, Rp);
			d_ExactExt(Rp, R, modulus);

			for (i = 0; i<= MODULI_SET_SIZE-1; i++)	Y[i] = R[i];
			for (i = 0; i<= MODULI_SET_SIZE; i++) Y[i+MODULI_SET_SIZE] = Rp[i];
			*/
		}
	}

	d_MMlast(Y, d_One, N, modulus, Result[bx]);
	/*
	d_FirstStep(Y, One, N, modulus, Q);
	d_ExactExtLast(Q, Qp, modulus);
	d_ThirdStep(Y, One, N, modulus, Qp, Rp);
	d_ExactExt(Rp, R, modulus);

	for (i = 0; i<= MODULI_SET_SIZE-1; i++)	Result[bx][i] = R[i];
	for (i = 0; i<= MODULI_SET_SIZE; i++) Result[bx][i+MODULI_SET_SIZE] = Rp[i];
	*/

}





//////////////////////////Final Montgomery multiplication///////////////////////

__device__ unsigned int d_MMlast(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Result)
{	int i;
	/*
	__shared__ unsigned int Q[MODULI_SET_SIZE]; // Variable for step 1
	__shared__ unsigned int Qp[MODULI_SET_SIZE+1]; // Varibale for step 2
	__shared__ unsigned int Rp[MODULI_SET_SIZE+1]; // Varibale for step 3
	__shared__ unsigned int R[MODULI_SET_SIZE]; // Varibale for step 4
	*/
	__shared__ unsigned int T[MODULI_SET_SIZE+1];
	__shared__ unsigned int TT[MODULI_SET_SIZE+1];

	/////////////////Step 1: Q <- ( -A * B) * N^(-1) /////////////////////////

	d_FirstStep(A, B, N, modulus, T);


	//////////////////Step 2: First base conversion using CRT /////////////////////

	//ApproxExt( Q, Qp, modulus, &*M, Mi, MiInverse, Mij);
	d_ExactExtLast( T, TT, modulus);
	//MRS( Q, Qp, modulus, MI);

	/////////////////Step 3: Rp <- (A * B + Qp * N) * M^(-1) /////////////////////////

	d_ThirdStep(A, B, N, modulus, TT, T);


	//////////////////Step 4: Second base conversion using Shenoy and Kumaresan /////////////////////

	d_ExactExt( T, TT, modulus);



	// Copy results back
	/*
	for (i = 0; i<= MODULI_SET_SIZE-1; i++)
	{
		Result[i] = R[i];
	}

	for (i = 0; i<= MODULI_SET_SIZE; i++)
	{
		Result[i+MODULI_SET_SIZE] = Rp[i];
	}
	*/

	for (i = 0; i<= MODULI_SET_SIZE-1; i++)
		{
			Result[i] = TT[i];
		}

		for (i = 0; i<= MODULI_SET_SIZE; i++)
		{
			Result[i+MODULI_SET_SIZE] = T[i];
	}

return 0;
}




/////////////////Step 1: Q <- ( -A * B) * N^(-1) /////////////////////////

__device__ unsigned int d_FirstStep(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Q)
{
	long long temp;
		//int i;
		int tx = threadIdx.x;

		/*for (i = 0; i <= MODULI_SET_SIZE-1; i++)
		{
			Q[i] = 0; // Initialize Q
		}*/

		//for (i = 0; i <= MODULI_SET_SIZE-1; i++)
		//{
			/*temp = -((long long)A[i] * (long long)B[i]) * (long long)ModularInverse( N[i], modulus[i]);
			Q[i] = (temp % (long long)modulus[i]) + (long long)modulus[i];*/ // Source :http://stackoverflow.com/questions/989943/weird-objective-c-mod-behavior
			if (tx <33)										 // Since temp is always negative here, use the first comment/answer in the source website.
			{
			temp = ((unsigned long long)A[tx] * (unsigned long long)B[tx]) % modulus[tx];
			temp = -((temp * (unsigned long long)d_ModularInverse( N[tx], modulus[tx])) % modulus[tx]);
			Q[tx] = (temp % modulus[tx]) + modulus[tx];
			}
		//}
		__syncthreads();

return 0;

}



/////////////////Step 3: Rp <- (A * B + Qp * N) * M^(-1) /////////////////////////

__device__ unsigned int d_ThirdStep(unsigned int *A, unsigned int *B, unsigned int *N, unsigned int *modulus, unsigned int *Qp, unsigned int *Rp)
{
	//int i;
	unsigned int temp, temp1, temp2;
	int tx = threadIdx.x;

	// Compute Rp
	//for (i = 0; i <= MODULI_SET_SIZE; i++)
	//{
		//Rp[i] = ((A[i+MODULI_SET_SIZE] * B[i+MODULI_SET_SIZE] + Qp[i] * N[i+MODULI_SET_SIZE]) * MInv[i]) % modulus[i+MODULI_SET_SIZE];
		temp = ((unsigned long long)A[tx+MODULI_SET_SIZE] * (unsigned long long)B[tx+MODULI_SET_SIZE]) % modulus[tx+MODULI_SET_SIZE];
		temp1 = ((unsigned long long)Qp[tx] * (unsigned long long)N[tx+MODULI_SET_SIZE]) % modulus[tx+MODULI_SET_SIZE];
		temp2 = ((unsigned long long)temp + (unsigned long long)temp1) % modulus[tx+MODULI_SET_SIZE];
		Rp[tx] = ((unsigned long long)temp2 * (unsigned long long)d_MInv[tx]) % modulus[tx+MODULI_SET_SIZE];

		__syncthreads();

	//}


	return 0;

}



//////////////////Approximate base conversion using CRT /////////////////////

__device__ unsigned int d_ApproxExt( unsigned int *Q, unsigned int *Qp, unsigned int *modulus)
{
	unsigned int sigma[MODULI_SET_SIZE], temp;
	int i, j;


	// Compute sigma
	for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	{
		sigma[i] = ((long long)Q[i]*(long long)d_MiInverse[i]) % modulus[i];
	}


	// Compute Q^

	for (i = 0; i <= MODULI_SET_SIZE; i++)
	{
		Qp[i] = 0; // Initialize Q^
	}

	for (j = 0; j <= MODULI_SET_SIZE; j++)
	{
		for (i = 0; i <= MODULI_SET_SIZE-1; i++)
		{
			//Qp[j] = ((long long)Qp[j] + (long long)Mij[j][i] * (long long)sigma[i]) % (long long)modulus[j+MODULI_SET_SIZE];
			  temp = ((long long)d_Mij[j][i] * (long long)sigma[i]) % (long long)modulus[j+MODULI_SET_SIZE];
			  Qp[j] = ((long long)temp + (long long)Qp[j]) % (long long)modulus[j+MODULI_SET_SIZE];
		}
	}



	return 0;
}




//////////////////Exact base conversion using Shenoy and Kumaresan /////////////////////

__device__ unsigned int d_ExactExt( unsigned int *Rp, unsigned int *R, unsigned int *modulus)
{
	__shared__ unsigned int Xi[MODULI_SET_SIZE];
	__shared__ unsigned int Beta;

	unsigned int temp, temp1, temp2;
	long long  temp3;
	int i, j;

	int tx = threadIdx.x;

	// Compute xi

	//for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	//{
		if(tx < 33)
	{
		Xi[tx] = ((unsigned long long)Rp[tx+1] * (unsigned long long)d_MpjInverse[tx]) % modulus[tx+MODULI_SET_SIZE+1];
	}

	__syncthreads();

	//}


	//Compute Beta
	//if(tx < 1)
	{
		temp = 0;

		for (i = 0; i <= MODULI_SET_SIZE-1; i++)
		{
			//temp = (temp + Mpjr[i] * Xi[i]) % modulus[MODULI_SET_SIZE];
			temp1 = ((unsigned long long)d_Mpjr[i] * (unsigned long long)Xi[i]) % modulus[MODULI_SET_SIZE];
			temp = ((unsigned long long)temp + (unsigned long long)temp1 ) % modulus[MODULI_SET_SIZE];

		}

		//temp = (modulus[MODULI_SET_SIZE] + (temp - Rp[0]) % modulus[MODULI_SET_SIZE]) % modulus[MODULI_SET_SIZE]; // Source :http://stackoverflow.com/questions/989943/weird-objective-c-mod-behavior
		temp3 = ((long long)temp - (long long)Rp[0]);
		//temp = temp % modulus[MODULI_SET_SIZE];
		temp3 = (modulus[MODULI_SET_SIZE] + temp3 % modulus[MODULI_SET_SIZE]) % modulus[MODULI_SET_SIZE];
		Beta = ((unsigned long long)temp3 * (unsigned long long)(d_MpInvR)) % modulus[MODULI_SET_SIZE];							// Since temp will sometimes be positive ans sometimes negative,
	}



	// Compute R
	//for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	//{
	if(tx < 33)
	{
		R[tx] = 0; // Initialize R

	//}

	//for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	//{
		for (j = 0; j <= MODULI_SET_SIZE-1; j++)
		{
			//R[i] = (R[i] + Mpji[i][j] * Xi[j]) % modulus[i];
			temp2 = ((unsigned long long)d_Mpji[tx][j] * (unsigned long long)Xi[j]) % modulus[tx];
			R[tx] = ((unsigned long long)R[tx] + (unsigned long long)temp2)% modulus[tx];
		}
		temp3 = R[tx] - ((long long)Beta * (long long)(d_Mpi[tx])) % modulus[tx];
		R[tx] = (modulus[tx] + (temp3 % modulus[tx])) % modulus[tx]; // Source :http://stackoverflow.com/questions/989943/weird-objective-c-mod-behavior
	}															// Since temp will sometimes be positive ans sometimes negative,
																// use the third comment/answer in the source website.


	__syncthreads();

	return 0;
}


//////////////////Exact base conversion using Kawamura, Harrison and Shenoy and Kumaresan /////////////////////

__device__ unsigned int d_ExactExtLast(unsigned int *Q, unsigned int *Qp, unsigned int *modulus)
{
	__shared__ unsigned int Xi[MODULI_SET_SIZE];
	__shared__ unsigned int Beta;
	unsigned int temp2;

	float temp;
	int i;
	long long temp1;

	int tx = threadIdx.x;


	// Compute xi

	//for (i = 0; i <= MODULI_SET_SIZE-1; i++)
	//{
	if(tx < 33)
	{
		Xi[tx] = ((unsigned long long)Q[tx]*(unsigned long long)d_MiInverse[tx]) % modulus[tx];
	}

	__syncthreads();
	//}


	//Compute Beta.
	//This part is done according to Kawamura et al (2000) page 5 equation 5 and idea from Harrison et al (2009) page 12.
	// Here Beta is essentially k as in Kawamura et al and Harrison et al.
	// This part needs further consideration and observation to see if the accumulation of results of the division below
	// actually works for large numbers of not. If not, alternative ways of doiing it would be (1) follow Bajard's base extension without conversion,
	// (2) Kawamura's approximate truncation method, or (3) Mixed Radix System

	//if(tx < 1)
	{
		temp = 0;

		for (i = 0; i <= MODULI_SET_SIZE-1; i++)
		{
			temp = temp + (float)Xi[i] / (float)modulus[i];
		}
		Beta = temp;
	}





	// Compute Qp
	// After the computation below, the validity of Qp[0] is questionalble; it needs to be seen what happens to this value for large numbers.

		//for (i = 0; i <= MODULI_SET_SIZE; i++)
		//{
			Qp[tx] = 0; // Initialize R
		//}


	//for (j = 0; j <= MODULI_SET_SIZE; j++)
	//{
		for (i = 0; i <= MODULI_SET_SIZE-1; i++)
		{
			//Qp[j] = ((unsigned long long)Qp[j] + (unsigned long long)Mij[j][i] * (unsigned long long)Xi[i]) % (unsigned long long)modulus[j+MODULI_SET_SIZE];
				temp2 = ((unsigned long long)d_Mij[tx][i] * (unsigned long long)Xi[i]) % (unsigned long long)modulus[tx+MODULI_SET_SIZE];
				Qp[tx] = ((unsigned long long)Qp[tx]+(unsigned long long)temp2) % (unsigned long long)modulus[tx+MODULI_SET_SIZE];
		}
		temp1 = (long long)Qp[tx] - (Beta * (long long)(d_Mj[tx])) % (long long)modulus[tx+MODULI_SET_SIZE];
		/*temp1 = (long long)Beta * (long long)(Mj[j]);
		temp1 = temp1 % (long long)modulus[j+MODULI_SET_SIZE];
		temp1 = (long long)Qp[j] - temp1;*/
		Qp[tx] = (modulus[tx+MODULI_SET_SIZE] + (temp1 % modulus[tx+MODULI_SET_SIZE])) % modulus[tx+MODULI_SET_SIZE]; // Source :http://stackoverflow.com/questions/989943/weird-objective-c-mod-behavior
	//}															 // Since temp will sometimes be positive ans sometimes negative,
																 // use the third comment/answer in the source website.


__syncthreads();


	return 0;
}



/*//////////Compute modular inverse///////////////////////////////////////////////
Code from http://en.wikipedia.org/wiki/Modular_multiplicative_inverse
and http://en.wikipedia.org/wiki/Extended_Euclidean_algorithm
*/

__device__ unsigned int d_ModularInverse(unsigned int N, unsigned int D)
{	long long temp_gcd, x, y;

	temp_gcd = d_gcd(N, D, &x, &y);

	if (temp_gcd == 1)
		return ((x+D)%D); // takes care of negative values of x
	else
		return 0;
}


/*//////////Compute gcd///////////////////////////////////////////////
Code from http://en.wikipedia.org/wiki/Modular_multiplicative_inverse
and http://en.wikipedia.org/wiki/Extended_Euclidean_algorithm
*/

__device__ unsigned int d_gcd(unsigned int A, unsigned int B, long long *X, long long *Y)
{
    long long x, y, u, v, m, n, a, b, q, r;

    /* B = A(0) + B(1) */
    x = 0; y = 1;

    /* A = A(1) + B(0) */
    u = 1; v = 0;

    for (a = A, b = B; 0 != a; b = a, a = r, x = u, y = v, u = m, v = n) {
        /* b = aq + r and 0 <= r < a */
        q = b / a;
        r = b % a;

        /* r = Ax + By - aq = Ax + By - (Au + Bv)q = A(x - uq) + B(y - vq) */
        m = x - (u * q);
        n = y - (v * q);
    }

    /* Ax + By = gcd(A, B) */
    *X = x; *Y = y;

    return b;
}



