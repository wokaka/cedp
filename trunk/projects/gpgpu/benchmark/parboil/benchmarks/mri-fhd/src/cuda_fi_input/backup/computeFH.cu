#include "hip/hip_runtime.h"
/*
**************************************************************************

 *cr

 *cr            (C) Copyright 2007 The Board of Trustees of the

 *cr                        University of Illinois

 *cr                         All Rights Reserved

 *cr

 **************************************************************************
*/
#include "gpufi.h" 
/* Adjustable parameters */
struct kValues
{
float Kx;
float Ky;
float Kz;
float RhoPhiR;
float RhoPhiI;
};

__constant__ __device__ struct kValues c[512];
/* & */
/* & */
/* & */
/* & */
void createDataStructs(int numK, int numX, float * realRhoPhi, float * imagRhoPhi, float * outR, float * outI)
{
realRhoPhi=((float * )calloc(numK, sizeof (float)));
imagRhoPhi=((float * )calloc(numK, sizeof (float)));
outR=((float * )calloc(numX, sizeof (float)));
outI=((float * )calloc(numX, sizeof (float)));
}

__global__ void ComputeRhoPhiGPU(int numK, float * phiR, float * phiI, float * dR, float * dI, float * realRhoPhi, float * imagRhoPhi,  struct _gpufi_data_ *gpufi_dev)
{

/* Automatically inserted by Cetus_Cuda */
GPUFI_KERNEL_KERNEL(gpufi_dev, 1);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1001, "A", (int*)&numK);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1002, "", (int*)&phiR);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1002, "", (int*)phiR, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1003, "", (int*)&phiI);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1003, "", (int*)phiI, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1004, "", (int*)&dR);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1004, "", (int*)dR, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1005, "", (int*)&dI);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1005, "", (int*)dI, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1006, "", (int*)&realRhoPhi);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1006, "", (int*)realRhoPhi, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1007, "", (int*)&imagRhoPhi);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1007, "", (int*)imagRhoPhi, 1 /* Array size */, 1 /* Number of faults */);
/* End of Cetus_Cuda code */

int indexK = ((blockIdx.x*512)+threadIdx.x);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1008, "", (int*)&indexK);


if ((indexK<numK))
{
float rPhiR = phiR[indexK];
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1009, "", (int*)&rPhiR);


float rPhiI = phiI[indexK];
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1010, "", (int*)&rPhiI);


float rDR = dR[indexK];
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1011, "", (int*)&rDR);


float rDI = dI[indexK];
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1012, "", (int*)&rDI);


realRhoPhi[indexK]=((rPhiR*rDR)+(rPhiI*rDI));
imagRhoPhi[indexK]=((rPhiR*rDI)-(rPhiI*rDR));
}
}

__global__ void ComputeFH_GPU(int numK, int kGlobalIndex, float * x, float * y, float * z, float * outR, float * outI,  struct _gpufi_data_ *gpufi_dev)
{

/* Automatically inserted by Cetus_Cuda */
GPUFI_KERNEL_KERNEL(gpufi_dev, 2);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1013, "", (int*)&numK);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1014, "", (int*)&kGlobalIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1015, "", (int*)&x);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1015, "", (int*)x, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1016, "", (int*)&y);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1016, "", (int*)y, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1017, "", (int*)&z);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1017, "", (int*)z, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1018, "", (int*)&outR);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1018, "", (int*)outR, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1019, "", (int*)&outI);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1019, "", (int*)outI, 1 /* Array size */, 1 /* Number of faults */);
/* End of Cetus_Cuda code */

float sX;
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1020, "", (int*)&sX);


float sY;
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1021, "", (int*)&sY);


float sZ;
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1022, "", (int*)&sZ);


float sOutR;
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1023, "", (int*)&sOutR);


float sOutI;
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1024, "", (int*)&sOutI);


/* Determine the element of the X arrays computed by this thread */
int xIndex = ((blockIdx.x*256)+threadIdx.x);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1025, "", (int*)&xIndex);


float * test = NULL;
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1026, "", (int*)&test);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1026, "", (int*)test, 1 /* Array size */, 1 /* Number of faults */);


int kIndex = 0;
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1027, "", (int*)&kIndex);


int kCnt;
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1028, "", (int*)&kCnt);


test=( & sOutR);
sX=x[xIndex];
sY=y[xIndex];
sZ=z[xIndex];
sOutR=outR[xIndex];
sOutI=outI[xIndex];
/* Loop over all elements of K in constant mem to compute a partial value */
/* for X. */
kCnt=(numK-kGlobalIndex);
if ((kCnt<512))
{
/* Automatically inserted by Cetus_Cuda */
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1013, "", (int*)&numK);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1014, "", (int*)&kGlobalIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1015, "", (int*)&x);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1015, "", (int*)x, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1016, "", (int*)&y);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1016, "", (int*)y, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1017, "", (int*)&z);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1017, "", (int*)z, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1018, "", (int*)&outR);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1018, "", (int*)outR, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1019, "", (int*)&outI);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1019, "", (int*)outI, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1020, "", (int*)&sX);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1021, "", (int*)&sY);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1022, "", (int*)&sZ);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1023, "", (int*)&sOutR);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1024, "", (int*)&sOutI);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1025, "", (int*)&xIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1026, "", (int*)&test);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1026, "", (int*)test, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1027, "", (int*)&kIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1028, "", (int*)&kCnt);
GPUFI_KERNEL_LOOP(gpufi_dev);
/* End of Cetus_Cuda code */

for (kIndex=0; (kIndex<(kCnt%4))&&(kGlobalIndex<numK); ((kIndex ++ ), (kGlobalIndex ++ )))
{
/* Automatically inserted by Cetus_Cuda */
GPUFI_KERNEL_ITERATION(gpufi_dev);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1013, "", (int*)&numK);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1014, "", (int*)&kGlobalIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1015, "", (int*)&x);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1015, "", (int*)x, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1016, "", (int*)&y);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1016, "", (int*)y, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1017, "", (int*)&z);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1017, "", (int*)z, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1018, "", (int*)&outR);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1018, "", (int*)outR, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1019, "", (int*)&outI);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1019, "", (int*)outI, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1020, "", (int*)&sX);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1021, "", (int*)&sY);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1022, "", (int*)&sZ);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1023, "", (int*)&sOutR);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1024, "", (int*)&sOutI);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1025, "", (int*)&xIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1026, "", (int*)&test);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1026, "", (int*)test, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1027, "", (int*)&kIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1028, "", (int*)&kCnt);
/* End of Cetus_Cuda code */

float expArg = (6.283185307179586*(((c[kIndex].Kx*sX)+(c[kIndex].Ky*sY))+(c[kIndex].Kz*sZ)));
float cosArg = cos(expArg);
float sinArg = sin(expArg);
sOutR+=((c[kIndex].RhoPhiR*cosArg)-(c[kIndex].RhoPhiI*sinArg));
sOutI+=((c[kIndex].RhoPhiI*cosArg)+(c[kIndex].RhoPhiR*sinArg));
}
/* Automatically inserted by Cetus_Cuda */
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1013, "", (int*)&numK);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1014, "", (int*)&kGlobalIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1015, "", (int*)&x);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1015, "", (int*)x, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1016, "", (int*)&y);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1016, "", (int*)y, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1017, "", (int*)&z);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1017, "", (int*)z, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1018, "", (int*)&outR);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1018, "", (int*)outR, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1019, "", (int*)&outI);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1019, "", (int*)outI, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1020, "", (int*)&sX);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1021, "", (int*)&sY);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1022, "", (int*)&sZ);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1023, "", (int*)&sOutR);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1024, "", (int*)&sOutI);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1025, "", (int*)&xIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1026, "", (int*)&test);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1026, "", (int*)test, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1027, "", (int*)&kIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1028, "", (int*)&kCnt);
/* End of Cetus_Cuda code */

}
/* Automatically inserted by Cetus_Cuda */
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1013, "", (int*)&numK);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1014, "", (int*)&kGlobalIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1015, "", (int*)&x);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1015, "", (int*)x, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1016, "", (int*)&y);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1016, "", (int*)y, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1017, "", (int*)&z);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1017, "", (int*)z, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1018, "", (int*)&outR);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1018, "", (int*)outR, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1019, "", (int*)&outI);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1019, "", (int*)outI, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1020, "", (int*)&sX);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1021, "", (int*)&sY);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1022, "", (int*)&sZ);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1023, "", (int*)&sOutR);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1024, "", (int*)&sOutI);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1025, "", (int*)&xIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1026, "", (int*)&test);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1026, "", (int*)test, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1027, "", (int*)&kIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1028, "", (int*)&kCnt);
GPUFI_KERNEL_LOOP(gpufi_dev);
/* End of Cetus_Cuda code */

for (; (kIndex<512)&&(kGlobalIndex<numK); ((kIndex+=4), (kGlobalIndex+=4)))
{
/* Automatically inserted by Cetus_Cuda */
GPUFI_KERNEL_ITERATION(gpufi_dev);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1013, "", (int*)&numK);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1014, "", (int*)&kGlobalIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1015, "", (int*)&x);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1015, "", (int*)x, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1016, "", (int*)&y);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1016, "", (int*)y, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1017, "", (int*)&z);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1017, "", (int*)z, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1018, "", (int*)&outR);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1018, "", (int*)outR, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1019, "", (int*)&outI);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1019, "", (int*)outI, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1020, "", (int*)&sX);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1021, "", (int*)&sY);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1022, "", (int*)&sZ);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1023, "", (int*)&sOutR);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1024, "", (int*)&sOutI);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1025, "", (int*)&xIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1026, "", (int*)&test);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1026, "", (int*)test, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1027, "", (int*)&kIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1028, "", (int*)&kCnt);
/* End of Cetus_Cuda code */

float expArg = (6.283185307179586*(((c[kIndex].Kx*sX)+(c[kIndex].Ky*sY))+(c[kIndex].Kz*sZ)));
float cosArg = cos(expArg);
float sinArg = sin(expArg);
int kIndex1;
float expArg1;
float cosArg1;
float sinArg1;
int kIndex2;
float expArg2;
float cosArg2;
float sinArg2;
int kIndex3;
float expArg3;
float cosArg3;
float sinArg3;
sOutR+=((c[kIndex].RhoPhiR*cosArg)-(c[kIndex].RhoPhiI*sinArg));
sOutI+=((c[kIndex].RhoPhiI*cosArg)+(c[kIndex].RhoPhiR*sinArg));
kIndex1=(kIndex+1);
expArg1=(6.283185307179586*(((c[kIndex1].Kx*sX)+(c[kIndex1].Ky*sY))+(c[kIndex1].Kz*sZ)));
cosArg1=cos(expArg1);
sinArg1=sin(expArg1);
sOutR+=((c[kIndex1].RhoPhiR*cosArg1)-(c[kIndex1].RhoPhiI*sinArg1));
sOutI+=((c[kIndex1].RhoPhiI*cosArg1)+(c[kIndex1].RhoPhiR*sinArg1));
kIndex2=(kIndex+2);
expArg2=(6.283185307179586*(((c[kIndex2].Kx*sX)+(c[kIndex2].Ky*sY))+(c[kIndex2].Kz*sZ)));
cosArg2=cos(expArg2);
sinArg2=sin(expArg2);
sOutR+=((c[kIndex2].RhoPhiR*cosArg2)-(c[kIndex2].RhoPhiI*sinArg2));
sOutI+=((c[kIndex2].RhoPhiI*cosArg2)+(c[kIndex2].RhoPhiR*sinArg2));
kIndex3=(kIndex+3);
expArg3=(6.283185307179586*(((c[kIndex3].Kx*sX)+(c[kIndex3].Ky*sY))+(c[kIndex3].Kz*sZ)));
cosArg3=cos(expArg3);
sinArg3=sin(expArg3);
sOutR+=((c[kIndex3].RhoPhiR*cosArg3)-(c[kIndex3].RhoPhiI*sinArg3));
sOutI+=((c[kIndex3].RhoPhiI*cosArg3)+(c[kIndex3].RhoPhiR*sinArg3));
}
/* Automatically inserted by Cetus_Cuda */
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1013, "", (int*)&numK);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1014, "", (int*)&kGlobalIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1015, "", (int*)&x);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1015, "", (int*)x, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1016, "", (int*)&y);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1016, "", (int*)y, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1017, "", (int*)&z);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1017, "", (int*)z, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1018, "", (int*)&outR);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1018, "", (int*)outR, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1019, "", (int*)&outI);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1019, "", (int*)outI, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1020, "", (int*)&sX);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1021, "", (int*)&sY);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1022, "", (int*)&sZ);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1023, "", (int*)&sOutR);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1024, "", (int*)&sOutI);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1025, "", (int*)&xIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1026, "", (int*)&test);
GPUFI_KERNEL_VARIABLE_STREAM(gpufi_dev, 1026, "", (int*)test, 1 /* Array size */, 1 /* Number of faults */);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1027, "", (int*)&kIndex);
GPUFI_KERNEL_VARIABLE(gpufi_dev, 1028, "", (int*)&kCnt);
/* End of Cetus_Cuda code */

outR[xIndex]=sOutR;
outI[xIndex]=sOutI;
}

void computeRhoPhi_GPU(int numK, float * phiR_d, float * phiI_d, float * dR_d, float * dI_d, float * realRhoPhi_d, float * imagRhoPhi_d)
{
int rhoPhiBlocks = (numK/512);
dim3 DimRhoPhiBlock;
dim3 DimPhoPhiGrid;
if ((numK%512))
{
rhoPhiBlocks ++ ;
}
DimRhoPhiBlock.x=512;
DimRhoPhiBlock.y=1;
DimRhoPhiBlock.z=1;
DimPhoPhiGrid.x=rhoPhiBlocks;
DimPhoPhiGrid.y=1;
DimPhoPhiGrid.z=1;
/* printf("Launch RhoPhi Kernel on GPU: Blocks (%d, %d), Threads Per Block %d\n", */
/*       rhoPhiBlocks, 1, KERNEL_RHO_PHI_THREADS_PER_BLOCK); */
/* Automatically inserted by Cetus_Cuda */
hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
CUDA_ERRCK
hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
CUDA_ERRCK
/* End of Cetus_Cuda code */

ComputeRhoPhiGPU<<<DimRhoPhiGrid, DimRhoPhiBlock>>> (numK, phiR_d, phiI_d, dR_d, dI_d, realRhoPhi_d, imagRhoPhi_d, gpufi_dev);
/* Automatically inserted by Cetus_Cuda */
hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
CUDA_ERRCK
hipFree(gpufi_dev);
CUDA_ERRCK
/* End of Cetus_Cuda code */

}

void computeFH_GPU(int numK, int numX, float * x_d, float * y_d, float * z_d, struct kValues * kVals, float * outR_d, float * outI_d)
{
int FHGrids = (numK/512);
int FHBlocks;
dim3 DimFHBlock;
dim3 DimFHGrid;
int FHGrid;
if ((numK%512))
{
FHGrids ++ ;
}
FHBlocks=(numX/256);
if ((numX%256))
{
FHBlocks ++ ;
}
DimFHBlock.x=256;
DimFHBlock.y=1;
DimFHBlock.z=1;
DimFHGrid.x=FHBlocks;
DimFHGrid.y=1;
DimFHGrid.z=1;
/* printf("Launch GPU Kernel: Grids %d, Blocks Per Grid (%d, %d), Threads Per Block (%d, %d), K Elems Per Thread %d\n", */
/*       FHGrids, DimFHGrid.x, DimFHGrid.y, DimFHBlock.x, DimFHBlock.y, KERNEL_FH_K_ELEMS_PER_GRID); */
for (FHGrid=0; FHGrid<FHGrids; FHGrid ++ )
{
/* Put the tile of K values into constant mem */
int FHGridBase = (FHGrid*512);
kValues * kValsTile = (kVals+FHGridBase);
int numElems = ((512<(numK-FHGridBase)) ? 512 : (numK-FHGridBase));
hipMemcpyToSymbol(HIP_SYMBOL(c), kValsTile, (numElems*sizeof (kValues)), 0);
{
hipError_t err;
if (((err=hipGetLastError())!=hipSuccess))
{
fprintf(stderr, "CUDA error on line %d: %s\n", 203, hipGetErrorString(err));
exit(( - 1));
}
}
;
/* Automatically inserted by Cetus_Cuda */
hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
CUDA_ERRCK
hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
CUDA_ERRCK
/* End of Cetus_Cuda code */

ComputeFH_GPU<<<DimFHGrid, DimFHBlock>>> (numK, FHGridBase, x_d, y_d, z_d, outR_d, outI_d, gpufi_dev);
/* Automatically inserted by Cetus_Cuda */
hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
CUDA_ERRCK
hipFree(gpufi_dev);
CUDA_ERRCK
/* End of Cetus_Cuda code */

{
hipError_t err;
if (((err=hipGetLastError())!=hipSuccess))
{
fprintf(stderr, "CUDA error on line %d: %s\n", 207, hipGetErrorString(err));
exit(( - 1));
}
}
;
}
}

