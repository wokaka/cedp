#include "hip/hip_runtime.h"
/*
**************************************************************************

 *cr

 *cr            (C) Copyright 2007 The Board of Trustees of the

 *cr                        University of Illinois

 *cr                         All Rights Reserved

 *cr

 **************************************************************************
*/
#include "gpufi.h" 
/* Adjustable parameters */
struct kValues
{
float Kx;
float Ky;
float Kz;
float RhoPhiR;
float RhoPhiI;
};

__constant__ __device__ struct kValues c[512];
/* & */
/* & */
/* & */
/* & */
void createDataStructs(int numK, int numX, float * realRhoPhi, float * imagRhoPhi, float * outR, float * outI)
{
realRhoPhi=((float * )calloc(numK, sizeof (float)));
imagRhoPhi=((float * )calloc(numK, sizeof (float)));
outR=((float * )calloc(numX, sizeof (float)));
outI=((float * )calloc(numX, sizeof (float)));
}

__global__ void ComputeRhoPhiGPU(int numK, float * phiR, float * phiI, float * dR, float * dI, float * realRhoPhi, float * imagRhoPhi)
{
int indexK = ((blockIdx.x*512)+threadIdx.x);
if ((indexK<numK))
{
float rPhiR = phiR[indexK];
float rPhiI = phiI[indexK];
float rDR = dR[indexK];
float rDI = dI[indexK];
realRhoPhi[indexK]=((rPhiR*rDR)+(rPhiI*rDI));
imagRhoPhi[indexK]=((rPhiR*rDI)-(rPhiI*rDR));
}
}

__global__ void ComputeFH_GPU(int numK, int kGlobalIndex, float * x, float * y, float * z, float * outR, float * outI)
{
float sX;
float sY;
float sZ;
float sOutR;
float sOutI;
/* Determine the element of the X arrays computed by this thread */
int xIndex = ((blockIdx.x*256)+threadIdx.x);
int kIndex = 0;
int kCnt;
sX=x[xIndex];
sY=y[xIndex];
sZ=z[xIndex];
sOutR=outR[xIndex];
sOutI=outI[xIndex];
/* Loop over all elements of K in constant mem to compute a partial value */
/* for X. */
kCnt=(numK-kGlobalIndex);
if ((kCnt<512))
{
#pragma loop name ComputeFH_GPU#0 
for (kIndex=0; (kIndex<(kCnt%4))&&(kGlobalIndex<numK); ((kIndex ++ ), (kGlobalIndex ++ )))
{
float expArg = (6.283185307179586*(((c[kIndex].Kx*sX)+(c[kIndex].Ky*sY))+(c[kIndex].Kz*sZ)));
float cosArg = cos(expArg);
float sinArg = sin(expArg);
sOutR+=((c[kIndex].RhoPhiR*cosArg)-(c[kIndex].RhoPhiI*sinArg));
sOutI+=((c[kIndex].RhoPhiI*cosArg)+(c[kIndex].RhoPhiR*sinArg));
}
}
#pragma loop name ComputeFH_GPU#1 
for (; (kIndex<512)&&(kGlobalIndex<numK); ((kIndex+=4), (kGlobalIndex+=4)))
{
float expArg = (6.283185307179586*(((c[kIndex].Kx*sX)+(c[kIndex].Ky*sY))+(c[kIndex].Kz*sZ)));
float cosArg = cos(expArg);
float sinArg = sin(expArg);
int kIndex1;
float expArg1;
float cosArg1;
float sinArg1;
int kIndex2;
float expArg2;
float cosArg2;
float sinArg2;
int kIndex3;
float expArg3;
float cosArg3;
float sinArg3;
sOutR+=((c[kIndex].RhoPhiR*cosArg)-(c[kIndex].RhoPhiI*sinArg));
sOutI+=((c[kIndex].RhoPhiI*cosArg)+(c[kIndex].RhoPhiR*sinArg));
kIndex1=(kIndex+1);
expArg1=(6.283185307179586*(((c[kIndex1].Kx*sX)+(c[kIndex1].Ky*sY))+(c[kIndex1].Kz*sZ)));
cosArg1=cos(expArg1);
sinArg1=sin(expArg1);
sOutR+=((c[kIndex1].RhoPhiR*cosArg1)-(c[kIndex1].RhoPhiI*sinArg1));
sOutI+=((c[kIndex1].RhoPhiI*cosArg1)+(c[kIndex1].RhoPhiR*sinArg1));
kIndex2=(kIndex+2);
expArg2=(6.283185307179586*(((c[kIndex2].Kx*sX)+(c[kIndex2].Ky*sY))+(c[kIndex2].Kz*sZ)));
cosArg2=cos(expArg2);
sinArg2=sin(expArg2);
sOutR+=((c[kIndex2].RhoPhiR*cosArg2)-(c[kIndex2].RhoPhiI*sinArg2));
sOutI+=((c[kIndex2].RhoPhiI*cosArg2)+(c[kIndex2].RhoPhiR*sinArg2));
kIndex3=(kIndex+3);
expArg3=(6.283185307179586*(((c[kIndex3].Kx*sX)+(c[kIndex3].Ky*sY))+(c[kIndex3].Kz*sZ)));
cosArg3=cos(expArg3);
sinArg3=sin(expArg3);
sOutR+=((c[kIndex3].RhoPhiR*cosArg3)-(c[kIndex3].RhoPhiI*sinArg3));
sOutI+=((c[kIndex3].RhoPhiI*cosArg3)+(c[kIndex3].RhoPhiR*sinArg3));
}
outR[xIndex]=sOutR;
outI[xIndex]=sOutI;
}

void computeRhoPhi_GPU(int numK, float * phiR_d, float * phiI_d, float * dR_d, float * dI_d, float * realRhoPhi_d, float * imagRhoPhi_d)
{
int rhoPhiBlocks = (numK/512);
dim3 DimRhoPhiBlock;
dim3 DimRhoPhiGrid;
if ((numK%512))
{
rhoPhiBlocks ++ ;
}
DimRhoPhiBlock.x=512;
DimRhoPhiBlock.y=1;
DimRhoPhiBlock.z=1;
DimPhoPhiGrid.x=rhoPhiBlocks;
DimPhoPhiGrid.y=1;
DimPhoPhiGrid.z=1;
/* printf("Launch RhoPhi Kernel on GPU: Blocks (%d, %d), Threads Per Block %d\n", */
/*       rhoPhiBlocks, 1, KERNEL_RHO_PHI_THREADS_PER_BLOCK); */
ComputeRhoPhiGPU<<<DimRhoPhiGrid, DimRhoPhiBlock>>>(numK, phiR_d, phiI_d, dR_d, dI_d, realRhoPhi_d, imagRhoPhi_d);
}

void computeFH_GPU(int numK, int numX, float * x_d, float * y_d, float * z_d, struct kValues * kVals, float * outR_d, float * outI_d)
{
int FHGrids = (numK/512);
int FHBlocks;
dim3 DimFHBlock;
dim3 DimFHGrid;
int FHGrid;
if ((numK%512))
{
FHGrids ++ ;
}
FHBlocks=(numX/256);
if ((numX%256))
{
FHBlocks ++ ;
}
DimFHBlock.x=256;
DimFHBlock.y=1;
DimFHBlock.z=1;
DimFHGrid.x=FHBlocks;
DimFHGrid.y=1;
DimFHGrid.z=1;
/* printf("Launch GPU Kernel: Grids %d, Blocks Per Grid (%d, %d), Threads Per Block (%d, %d), K Elems Per Thread %d\n", */
/*       FHGrids, DimFHGrid.x, DimFHGrid.y, DimFHBlock.x, DimFHBlock.y, KERNEL_FH_K_ELEMS_PER_GRID); */
#pragma loop name computeFH_GPU#0 
for (FHGrid=0; FHGrid<FHGrids; FHGrid ++ )
{
/* Put the tile of K values into constant mem */
int FHGridBase = (FHGrid*512);
kValues * kValsTile = (kVals+FHGridBase);
int numElems = ((512<(numK-FHGridBase)) ? 512 : (numK-FHGridBase));
hipMemcpyToSymbol(HIP_SYMBOL(c), kValsTile, (numElems*sizeof (kValues)), 0);
{
hipError_t err;
if (((err=hipGetLastError())!=hipSuccess))
{
fprintf(stderr, "CUDA error on line %d: %s\n", 201, hipGetErrorString(err));
exit(( - 1));
}
}
;
ComputeFH_GPU<<<DimFHGrid, DimFHBlock>>>(numK, FHGridBase, x_d, y_d, z_d, outR_d, outI_d);
{
hipError_t err;
if (((err=hipGetLastError())!=hipSuccess))
{
fprintf(stderr, "CUDA error on line %d: %s\n", 205, hipGetErrorString(err));
exit(( - 1));
}
}
;
}
}

