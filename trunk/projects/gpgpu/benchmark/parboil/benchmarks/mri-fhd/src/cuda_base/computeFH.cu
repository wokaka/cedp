
#include <hip/hip_runtime.h>
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#define PI   3.1415926535897932384626433832795029
#define PIx2 6.2831853071795864769252867665590058

/* Adjustable parameters */
#define KERNEL_RHO_PHI_THREADS_PER_BLOCK 512
#define KERNEL_FH_THREADS_PER_BLOCK 256
#define KERNEL_FH_K_ELEMS_PER_GRID 512
#define KERNEL_FH_X_ELEMS_PER_THREAD 1

#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))

#define CUDA_ERRCK							\
  {hipError_t err;							\
    if ((err = hipGetLastError()) != hipSuccess) {			\
      fprintf(stderr, "CUDA error on line %d: %s\n", __LINE__, hipGetErrorString(err)); \
      exit(-1);								\
    }									\
  }

struct kValues {
  float Kx;
  float Ky;
  float Kz;
  float RhoPhiR;
  float RhoPhiI;
};

__constant__ __device__ kValues c[KERNEL_FH_K_ELEMS_PER_GRID];

void createDataStructs(int numK, int numX, 
                       float*& outR, float*& outI)
{
  outR = (float*) calloc (numX, sizeof (float));
  outI = (float*) calloc (numX, sizeof (float));
}

__global__ void
ComputeRhoPhiGPU(int numK,
                 float* phiR, float* phiI, 
                 float* dR, float* dI, 
                 float* realRhoPhi, float* imagRhoPhi)
{
  int indexK = blockIdx.x*KERNEL_RHO_PHI_THREADS_PER_BLOCK + threadIdx.x;
  if (indexK < numK) {
    float rPhiR = phiR[indexK];
    float rPhiI = phiI[indexK];
    float rDR = dR[indexK];
    float rDI = dI[indexK];
    realRhoPhi[indexK] = rPhiR * rDR + rPhiI * rDI;
    imagRhoPhi[indexK] = rPhiR * rDI - rPhiI * rDR;
  }
}

__global__ void
ComputeFH_GPU(int numK, int kGlobalIndex,
              float* cKx, float* cKy, float* cKz,
              float* x, float* y, float* z,
              float* cRhoPhiR, float* cRhoPhiI,
              float* outR, float* outI) {
  float sX;
  float sY;
  float sZ;
  float sOutR;
  float sOutI;

  // Determine the element of the X arrays computed by this thread
  int xIndex = blockIdx.x*KERNEL_FH_THREADS_PER_BLOCK + threadIdx.x;

  // Read block's X values from global mem to shared mem
  sX = x[xIndex];
  sY = y[xIndex];
  sZ = z[xIndex];
  sOutR = outR[xIndex];
  sOutI = outI[xIndex];

  // Loop over all elements of K in constant mem to compute a partial value
  // for X.
  for (int kIndex = 0;
       (kIndex < KERNEL_FH_K_ELEMS_PER_GRID) && (kGlobalIndex < numK);
       kIndex++, kGlobalIndex++) {
    float exponentArg = PIx2 * (cKx[kIndex] * sX +
				cKy[kIndex] * sY +
				cKz[kIndex] * sZ);

    float cosArg = cos(exponentArg);
    float sinArg = sin(exponentArg);

    sOutR += cRhoPhiR[kIndex] * cosArg - cRhoPhiI[kIndex] * sinArg;
    sOutI += cRhoPhiI[kIndex] * cosArg + cRhoPhiR[kIndex] * sinArg;
  }

 outR[xIndex] = sOutR;
 outI[xIndex] = sOutI;
}

void computeRhoPhi_GPU(int numK, 
                       float* phiR_d, float* phiI_d, float* dR_d, float* dI_d,
                       float* realRhoPhi_d, float* imagRhoPhi_d)
{
  int rhoPhiBlocks = numK / KERNEL_RHO_PHI_THREADS_PER_BLOCK;

  if (numK % KERNEL_RHO_PHI_THREADS_PER_BLOCK)
    rhoPhiBlocks++;

  dim3 DimRhoPhiBlock(KERNEL_RHO_PHI_THREADS_PER_BLOCK, 1);
  dim3 DimRhoPhiGrid(rhoPhiBlocks, 1);

  ComputeRhoPhiGPU <<< DimRhoPhiGrid, DimRhoPhiBlock >>> 
      (numK, phiR_d, phiI_d, dR_d, dI_d, realRhoPhi_d, imagRhoPhi_d);
}

void computeFH_GPU(int numK, int numX, 
                   float* kx_d, float* ky_d, float* kz_d,
                   float* realRhoPhi_d, float* imagRhoPhi_d,
                   float* x_d, float* y_d, float* z_d,
                   float* outR_d, float* outI_d)
{
  int FHGrids = numK / KERNEL_FH_K_ELEMS_PER_GRID;
  if (numK % KERNEL_FH_K_ELEMS_PER_GRID)
    FHGrids++;
  int FHBlocks = numX / KERNEL_FH_THREADS_PER_BLOCK;
  if (numX % KERNEL_FH_THREADS_PER_BLOCK)
    FHBlocks++;
  dim3 DimFHBlock(KERNEL_FH_THREADS_PER_BLOCK, 1);
  dim3 DimFHGrid(FHBlocks, 1);

  for (int FHGrid = 0; FHGrid < FHGrids; FHGrid++) {
    int FHGridBase = FHGrid * KERNEL_FH_K_ELEMS_PER_GRID;
    float *rhoPhiRTile = realRhoPhi_d + FHGridBase;
    float *rhoPhiITile = imagRhoPhi_d + FHGridBase;
    float *kxTile = kx_d + FHGridBase;
    float *kyTile = ky_d + FHGridBase;
    float *kzTile = kz_d + FHGridBase;

    ComputeFH_GPU <<< DimFHGrid, DimFHBlock >>>
      (numK, FHGridBase, kxTile, kyTile, kzTile, x_d, y_d, z_d,
       rhoPhiRTile, rhoPhiITile, outR_d, outI_d);
    CUDA_ERRCK;
  }
}
