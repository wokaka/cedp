#include "hip/hip_runtime.h"
/* Instrumented by FaultInjector */
#include "gpufi.h"

/*
**************************************************************************
 
 *cr
 
 *cr            (C) Copyright 2007 The Board of Trustees of the
 
 *cr                        University of Illinois
 
 *cr                         All Rights Reserved
 
 *cr
 
 **************************************************************************
*/
#include "gpufi.h"

/* Adjustable parameters */
struct kValues
{
    float Kx;
    float Ky;
    float Kz;
    float RhoPhiR;
    float RhoPhiI;
};

__constant__ __device__ struct kValues c[512];
/* & */
/* & */
/* & */
/* & */
void createDataStructs(int numK, int numX, float *  * realRhoPhi, float *  * imagRhoPhi, float *  * outR, float *  * outI)
{
    ( * realRhoPhi)=((float * )calloc(numK, sizeof (float)));
    ( * imagRhoPhi)=((float * )calloc(numK, sizeof (float)));
    ( * outR)=((float * )calloc(numX, sizeof (float)));
    ( * outI)=((float * )calloc(numX, sizeof (float)));
}

__global__ void ComputeRhoPhiGPU(int numK, float * phiR, float * phiI, float * dR, float * dI, float * realRhoPhi, float * imagRhoPhi,  struct _gpufi_data_ *gpufi_dev)
{
    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_BEGIN, 0, "ComputeRhoPhiGPU");
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 0, "numK", (int*)&numK, GPUFI_DATATYPE_INTEGER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 1, "phiR", (int*)&phiR, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 2, "phiI", (int*)&phiI, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 3, "dR", (int*)&dR, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 4, "dI", (int*)&dI, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 5, "realRhoPhi", (int*)&realRhoPhi, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 6, "imagRhoPhi", (int*)&imagRhoPhi, GPUFI_DATATYPE_FLOAT_POINTER);


    int indexK = ((blockIdx.x*512)+threadIdx.x);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 14, "indexK", (int*)&indexK, GPUFI_DATATYPE_INTEGER);


    if ((indexK<numK))
    {
        float rPhiR = phiR[indexK];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 15, "rPhiR", (int*)&rPhiR, GPUFI_DATATYPE_FLOAT);


        float rPhiI = phiI[indexK];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 16, "rPhiI", (int*)&rPhiI, GPUFI_DATATYPE_FLOAT);


        float rDR = dR[indexK];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 17, "rDR", (int*)&rDR, GPUFI_DATATYPE_FLOAT);


        float rDI = dI[indexK];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 18, "rDI", (int*)&rDI, GPUFI_DATATYPE_FLOAT);


        realRhoPhi[indexK]=((rPhiR*rDR)+(rPhiI*rDI));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 19, "realRhoPhi[indexK]", (int*)&realRhoPhi[indexK], GPUFI_DATATYPE_FLOAT);


        imagRhoPhi[indexK]=((rPhiR*rDI)-(rPhiI*rDR));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 20, "imagRhoPhi[indexK]", (int*)&imagRhoPhi[indexK], GPUFI_DATATYPE_FLOAT);


    }
    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_END, 0, "ComputeRhoPhiGPU");
}

__global__ void ComputeFH_GPU(int numK, int kGlobalIndex, float * x, float * y, float * z, float * outR, float * outI,  struct _gpufi_data_ *gpufi_dev)
{
    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_BEGIN, 1, "ComputeFH_GPU");
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 10, "y", (int*)&y, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 11, "z", (int*)&z, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 12, "outR", (int*)&outR, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 13, "outI", (int*)&outI, GPUFI_DATATYPE_FLOAT_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 7, "numK", (int*)&numK, GPUFI_DATATYPE_INTEGER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 8, "kGlobalIndex", (int*)&kGlobalIndex, GPUFI_DATATYPE_INTEGER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 9, "x", (int*)&x, GPUFI_DATATYPE_FLOAT_POINTER);


    float sX;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 21, "sX", (int*)&sX, GPUFI_DATATYPE_FLOAT);


    float sY;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 22, "sY", (int*)&sY, GPUFI_DATATYPE_FLOAT);


    float sZ;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 23, "sZ", (int*)&sZ, GPUFI_DATATYPE_FLOAT);


    float sOutR;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 24, "sOutR", (int*)&sOutR, GPUFI_DATATYPE_FLOAT);


    float sOutI;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 25, "sOutI", (int*)&sOutI, GPUFI_DATATYPE_FLOAT);


    /* Determine the element of the X arrays computed by this thread
     */
    int xIndex = ((blockIdx.x*256)+threadIdx.x);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 26, "xIndex", (int*)&xIndex, GPUFI_DATATYPE_INTEGER);


    float * test = NULL;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 27, "test", (int*)&test, GPUFI_DATATYPE_FLOAT_POINTER);


    int kIndex = 0;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 28, "kIndex", (int*)&kIndex, GPUFI_DATATYPE_INTEGER);


    int kCnt;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 29, "kCnt", (int*)&kCnt, GPUFI_DATATYPE_INTEGER);


    test=( & sOutR);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 30, "test", (int*)&test, GPUFI_DATATYPE_FLOAT_POINTER);


    sX=x[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 31, "sX", (int*)&sX, GPUFI_DATATYPE_FLOAT);


    sY=y[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 32, "sY", (int*)&sY, GPUFI_DATATYPE_FLOAT);


    sZ=z[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 33, "sZ", (int*)&sZ, GPUFI_DATATYPE_FLOAT);


    sOutR=outR[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 34, "sOutR", (int*)&sOutR, GPUFI_DATATYPE_FLOAT);


    sOutI=outI[xIndex];
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 35, "sOutI", (int*)&sOutI, GPUFI_DATATYPE_FLOAT);


    /* Loop over all elements of K in constant mem to compute a partial value
     */
    /* for X.
     */
    kCnt=(numK-kGlobalIndex);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 36, "kCnt", (int*)&kCnt, GPUFI_DATATYPE_INTEGER);


    if ((kCnt<512))
    {
        /* Fault Injector Instrumentation Begin */
        GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_BEGIN);
        /* Fault Injector Instrumentation End */

        for (kIndex=0; (kIndex<(kCnt%4))&&(kGlobalIndex<numK); ((kIndex ++ ), (kGlobalIndex ++ )))
        {
            /* Fault Injector Instrumentation Begin */
            GPUFI_KERNEL_ITERATION(gpufi_dev);
            /* Fault Injector Instrumentation End */

            float expArg = (6.283185307179586*(((c[kIndex].Kx*sX)+(c[kIndex].Ky*sY))+(c[kIndex].Kz*sZ)));
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 40, "expArg", (int*)&expArg, GPUFI_DATATYPE_FLOAT);


            float cosArg = cos(expArg);
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 41, "cosArg", (int*)&cosArg, GPUFI_DATATYPE_FLOAT);


            float sinArg = sin(expArg);
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 42, "sinArg", (int*)&sinArg, GPUFI_DATATYPE_FLOAT);


            sOutR+=((c[kIndex].RhoPhiR*cosArg)-(c[kIndex].RhoPhiI*sinArg));
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 43, "sOutR", (int*)&sOutR, GPUFI_DATATYPE_FLOAT);


            sOutI+=((c[kIndex].RhoPhiI*cosArg)+(c[kIndex].RhoPhiR*sinArg));
            GPUFI_KERNEL_VARIABLE(gpufi_dev, 44, "sOutI", (int*)&sOutI, GPUFI_DATATYPE_FLOAT);


        }
        /* Automatically inserted by Cetus_Cuda */
        GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_END);
        /* Fault Injector Instrumentation End *//* End of Cetus_Cuda code */

        GPUFI_KERNEL_VARIABLE(gpufi_dev, 38, "kIndex", (int*)&kIndex, GPUFI_DATATYPE_INTEGER);


        GPUFI_KERNEL_VARIABLE(gpufi_dev, 39, "kGlobalIndex", (int*)&kGlobalIndex, GPUFI_DATATYPE_INTEGER);


    }
    /* Fault Injector Instrumentation Begin */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_BEGIN);
    /* Fault Injector Instrumentation End */

    for (; (kIndex<512)&&(kGlobalIndex<numK); ((kIndex+=4), (kGlobalIndex+=4)))
    {
        /* Fault Injector Instrumentation Begin */
        GPUFI_KERNEL_ITERATION(gpufi_dev);
        /* Fault Injector Instrumentation End */

        float expArg = (6.283185307179586*(((c[kIndex].Kx*sX)+(c[kIndex].Ky*sY))+(c[kIndex].Kz*sZ)));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 47, "expArg", (int*)&expArg, GPUFI_DATATYPE_FLOAT);


        float cosArg = cos(expArg);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 48, "cosArg", (int*)&cosArg, GPUFI_DATATYPE_FLOAT);


        float sinArg = sin(expArg);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 49, "sinArg", (int*)&sinArg, GPUFI_DATATYPE_FLOAT);


        int kIndex1;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 50, "kIndex1", (int*)&kIndex1, GPUFI_DATATYPE_INTEGER);


        float expArg1;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 51, "expArg1", (int*)&expArg1, GPUFI_DATATYPE_FLOAT);


        float cosArg1;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 52, "cosArg1", (int*)&cosArg1, GPUFI_DATATYPE_FLOAT);


        float sinArg1;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 53, "sinArg1", (int*)&sinArg1, GPUFI_DATATYPE_FLOAT);


        int kIndex2;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 54, "kIndex2", (int*)&kIndex2, GPUFI_DATATYPE_INTEGER);


        float expArg2;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 55, "expArg2", (int*)&expArg2, GPUFI_DATATYPE_FLOAT);


        float cosArg2;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 56, "cosArg2", (int*)&cosArg2, GPUFI_DATATYPE_FLOAT);


        float sinArg2;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 57, "sinArg2", (int*)&sinArg2, GPUFI_DATATYPE_FLOAT);


        int kIndex3;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 58, "kIndex3", (int*)&kIndex3, GPUFI_DATATYPE_INTEGER);


        float expArg3;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 59, "expArg3", (int*)&expArg3, GPUFI_DATATYPE_FLOAT);


        float cosArg3;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 60, "cosArg3", (int*)&cosArg3, GPUFI_DATATYPE_FLOAT);


        float sinArg3;
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 61, "sinArg3", (int*)&sinArg3, GPUFI_DATATYPE_FLOAT);


        sOutR+=((c[kIndex].RhoPhiR*cosArg)-(c[kIndex].RhoPhiI*sinArg));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 62, "sOutR", (int*)&sOutR, GPUFI_DATATYPE_FLOAT);


        sOutI+=((c[kIndex].RhoPhiI*cosArg)+(c[kIndex].RhoPhiR*sinArg));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 63, "sOutI", (int*)&sOutI, GPUFI_DATATYPE_FLOAT);


        kIndex1=(kIndex+1);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 64, "kIndex1", (int*)&kIndex1, GPUFI_DATATYPE_INTEGER);


        expArg1=(6.283185307179586*(((c[kIndex1].Kx*sX)+(c[kIndex1].Ky*sY))+(c[kIndex1].Kz*sZ)));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 65, "expArg1", (int*)&expArg1, GPUFI_DATATYPE_FLOAT);


        cosArg1=cos(expArg1);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 66, "cosArg1", (int*)&cosArg1, GPUFI_DATATYPE_FLOAT);


        sinArg1=sin(expArg1);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 67, "sinArg1", (int*)&sinArg1, GPUFI_DATATYPE_FLOAT);


        sOutR+=((c[kIndex1].RhoPhiR*cosArg1)-(c[kIndex1].RhoPhiI*sinArg1));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 68, "sOutR", (int*)&sOutR, GPUFI_DATATYPE_FLOAT);


        sOutI+=((c[kIndex1].RhoPhiI*cosArg1)+(c[kIndex1].RhoPhiR*sinArg1));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 69, "sOutI", (int*)&sOutI, GPUFI_DATATYPE_FLOAT);


        kIndex2=(kIndex+2);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 70, "kIndex2", (int*)&kIndex2, GPUFI_DATATYPE_INTEGER);


        expArg2=(6.283185307179586*(((c[kIndex2].Kx*sX)+(c[kIndex2].Ky*sY))+(c[kIndex2].Kz*sZ)));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 71, "expArg2", (int*)&expArg2, GPUFI_DATATYPE_FLOAT);


        cosArg2=cos(expArg2);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 72, "cosArg2", (int*)&cosArg2, GPUFI_DATATYPE_FLOAT);


        sinArg2=sin(expArg2);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 73, "sinArg2", (int*)&sinArg2, GPUFI_DATATYPE_FLOAT);


        sOutR+=((c[kIndex2].RhoPhiR*cosArg2)-(c[kIndex2].RhoPhiI*sinArg2));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 74, "sOutR", (int*)&sOutR, GPUFI_DATATYPE_FLOAT);


        sOutI+=((c[kIndex2].RhoPhiI*cosArg2)+(c[kIndex2].RhoPhiR*sinArg2));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 75, "sOutI", (int*)&sOutI, GPUFI_DATATYPE_FLOAT);


        kIndex3=(kIndex+3);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 76, "kIndex3", (int*)&kIndex3, GPUFI_DATATYPE_INTEGER);


        expArg3=(6.283185307179586*(((c[kIndex3].Kx*sX)+(c[kIndex3].Ky*sY))+(c[kIndex3].Kz*sZ)));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 77, "expArg3", (int*)&expArg3, GPUFI_DATATYPE_FLOAT);


        cosArg3=cos(expArg3);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 78, "cosArg3", (int*)&cosArg3, GPUFI_DATATYPE_FLOAT);


        sinArg3=sin(expArg3);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 79, "sinArg3", (int*)&sinArg3, GPUFI_DATATYPE_FLOAT);


        sOutR+=((c[kIndex3].RhoPhiR*cosArg3)-(c[kIndex3].RhoPhiI*sinArg3));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 80, "sOutR", (int*)&sOutR, GPUFI_DATATYPE_FLOAT);


        sOutI+=((c[kIndex3].RhoPhiI*cosArg3)+(c[kIndex3].RhoPhiR*sinArg3));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 81, "sOutI", (int*)&sOutI, GPUFI_DATATYPE_FLOAT);


    }
    /* Automatically inserted by Cetus_Cuda */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_END);
    /* Fault Injector Instrumentation End *//* End of Cetus_Cuda code */

    GPUFI_KERNEL_VARIABLE(gpufi_dev, 45, "kIndex", (int*)&kIndex, GPUFI_DATATYPE_INTEGER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 46, "kGlobalIndex", (int*)&kGlobalIndex, GPUFI_DATATYPE_INTEGER);


    outR[xIndex]=sOutR;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 82, "outR[xIndex]", (int*)&outR[xIndex], GPUFI_DATATYPE_FLOAT);


    outI[xIndex]=sOutI;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 83, "outI[xIndex]", (int*)&outI[xIndex], GPUFI_DATATYPE_FLOAT);


    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_END, 1, "ComputeFH_GPU");
}

void computeRhoPhi_GPU(int numK, float * phiR_d, float * phiI_d, float * dR_d, float * dI_d, float * realRhoPhi_d, float * imagRhoPhi_d)
{
    int rhoPhiBlocks = (numK/512);
    dim3 DimRhoPhiBlock;
    dim3 DimRhoPhiGrid;
    if ((numK%512))
    {
        rhoPhiBlocks ++ ;
    }
    DimRhoPhiBlock.x=512;
    DimRhoPhiBlock.y=1;
    DimRhoPhiBlock.z=1;
    DimRhoPhiGrid.x=rhoPhiBlocks;
    DimRhoPhiGrid.y=1;
    DimRhoPhiGrid.z=1;
    /* printf("Launch RhoPhi Kernel on GPU: Blocks (%d, %d), Threads Per Block %d\n",
     */
    /*       rhoPhiBlocks, 1, KERNEL_RHO_PHI_THREADS_PER_BLOCK);
     */
    /* Fault Injection Instrumentation Begin */
    hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
    CUDA_ERRCK
    hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
    CUDA_ERRCK
    /* Fault Injection Instrumentation End */

    ComputeRhoPhiGPU<<<DimRhoPhiGrid, DimRhoPhiBlock>>> (numK, phiR_d, phiI_d, dR_d, dI_d, realRhoPhi_d, imagRhoPhi_d, gpufi_dev);
    /* Fault Injection Instrumentation Begin */
    hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
    CUDA_ERRCK
    hipFree(gpufi_dev);
    CUDA_ERRCK
    /* Fault Injection Instrumentation End */

}

void computeFH_GPU(int numK, int numX, float * x_d, float * y_d, float * z_d, struct kValues * kVals, float * outR_d, float * outI_d)
{
    int FHGrids = (numK/512);
    int FHBlocks;
    dim3 DimFHBlock;
    dim3 DimFHGrid;
    int FHGrid;
    if ((numK%512))
    {
        FHGrids ++ ;
    }
    FHBlocks=(numX/256);
    if ((numX%256))
    {
        FHBlocks ++ ;
    }
    DimFHBlock.x=256;
    DimFHBlock.y=1;
    DimFHBlock.z=1;
    DimFHGrid.x=FHBlocks;
    DimFHGrid.y=1;
    DimFHGrid.z=1;
    /* printf("Launch GPU Kernel: Grids %d, Blocks Per Grid (%d, %d), Threads Per Block (%d, %d), K Elems Per Thread %d\n",
     */
    /*       FHGrids, DimFHGrid.x, DimFHGrid.y, DimFHBlock.x, DimFHBlock.y, KERNEL_FH_K_ELEMS_PER_GRID);
     */
    for (FHGrid=0; FHGrid<FHGrids; FHGrid ++ )
    {
        /* Put the tile of K values into constant mem
         */
        int FHGridBase = (FHGrid*512);
        kValues * kValsTile = (kVals+FHGridBase);
        int numElems = ((512<(numK-FHGridBase)) ? 512 : (numK-FHGridBase));
        hipMemcpyToSymbol(HIP_SYMBOL(c), kValsTile, (numElems*sizeof (kValues)), 0);
        {
            hipError_t err;
            if (((err=hipGetLastError())!=hipSuccess))
            {
                fprintf(stderr, "CUDA error on line %d: %s\n", 202, hipGetErrorString(err));
                exit(( - 1));
            }
        }
        ;
        /* Fault Injection Instrumentation Begin */
        hipMalloc((void**)&gpufi_dev, sizeof(struct _gpufi_data_));
        CUDA_ERRCK
        hipMemcpy(gpufi_dev, &gpufi_host, sizeof(struct _gpufi_data_), hipMemcpyHostToDevice);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

        ComputeFH_GPU<<<DimFHGrid, DimFHBlock>>> (numK, FHGridBase, x_d, y_d, z_d, outR_d, outI_d, gpufi_dev);
        /* Fault Injection Instrumentation Begin */
        hipMemcpy(&gpufi_host, gpufi_dev, sizeof(struct _gpufi_data_), hipMemcpyDeviceToHost);
        CUDA_ERRCK
        hipFree(gpufi_dev);
        CUDA_ERRCK
        /* Fault Injection Instrumentation End */

        {
            hipError_t err;
            if (((err=hipGetLastError())!=hipSuccess))
        {
            fprintf(stderr, "CUDA error on line %d: %s\n", 206, hipGetErrorString(err));
                exit(( - 1));
            }
        }
        ;
    }
}
