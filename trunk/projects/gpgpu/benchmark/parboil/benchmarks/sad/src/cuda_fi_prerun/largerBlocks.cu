#include "hip/hip_runtime.h"
/* Instrumented by FaultInjector */
#include "gpufi.h"

/*
**************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 **************************************************************************
*/
#include "sad.h"

#include "largerBlocks.h"

/* typedef */
struct uhvec
{
    unsigned short x;
    unsigned short y;
};

/* __align__(4) */
typedef unsigned int uint;
__global__ void larger_sad_calc_8(unsigned short * blk_sad, int mb_width, int mb_height,  struct _gpufi_data_ *gpufi_dev)
{
    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_BEGIN, 0, "larger_sad_calc_8");
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 0, "blk_sad", (int*)&blk_sad, GPUFI_DATATYPE_UNKNOWN_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 1, "mb_width", (int*)&mb_width, GPUFI_DATATYPE_INTEGER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 2, "mb_height", (int*)&mb_height, GPUFI_DATATYPE_INTEGER);


    int tx = (threadIdx.y&1);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 6, "tx", (int*)&tx, GPUFI_DATATYPE_INTEGER);


    int ty = (threadIdx.y>>1);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 7, "ty", (int*)&ty, GPUFI_DATATYPE_INTEGER);


    /* Macroblock and sub-block coordinates */
    int mb_x = blockIdx.x;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 8, "mb_x", (int*)&mb_x, GPUFI_DATATYPE_INTEGER);


    int mb_y = blockIdx.y;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 9, "mb_y", (int*)&mb_y, GPUFI_DATATYPE_INTEGER);


    /* Number of macroblocks in a frame */
    int macroblocks = __mul24(mb_width, mb_height);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 10, "macroblocks", (int*)&macroblocks, GPUFI_DATATYPE_INTEGER);


    int macroblock_index = ((__mul24(mb_y, mb_width)+mb_x)*1096);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 11, "macroblock_index", (int*)&macroblock_index, GPUFI_DATATYPE_INTEGER);


    int search_pos;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 12, "search_pos", (int*)&search_pos, GPUFI_DATATYPE_INTEGER);


    unsigned short * bi;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 13, "bi", (int*)&bi, GPUFI_DATATYPE_UNKNOWN_POINTER);


    unsigned short * bo_6, * bo_5, * bo_4;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 14, "bo_6", (int*)&bo_6, GPUFI_DATATYPE_UNKNOWN_POINTER);


    bi=((blk_sad+((__mul24(macroblocks, 25)+((ty*8)+(tx*2)))*1096))+(macroblock_index*16));
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 15, "bi", (int*)&bi, GPUFI_DATATYPE_UNKNOWN_POINTER);


    /* Block type 6: 4x8
     */
    bo_6=((blk_sad+((((macroblocks<<4)+macroblocks)+((ty*4)+(tx*2)))*1096))+(macroblock_index*8));
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 16, "bo_6", (int*)&bo_6, GPUFI_DATATYPE_UNKNOWN_POINTER);


    /* always true, but improves register allocation
     */
    if ((ty<100))
    {
        /* Block type 5: 8x4
         */
        bo_5=((blk_sad+((((macroblocks<<3)+macroblocks)+((ty*4)+tx))*1096))+(macroblock_index*8));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 17, "bo_5", (int*)&bo_5, GPUFI_DATATYPE_UNKNOWN_POINTER);


        /* Block type 4: 8x8
         */
        bo_4=((blk_sad+((((macroblocks<<2)+macroblocks)+((ty*2)+tx))*1096))+(macroblock_index*4));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 18, "bo_4", (int*)&bo_4, GPUFI_DATATYPE_UNKNOWN_POINTER);


    }
    /* Fault Injector Instrumentation Begin */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_BEGIN);
    /* Fault Injector Instrumentation End */

    for (search_pos=threadIdx.x; search_pos<((1089+1)/2); search_pos+=32)
    {
        /*
        Each uint is actually two 2-byte integers packed together.
               * Only addition is used and there is no chance of integer overflow
               * so this can be done to reduce computation time.
        */
        /* Fault Injector Instrumentation Begin */
        GPUFI_KERNEL_ITERATION(gpufi_dev);
        /* Fault Injector Instrumentation End */

        uint i00 = ((uint * )bi)[search_pos];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 21, "i00", (int*)&i00, GPUFI_DATATYPE_UNKNOWN);


        uint i01 = ((uint * )bi)[(search_pos+(1096/2))];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 22, "i01", (int*)&i01, GPUFI_DATATYPE_UNKNOWN);


        uint i10 = ((uint * )bi)[(search_pos+((4*1096)/2))];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 23, "i10", (int*)&i10, GPUFI_DATATYPE_UNKNOWN);


        uint i11 = ((uint * )bi)[(search_pos+((5*1096)/2))];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 24, "i11", (int*)&i11, GPUFI_DATATYPE_UNKNOWN);


        ((uint * )bo_6)[search_pos]=(i00+i10);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 25, "((uint * )bo_6)[search_pos]", (int*)&((uint * )bo_6)[search_pos], GPUFI_DATATYPE_UNKNOWN);


        ((uint * )bo_6)[(search_pos+(1096/2))]=(i01+i11);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 26, "((uint * )bo_6)[(search_pos+(1096/2))]", (int*)&((uint * )bo_6)[(search_pos+(1096/2))], GPUFI_DATATYPE_UNKNOWN);


        ((uint * )bo_5)[search_pos]=(i00+i01);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 27, "((uint * )bo_5)[search_pos]", (int*)&((uint * )bo_5)[search_pos], GPUFI_DATATYPE_UNKNOWN);


        ((uint * )bo_5)[(search_pos+((2*1096)/2))]=(i10+i11);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 28, "((uint * )bo_5)[(search_pos+((2*1096)/2))]", (int*)&((uint * )bo_5)[(search_pos+((2*1096)/2))], GPUFI_DATATYPE_UNKNOWN);


        ((uint * )bo_4)[search_pos]=((i00+i01)+(i10+i11));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 29, "((uint * )bo_4)[search_pos]", (int*)&((uint * )bo_4)[search_pos], GPUFI_DATATYPE_UNKNOWN);


    }
    /* Automatically inserted by Cetus_Cuda */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_END);
    /* Fault Injector Instrumentation End *//* End of Cetus_Cuda code */

    GPUFI_KERNEL_VARIABLE(gpufi_dev, 20, "search_pos", (int*)&search_pos, GPUFI_DATATYPE_INTEGER);


    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_END, 0, "larger_sad_calc_8");
}

__global__ void larger_sad_calc_16(unsigned short * blk_sad, int mb_width, int mb_height,  struct _gpufi_data_ *gpufi_dev)
{
    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_BEGIN, 1, "larger_sad_calc_16");
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 3, "blk_sad", (int*)&blk_sad, GPUFI_DATATYPE_UNKNOWN_POINTER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 4, "mb_width", (int*)&mb_width, GPUFI_DATATYPE_INTEGER);


    GPUFI_KERNEL_VARIABLE(gpufi_dev, 5, "mb_height", (int*)&mb_height, GPUFI_DATATYPE_INTEGER);


    /* Macroblock coordinates */
    int mb_x = blockIdx.x;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 30, "mb_x", (int*)&mb_x, GPUFI_DATATYPE_INTEGER);


    int mb_y = blockIdx.y;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 31, "mb_y", (int*)&mb_y, GPUFI_DATATYPE_INTEGER);


    /* Number of macroblocks in a frame */
    int macroblocks = (__mul24(mb_width, mb_height)*1096);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 32, "macroblocks", (int*)&macroblocks, GPUFI_DATATYPE_INTEGER);


    int macroblock_index = ((__mul24(mb_y, mb_width)+mb_x)*1096);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 33, "macroblock_index", (int*)&macroblock_index, GPUFI_DATATYPE_INTEGER);


    int search_pos;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 34, "search_pos", (int*)&search_pos, GPUFI_DATATYPE_INTEGER);


    unsigned short * bi;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 35, "bi", (int*)&bi, GPUFI_DATATYPE_UNKNOWN_POINTER);


    unsigned short * bo_3, * bo_2, * bo_1;
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 36, "bo_3", (int*)&bo_3, GPUFI_DATATYPE_UNKNOWN_POINTER);


    /* bi = blk_sad + macroblocks * 5 + macroblock_index * 4;
     */
    bi=((blk_sad+((macroblocks+macroblock_index)<<2))+macroblocks);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 37, "bi", (int*)&bi, GPUFI_DATATYPE_UNKNOWN_POINTER);


    /* Block type 3: 8x16
     */
    /* bo_3 = blk_sad + macroblocks * 3 + macroblock_index * 2;
     */
    bo_3=((blk_sad+((macroblocks+macroblock_index)<<1))+macroblocks);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 38, "bo_3", (int*)&bo_3, GPUFI_DATATYPE_UNKNOWN_POINTER);


    /* Block type 5: 8x4
     */
    bo_2=((blk_sad+macroblocks)+(macroblock_index*2));
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 39, "bo_2", (int*)&bo_2, GPUFI_DATATYPE_UNKNOWN_POINTER);


    /* Block type 4: 8x8
     */
    bo_1=(blk_sad+macroblock_index);
    GPUFI_KERNEL_VARIABLE(gpufi_dev, 40, "bo_1", (int*)&bo_1, GPUFI_DATATYPE_UNKNOWN_POINTER);


    /* Fault Injector Instrumentation Begin */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_BEGIN);
    /* Fault Injector Instrumentation End */

    for (search_pos=threadIdx.x; search_pos<((1089+1)/2); search_pos+=32)
    {
        /*
        Each uint is actually two 2-byte integers packed together.
               * Only addition is used and there is no chance of integer overflow
               * so this can be done to reduce computation time.
        */
        /* Fault Injector Instrumentation Begin */
        GPUFI_KERNEL_ITERATION(gpufi_dev);
        /* Fault Injector Instrumentation End */

        uint i00 = ((uint * )bi)[search_pos];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 43, "i00", (int*)&i00, GPUFI_DATATYPE_UNKNOWN);


        uint i01 = ((uint * )bi)[(search_pos+(1096/2))];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 44, "i01", (int*)&i01, GPUFI_DATATYPE_UNKNOWN);


        uint i10 = ((uint * )bi)[(search_pos+((2*1096)/2))];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 45, "i10", (int*)&i10, GPUFI_DATATYPE_UNKNOWN);


        uint i11 = ((uint * )bi)[(search_pos+((3*1096)/2))];
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 46, "i11", (int*)&i11, GPUFI_DATATYPE_UNKNOWN);


        ((uint * )bo_3)[search_pos]=(i00+i10);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 47, "((uint * )bo_3)[search_pos]", (int*)&((uint * )bo_3)[search_pos], GPUFI_DATATYPE_UNKNOWN);


        ((uint * )bo_3)[(search_pos+(1096/2))]=(i01+i11);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 48, "((uint * )bo_3)[(search_pos+(1096/2))]", (int*)&((uint * )bo_3)[(search_pos+(1096/2))], GPUFI_DATATYPE_UNKNOWN);


        ((uint * )bo_2)[search_pos]=(i00+i01);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 49, "((uint * )bo_2)[search_pos]", (int*)&((uint * )bo_2)[search_pos], GPUFI_DATATYPE_UNKNOWN);


        ((uint * )bo_2)[(search_pos+(1096/2))]=(i10+i11);
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 50, "((uint * )bo_2)[(search_pos+(1096/2))]", (int*)&((uint * )bo_2)[(search_pos+(1096/2))], GPUFI_DATATYPE_UNKNOWN);


        ((uint * )bo_1)[search_pos]=((i00+i01)+(i10+i11));
        GPUFI_KERNEL_VARIABLE(gpufi_dev, 51, "((uint * )bo_1)[search_pos]", (int*)&((uint * )bo_1)[search_pos], GPUFI_DATATYPE_UNKNOWN);


    }
    /* Automatically inserted by Cetus_Cuda */
    GPUFI_KERNEL_LOOP(gpufi_dev, GPUFI_LOOP_END);
    /* Fault Injector Instrumentation End *//* End of Cetus_Cuda code */

    GPUFI_KERNEL_VARIABLE(gpufi_dev, 42, "search_pos", (int*)&search_pos, GPUFI_DATATYPE_INTEGER);


    GPUFI_KERNEL(gpufi_dev, GPUFI_KERNEL_END, 1, "larger_sad_calc_16");
}
